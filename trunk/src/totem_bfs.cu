#include "hip/hip_runtime.h"
/* TODO(lauro,abdullah,elizeu): Add license.
 *
 * This file contains an implementation of the breadth-first search (BFS) graph
 * search algorithm based as described in [Harish07].
 * [Harish07] P. Harish and P. Narayanan, "Accelerating large graph algorithms
 *   on the GPU using CUDA," in High Performance Computing - HiPC 2007,
 *   LNCS v. 4873, ch. 21, doi: http://dx.doi.org/10.1007/978-3-540-77220-0_21
 *
 *  Created on: 2011-02-28
 *      Author: Lauro Beltrão Costa
 */

// system includes
#include <hip/hip_runtime.h>

// totem includes
#include "totem_comdef.h"
#include "totem_comkernel.cuh"
#include "totem_graph.h"
#include "totem_mem.h"

// TODO(elizeu): TODO to change the type from uint32_t to id_t for
// graph->vertices and graph->edges.

/* This comment describes implementation details of the next two functions.

 Modified from [Harish07].
 Breadth First Search
 This implementation uses level synchronization. BFS traverses the
 graph in levels; once a level is visited it is not visited again.
 The BFS frontier corresponds to all the nodes being processed at the current
 level.
 Each thread process a vertex (and in the following text the terms are used
 in a similar way).
 Two boolean arrays, frontier and visited of size |V| stores the BFS frontier
 and the visited vertices. An integer array, cost, stores the minimal number
 of edges of each vertex from the source vertex S. In each iteration, each
 vertex looks at its entry in the frontier array. If true, it fetches its cost
 from the cost array and updates all the costs of its neighbors if more than
 its own cost plus one. The vertex removes its own entry from the frontier
 array and adds to the visited array. It also adds its neighbors to the
 frontier array if the neighbor is not already visited. This process is
 repeated until the frontier is empty (i.e., it does not contain any vertex to
 be processed).
 */
__global__
void bfs_kernel(graph_t graph, uint32_t level, bool* finished, uint32_t* cost) {
  const int vertex_id = THREAD_GLOBAL_INDEX;
  if (vertex_id >= graph.vertex_count) return;
  if (cost[vertex_id] != level) return;

  // TODO(lauro, abdullah): one optimization is to load the neighbors ids to
  // shared memory to facilitate  coalesced memory access.
  // for all neighbors of vertex_id
  for (uint32_t i = graph.vertices[vertex_id];
       i < graph.vertices[vertex_id + 1]; i++) {
    const uint32_t neighbor_id = graph.edges[i];
    if (cost[neighbor_id] == INFINITE) {
      // Threads may update finished and the same position in the cost array
      // concurrently. It does not affect correctness since all
      // threads would update with the same value.
      *finished = false;
      cost[neighbor_id] = level + 1;
    }
  } // for
}

// TODO(lauro): Add CHECK_ERR for CUDA functions.
// TODO(lauro): Return an error_t and have the a yuck out param.
__host__
uint32_t* bfs(uint32_t source_id, const graph_t* graph) {
  if( (graph == NULL) || (source_id >= graph->vertex_count) ) {
    return NULL;
  } else if( graph->vertex_count == 1 ) {
    uint32_t* cost = (uint32_t*) mem_alloc(sizeof(uint32_t));
    cost[0] = 0;
    return cost;
  }
  // TODO(lauro): More optimizations can be performed here. For example, if
  // there is no edge. It can return the cost array initialize as INFINITE.

  // Create graph on GPU memory.
  // TODO(lauro): Move to some utility library. We will often need this.
  graph_t graph_d = *graph;
  hipMalloc((void**) &(graph_d.vertices),
             (graph->vertex_count + 1) * sizeof(uint32_t));
  hipMalloc((void**) &(graph_d.edges),
             graph->edge_count * sizeof(uint32_t));
  hipMemcpy(graph_d.vertices, graph->vertices,
            (graph->vertex_count + 1)  * sizeof(uint32_t),
             hipMemcpyHostToDevice);
  hipMemcpy(graph_d.edges, graph->edges, graph->edge_count * sizeof(uint32_t),
             hipMemcpyHostToDevice);

  // TODO(lauro) Next three lines are not directly related to this function and
  // should have a better location.
  dim3 blocks;
  dim3 threads_per_block;
  KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);

  // Create cost array only on GPU.
  uint32_t* cost_d;
  hipMalloc((void**) &cost_d, graph->vertex_count * sizeof(uint32_t));
  // Initialize cost to INFINITE.
  memset_device<<<blocks, threads_per_block>>>(cost_d, INFINITE,
                                               graph->vertex_count);

  // For the source vertex, initialize cost.
  hipMemset(&(cost_d[source_id]), 0, sizeof(uint32_t));

  // while the current level have vertices to be processed.
  bool finished = false;
  bool* finished_d;
  hipMalloc((void**) &finished_d, sizeof(bool));
  for (uint32_t level = 0; !finished; level++) {
    hipMemset(finished_d, true, 1);
    // for each vertex V in parallel do
    bfs_kernel<<<blocks, threads_per_block>>>(graph_d, level, finished_d,
                                              cost_d);
    hipMemcpy(&finished, finished_d, sizeof(bool), hipMemcpyDeviceToHost);
  }

  hipFree(graph_d.vertices);
  hipFree(graph_d.edges);

  uint32_t* cost = (uint32_t*) mem_alloc(graph->vertex_count *
                                         sizeof(uint32_t));
  hipMemcpy(cost, cost_d, graph->vertex_count * sizeof(uint32_t),
             hipMemcpyDeviceToHost);
  hipFree(cost_d);
  return cost;
}
