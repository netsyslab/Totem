#include "hip/hip_runtime.h"
/**
 * Hybrid implementation of the incoming-based PageRank algorithm
 *
 *  Created on: 2012-09-02
 *  Author: Abdullah Gharaibeh
 */

// totem includes
#include "totem_alg.h"
#include "totem_engine.cuh"

/**
 * PageRank specific state
 */
typedef struct pagestate_s {
  rank_t* rank;
  rank_t* rank_s[MAX_PARTITION_COUNT];
  dim3 blocks;
  dim3 threads;
} page_rank_state_t;

/**
 * final result
 */
PRIVATE rank_t* rank_final = NULL;

/**
 * Used as a temporary buffer to host the final result produced by
 * GPU partitions
 */
PRIVATE rank_t* rank_host = NULL;

/**
 * Checks for input parameters and special cases. This is invoked at the
 * beginning of public interfaces (GPU and CPU)
*/
PRIVATE
error_t check_special_cases(float* rank, bool* finished) {
  *finished = true;
  if (engine_vertex_count() == 0) {
    return FAILURE;
  } else if (engine_vertex_count() == 1) {
    rank[0] = 1.0;
    return SUCCESS;
  }
  *finished = false;
  return SUCCESS;
}

/**
 * Sum the rank of the neighbors.
 * @param[in] graph the graph to apply page rank on
 * @param[in] rank an array storing the current rank of each vertex in the graph
 * @return sum of neighbors' ranks
 */
inline __device__ __host__
rank_t sum_neighbors_ranks(graph_t* graph, vid_t vid, page_rank_state_t* ps) {
  rank_t sum = 0;
  for (eid_t i = graph->vertices[vid];
       i < graph->vertices[vid + 1]; i++) {
    rank_t* nbr_rank_s = ps->rank_s[GET_PARTITION_ID(graph->edges[i])];
    sum += nbr_rank_s[GET_VERTEX_ID(graph->edges[i])];
  }
  return sum;
}

/**
 * The PageRank kernel. Based on the algorithm described in [Malewicz2010].
 * For each round, each vertex broadcasts along each outgoing edge its tentative
 * PageRank divided by the number of outgoing edges. The tentative PageRank of
 * vertex is calculated as follows: the vertex sums up the values arriving into
 * sum and sets its own tentative PageRank to
 * ((1 - DAMPING_FACTOR) / vertex_count + DAMPING_FACTOR * sum).
 * Broadcasting messages over outgoing edges is done as follows: the value is
 * placed in the outbox buffer. In the next round the inbox and outbox are
 * swapped, and the message will be accessed in the next round via the
 * inbox buffer. This operation simulates a broadcast because all the neighbors
 * of vertex v will access the same location (i.e., inbox[v]) to get the message
 * (i.e., tentative_PageRank_of_v/neighbor_count). In the last round, outbox
 * will contain the PageRank of each vertex.
 * @param[in] graph the graph to apply page rank on
 * @param[in] inbox messages broadcasted to vertices
 * @param[in] outbox messages to be broadcasted in the next round
 */
__global__
void page_rank_incoming_kernel(partition_t par, page_rank_state_t ps, 
                               vid_t vcount, bool last_round) {
  vid_t vid = THREAD_GLOBAL_INDEX;
  if (vid >= par.subgraph.vertex_count) return;
  rank_t sum = sum_neighbors_ranks(&par.subgraph, vid, &ps);
  rank_t my_rank = ((1 - PAGE_RANK_DAMPING_FACTOR) / vcount) + 
    (PAGE_RANK_DAMPING_FACTOR * sum);
  ps.rank[vid] = last_round ? my_rank : 
    my_rank / (par.subgraph.vertices[vid + 1] - par.subgraph.vertices[vid]);
}

PRIVATE void page_rank_incoming_gpu(partition_t* par, bool last_round) {
  page_rank_state_t* ps = (page_rank_state_t*)par->algo_state;
  page_rank_incoming_kernel<<<ps->blocks, ps->threads, 0,
    par->streams[1]>>>(*par, *ps, engine_vertex_count(), last_round);
  CALL_CU_SAFE(hipGetLastError());
}

PRIVATE void page_rank_incoming_cpu(partition_t* par, bool last_round) {
  page_rank_state_t* ps = (page_rank_state_t*)par->algo_state;
  graph_t* subgraph = &(par->subgraph);
  vid_t vcount = engine_vertex_count();  
  OMP(omp parallel for schedule(runtime))
  for(vid_t vid = 0; vid < subgraph->vertex_count; vid++) {
    rank_t sum = sum_neighbors_ranks(subgraph, vid, ps);
    rank_t my_rank = ((1 - PAGE_RANK_DAMPING_FACTOR) / vcount) + 
      (PAGE_RANK_DAMPING_FACTOR * sum);
    ps->rank[vid] = last_round ? my_rank : 
      my_rank / (subgraph->vertices[vid + 1] - subgraph->vertices[vid]);
  }
}

PRIVATE void page_rank_incoming(partition_t* par) {
  if (engine_superstep() > 1) {
    page_rank_state_t* ps = (page_rank_state_t*)par->algo_state;
    for (int pid = 0; pid < engine_partition_count(); pid++) {
      if (pid == par->id) continue;
      ps->rank_s[pid] = (rank_t*)par->outbox[pid].pull_values;
    }

    rank_t* tmp = ps->rank;
    ps->rank = ps->rank_s[par->id];
    ps->rank_s[par->id] = tmp;
    bool last_round = (engine_superstep() == (PAGE_RANK_ROUNDS + 1));
    if (par->processor.type == PROCESSOR_GPU) {
      page_rank_incoming_gpu(par, last_round);
    } else {
      assert(par->processor.type == PROCESSOR_CPU);
      page_rank_incoming_cpu(par, last_round);
    }
  }
  if (engine_superstep() < (PAGE_RANK_ROUNDS + 1)) {
    engine_report_not_finished();
  }
}

PRIVATE void page_rank_incoming_gather(partition_t* partition) {
  page_rank_state_t* ps = (page_rank_state_t*)partition->algo_state;
  engine_gather_inbox(partition->id, ps->rank);
}

PRIVATE void page_rank_incoming_aggr(partition_t* partition) {
  if (!partition->subgraph.vertex_count) return;
  page_rank_state_t* ps = (page_rank_state_t*)partition->algo_state;
  graph_t* subgraph = &partition->subgraph;
  rank_t* src_rank = NULL;
  if (partition->processor.type == PROCESSOR_GPU) {
    CALL_CU_SAFE(hipMemcpy(rank_host, ps->rank, 
                            subgraph->vertex_count * sizeof(rank_t),
                            hipMemcpyDefault));
    src_rank = rank_host;
  } else {
    assert(partition->processor.type == PROCESSOR_CPU);
    src_rank = ps->rank;
  }
  // aggregate the results
  for (vid_t v = 0; v < subgraph->vertex_count; v++) {
    rank_final[partition->map[v]] = src_rank[v];
  }
}

PRIVATE void page_rank_incoming_init(partition_t* par) {
  vid_t vcount = par->subgraph.vertex_count;
  if (vcount == 0) return;
  page_rank_state_t* ps = NULL;
  CALL_SAFE(totem_calloc(sizeof(page_rank_state_t), TOTEM_MEM_HOST, 
                         (void**)&ps));
  totem_mem_t type = TOTEM_MEM_HOST;
  if (par->processor.type == PROCESSOR_GPU) {
    type = TOTEM_MEM_DEVICE;
    KERNEL_CONFIGURE(vcount, ps->blocks, ps->threads);
  }
  /* for (int pid = 0; pid < engine_partition_count(); pid++) { */
  /*   if (pid == par->id) { */
  CALL_SAFE(totem_malloc(vcount * sizeof(rank_t), type, 
                         (void**)&(ps->rank_s[par->id])));
  /*   } else { */
  /*     ps->rank_s[pid] = (rank_t*)par->outbox[pid].pull_values; */
  /*   } */
  /* } */
  CALL_SAFE(totem_malloc(vcount * sizeof(rank_t), type, (void**)&(ps->rank)));
  rank_t init_value = 1 / (rank_t)engine_vertex_count();
  totem_memset(ps->rank, init_value, vcount, type, par->streams[1]);
  par->algo_state = ps;
}

PRIVATE void page_rank_incoming_finalize(partition_t* partition) {
  assert(partition->algo_state);
  page_rank_state_t* ps = (page_rank_state_t*)partition->algo_state;
  totem_mem_t type = TOTEM_MEM_HOST;
  if (partition->processor.type == PROCESSOR_GPU) {
    type = TOTEM_MEM_DEVICE;
  } 
  totem_free(ps->rank, type);
  totem_free(ps->rank_s[partition->id], type);
  totem_free(ps, TOTEM_MEM_HOST);
  partition->algo_state = NULL;
}

error_t page_rank_incoming_hybrid(float *rank_i, float* rank) {
  // check for special cases
  bool finished = false;
  error_t rc = check_special_cases(rank, &finished);
  if (finished) return rc;

  // initialize global state
  rank_final = rank;

  // initialize the engine
  engine_config_t config = {
    NULL, page_rank_incoming, NULL, page_rank_incoming_gather, 
    page_rank_incoming_init, page_rank_incoming_finalize, 
    page_rank_incoming_aggr, GROOVES_PULL
  };
  engine_config(&config);
  if (engine_largest_gpu_partition()) {
    CALL_SAFE(totem_malloc(engine_largest_gpu_partition() * sizeof(float), 
                           TOTEM_MEM_HOST_PINNED, (void**)&rank_host));
  }
  engine_execute();

  // clean up and return
  if (engine_largest_gpu_partition()) {
    totem_free(rank_host, TOTEM_MEM_HOST_PINNED);
  }
  return SUCCESS;
}
