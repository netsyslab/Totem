#include "hip/hip_runtime.h"
/**
 * Hybrid implementation of the incoming-based PageRank algorithm
 *
 *  Created on: 2012-09-02
 *  Author: Abdullah Gharaibeh
 */

// totem includes
#include "totem_alg.h"
#include "totem_engine.cuh"

/**
 * In case the vertex ids are sorted by degree, this constant specifies the 
 * number of groups of vertices of similar degree to be processed together
 */
PRIVATE const int section_count = 6;

/**
 * final result
 */
PRIVATE rank_t* rank_final = NULL;

/**
 * Used as a temporary buffer to host the final result produced by
 * GPU partitions
 */
PRIVATE rank_t* rank_host = NULL;

/**
 * A constant used as part of calculating the rank in each round. The value
 * depends on the number of vertices in the graph, and is equal to:
 * ((1 - PAGE_RANK_DAMPING_FACTOR) / vertex_count)
 */
PRIVATE rank_t c1 = 0;

/**
 * PageRank specific state
 */
typedef struct pagestate_s {
  rank_t* rank;
  rank_t* rank_s[MAX_PARTITION_COUNT];
  vid_t   sections[section_count + 1];
} page_rank_state_t;

/**
 * Checks for input parameters and special cases. This is invoked at the
 * beginning of public interfaces (GPU and CPU)
*/
PRIVATE
error_t check_special_cases(rank_t* rank, bool* finished) {
  *finished = true;
  if (engine_vertex_count() == 0) {
    return FAILURE;
  } else if (engine_vertex_count() == 1) {
    rank[0] = 1.0;
    return SUCCESS;
  }
  *finished = false;
  return SUCCESS;
}

template<int VWARP_WIDTH>
PRIVATE __device__ void 
sum_neighbors(const vid_t* __restrict nbrs, const vid_t nbr_count,
              rank_t** rank_s, rank_t* vwarp_rank, int warp_offset) {
  if (VWARP_WIDTH > 32) __syncthreads();
  rank_t sum = 0;
  for (vid_t i = warp_offset; i < nbr_count; i+= VWARP_WIDTH) {
    vid_t nbr = GET_VERTEX_ID(nbrs[i]);
    int nbr_pid = GET_PARTITION_ID(nbrs[i]);
    rank_t* nbr_rank = rank_s[nbr_pid];
    sum += nbr_rank[nbr];
  }
  vwarp_rank[warp_offset] = sum;
  if (VWARP_WIDTH > 32) __syncthreads();

  // completely unrolled parallel reduction
  if (warp_offset < VWARP_WIDTH / 2) {      
    // do reduction in shared mem
    if (VWARP_WIDTH > 1024) assert(false);
    if (VWARP_WIDTH == 1024) {
      if (warp_offset < 512) {
        vwarp_rank[warp_offset] = sum = sum + vwarp_rank[warp_offset + 512];
      }
      __syncthreads();
    }      
    if (VWARP_WIDTH >= 512) {
      if (warp_offset < 256) {
        vwarp_rank[warp_offset] = sum = sum + vwarp_rank[warp_offset + 256];
      }
      __syncthreads();
    }
    if (VWARP_WIDTH >= 256) {
      if (warp_offset < 128) {
        vwarp_rank[warp_offset] = sum = sum + vwarp_rank[warp_offset + 128];
      }
      __syncthreads();
    }
    if (VWARP_WIDTH >= 128) {
      if (warp_offset <  64) {
        vwarp_rank[warp_offset] = sum = sum + vwarp_rank[warp_offset + 64];
      }
      __syncthreads();
    }
    
    if (warp_offset < 32) {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile rank_t *smem = vwarp_rank;
      
      if (VWARP_WIDTH >= 64) {
        smem[warp_offset] = sum = sum + smem[warp_offset + 32];
      }
      if (VWARP_WIDTH >= 32) {
        smem[warp_offset] = sum = sum + smem[warp_offset + 16];
      }
      if (VWARP_WIDTH >= 16) {
        smem[warp_offset] = sum = sum + smem[warp_offset + 8];
      }
      if (VWARP_WIDTH >= 8) {
        smem[warp_offset] = sum = sum + smem[warp_offset + 4];
      }
      if (VWARP_WIDTH >= 4) {
        smem[warp_offset] = sum = sum + smem[warp_offset + 2];
      }
      if (VWARP_WIDTH >= 2) {
        smem[warp_offset] = sum = sum + smem[warp_offset + 1];
      }
    }
  }
}

/**
 * The PageRank kernel. Based on the algorithm described in [Malewicz2010].
 */
template<int VWARP_WIDTH, int VWARP_BATCH, int THREADS_PER_BLOCK>
PRIVATE __global__
void page_rank_incoming_kernel(partition_t par, page_rank_state_t ps, 
                               const vid_t start, const vid_t vertex_count,
                               const bool last_round, const rank_t c1) {
  if (THREAD_GLOBAL_INDEX >= 
      vwarp_thread_count(vertex_count, VWARP_WIDTH, VWARP_BATCH)) return;

  const eid_t* __restrict vertices = par.subgraph.vertices;

  vid_t start_vertex = start + 
    vwarp_block_start_vertex(VWARP_WIDTH, VWARP_BATCH) + 
    vwarp_warp_start_vertex(VWARP_WIDTH, VWARP_BATCH);
  vid_t end_vertex = start_vertex +
    vwarp_warp_batch_size(vertex_count, VWARP_WIDTH, VWARP_BATCH);
  int warp_offset = vwarp_thread_index(VWARP_WIDTH);

  // Each thread in every warp has an entry in the following array which will be
  // used to calculate intermediary delta values in shared memory
  __shared__ rank_t block_rank[THREADS_PER_BLOCK];
  int index = THREAD_BLOCK_INDEX / VWARP_WIDTH;
  rank_t* vwarp_rank = &block_rank[index * VWARP_WIDTH];

  for(vid_t v = start_vertex; v < end_vertex; v++) {
    const eid_t nbr_count = vertices[v + 1] - vertices[v];
    vid_t* nbrs = par.subgraph.edges + vertices[v];
    if (v >= par.subgraph.vertex_ext) {
      nbrs = par.subgraph.edges_ext + 
        (vertices[v] - par.subgraph.edge_count_ext);
    }
    sum_neighbors<VWARP_WIDTH>
      (nbrs, nbr_count, ps.rank_s, vwarp_rank, warp_offset);
    if (warp_offset == 0) {
      rank_t my_rank = c1 + (PAGE_RANK_DAMPING_FACTOR * vwarp_rank[0]);
      if (!last_round) {
        my_rank /= nbr_count;
      }
      ps.rank[v] = my_rank;
    }
  }
}

template<int VWARP_WIDTH, int BATCH_SIZE>
PRIVATE void
page_rank_gpu_launch(partition_t* par, page_rank_state_t* ps, 
                     const vid_t start, const vid_t vertex_count,
                     const bool last, const rank_t c1) {
  const int threads = MAX_THREADS_PER_BLOCK;
  dim3 blocks;
  assert(VWARP_WIDTH <= threads);
  kernel_configure(vwarp_thread_count(vertex_count, VWARP_WIDTH, BATCH_SIZE),
                   blocks, threads);
  page_rank_incoming_kernel<VWARP_WIDTH, BATCH_SIZE, threads> <<<blocks, 
    threads, 0, par->streams[1]>>>(*par, *ps, start, vertex_count, last, c1);
}

typedef void(*page_rank_gpu_func_t)(partition_t*, page_rank_state_t*, 
                                    const vid_t, const vid_t, const bool,
                                    const rank_t);

PRIVATE void page_rank_incoming_gpu_sorted(partition_t* par, bool last_round) {
  page_rank_state_t* ps = (page_rank_state_t*)par->algo_state;
  const page_rank_gpu_func_t section_launch[] = {
    page_rank_gpu_launch<1,   2>,    // (0) < 2
    page_rank_gpu_launch<2,   2>,    // (1) > 2    && < 8
    page_rank_gpu_launch<4,   4>,    // (2) > 8   && < 16
    page_rank_gpu_launch<8,   8>,    // (3) > 16  && < 64
    page_rank_gpu_launch<16,  8>,    // (4) > 64   && < 128
    page_rank_gpu_launch<32,  4>,    // (5) > 1024 && < 4 * 1024
    page_rank_gpu_launch<256, 2>,    // (6) >= 4 * 1024
  };

  int64_t end = par->subgraph.vertex_count;
  for (int i = section_count; i >= 0; i--) {
    int64_t start = ps->sections[i];
    int64_t count = end - start;
    if (count > 0) {
      stopwatch_t stopwatch;
      stopwatch_start(&stopwatch);
      section_launch[i](par, ps, start, count, last_round, c1);
      CALL_CU_SAFE(hipGetLastError());
      end = start;
    }
  }
}

PRIVATE void page_rank_incoming_gpu(partition_t* par, bool last_round) {
  if (engine_sorted()) {
    page_rank_incoming_gpu_sorted(par, last_round);
    return;      
  }

  PRIVATE const page_rank_gpu_func_t PAGE_RANK_GPU_FUNC[] = {
    // RANDOM algorithm
    page_rank_gpu_launch<VWARP_MEDIUM_WARP_WIDTH, VWARP_MEDIUM_BATCH_SIZE>,
    // HIGH partitioning
    page_rank_gpu_launch<VWARP_MEDIUM_WARP_WIDTH, VWARP_MEDIUM_BATCH_SIZE>,
    // LOW partitioning
    page_rank_gpu_launch<MAX_THREADS_PER_BLOCK, VWARP_MEDIUM_WARP_WIDTH>
  };
  page_rank_state_t* ps = (page_rank_state_t*)par->algo_state;  
  int par_alg = engine_partition_algorithm();
  PAGE_RANK_GPU_FUNC[par_alg](par, ps, 0, par->subgraph.vertex_count,
                              last_round, c1);
}


PRIVATE void page_rank_incoming_cpu(partition_t* par, bool last_round) {
  page_rank_state_t* ps = (page_rank_state_t*)par->algo_state;
  graph_t* subgraph = &(par->subgraph);
  vid_t vcount = engine_vertex_count();  

  OMP(omp parallel for schedule(runtime))
  for(vid_t vid = 0; vid < subgraph->vertex_count; vid++) {
    rank_t sum = 0;
    for (eid_t i = subgraph->vertices[vid];
         i < subgraph->vertices[vid + 1]; i++) {
      rank_t* nbr_rank_s = ps->rank_s[GET_PARTITION_ID(subgraph->edges[i])];
      sum += nbr_rank_s[GET_VERTEX_ID(subgraph->edges[i])];
    }
    rank_t my_rank = c1 + (PAGE_RANK_DAMPING_FACTOR * sum);
    if (!last_round) {
      my_rank /= (subgraph->vertices[vid + 1] - subgraph->vertices[vid]); 
    }
    ps->rank[vid] = my_rank;
  }  
}

PRIVATE void page_rank_incoming(partition_t* par) {
  if (!par->subgraph.vertex_count) return;
  if (engine_superstep() > 1) {
    page_rank_state_t* ps = (page_rank_state_t*)par->algo_state;
    for (int pid = 0; pid < engine_partition_count(); pid++) {
      if (pid != par->id) {
        ps->rank_s[pid] = (rank_t*)par->outbox[pid].pull_values;
      }
    }
    rank_t* tmp = ps->rank;
    ps->rank = ps->rank_s[par->id];
    ps->rank_s[par->id] = tmp;
    bool last_round = (engine_superstep() == (PAGE_RANK_ROUNDS + 1));
    if (par->processor.type == PROCESSOR_GPU) {
      page_rank_incoming_gpu(par, last_round);
    } else {
      assert(par->processor.type == PROCESSOR_CPU);
      page_rank_incoming_cpu(par, last_round);
    }
  }
  if (engine_superstep() < (PAGE_RANK_ROUNDS + 1)) {
    engine_report_not_finished();
  }
}

PRIVATE void page_rank_incoming_gather(partition_t* partition) {
  if (!partition->subgraph.vertex_count) return;
  page_rank_state_t* ps = (page_rank_state_t*)partition->algo_state;
  engine_gather_inbox(partition->id, ps->rank);
}

PRIVATE void page_rank_incoming_aggr(partition_t* partition) {
  if (!partition->subgraph.vertex_count) return;
  page_rank_state_t* ps = (page_rank_state_t*)partition->algo_state;
  graph_t* subgraph = &partition->subgraph;
  rank_t* src_rank = NULL;
  if (partition->processor.type == PROCESSOR_GPU) {
    CALL_CU_SAFE(hipMemcpy(rank_host, ps->rank, 
                            subgraph->vertex_count * sizeof(rank_t),
                            hipMemcpyDefault));
    src_rank = rank_host;
  } else {
    assert(partition->processor.type == PROCESSOR_CPU);
    src_rank = ps->rank;
  }
  // aggregate the results
  for (vid_t v = 0; v < subgraph->vertex_count; v++) {
    rank_final[partition->map[v]] = src_rank[v];
  }
}

PRIVATE __global__
void sections_build_kernel(const eid_t* __restrict vertices, 
                           vid_t vertex_count, vid_t* sections) {
  const vid_t v = THREAD_GLOBAL_INDEX;
  if (v >= vertex_count) return;
  vid_t nbr_count = vertices[v + 1] - vertices[v];

  if (nbr_count > 1 && nbr_count < 8) {
    atomicMin(&sections[0], v);
  }
  if (nbr_count > 7 && nbr_count < 16) {
    atomicMin(&sections[1], v);
  }
  if (nbr_count > 15 && nbr_count < 64) {
    atomicMin(&sections[2], v);
  }
  if (nbr_count > 63 && nbr_count < 128) {
    atomicMin(&sections[3], v);
  }
  if (nbr_count > 127 && nbr_count < (4 * 1024)) {
    atomicMin(&sections[4], v);
  }
  if (nbr_count >= (4 * 1024)) {
    atomicMin(&sections[5], v);
  }
}

PRIVATE void page_rank_incoming_init(partition_t* par) {
  vid_t vcount = par->subgraph.vertex_count;
  if (vcount == 0) return;
  page_rank_state_t* ps = NULL;
  CALL_SAFE(totem_calloc(sizeof(page_rank_state_t), TOTEM_MEM_HOST, 
                         (void**)&ps));
  totem_mem_t type = TOTEM_MEM_HOST;
  if (par->processor.type == PROCESSOR_GPU) {
    type = TOTEM_MEM_DEVICE;
    if (engine_sorted()) {
      vid_t* sections = NULL;
      CALL_SAFE(totem_malloc(section_count * sizeof(vid_t), type, 
                             (void**)&sections));
      CALL_SAFE(totem_memset(sections, vcount, section_count, 
                             TOTEM_MEM_DEVICE, par->streams[1]));
      dim3 blocks;
      kernel_configure(vcount, blocks);
      sections_build_kernel<<<blocks, DEFAULT_THREADS_PER_BLOCK, 0, 
        par->streams[1]>>>(par->subgraph.vertices, vcount, sections);
      CALL_CU_SAFE(hipMemcpyAsync(&ps->sections[1], sections,
                                   section_count * sizeof(vid_t),
                                   hipMemcpyDeviceToHost, par->streams[1]));
      ps->sections[0] = 0;
      CALL_CU_SAFE(hipStreamSynchronize(par->streams[1]));
      totem_free(sections, type);
    }
  }

  CALL_SAFE(totem_malloc(vcount * sizeof(rank_t), type, 
                         (void**)&(ps->rank_s[par->id])));
  CALL_SAFE(totem_malloc(vcount * sizeof(rank_t), type, (void**)&(ps->rank)));
  rank_t init_value = 1 / (rank_t)engine_vertex_count();
  totem_memset(ps->rank, init_value, vcount, type, par->streams[1]);
  CALL_CU_SAFE(hipStreamSynchronize(par->streams[1]));
  par->algo_state = ps;
}

PRIVATE void page_rank_incoming_finalize(partition_t* partition) {
  if (!partition->subgraph.vertex_count) return;
  assert(partition->algo_state);
  page_rank_state_t* ps = (page_rank_state_t*)partition->algo_state;
  totem_mem_t type = TOTEM_MEM_HOST;
  if (partition->processor.type == PROCESSOR_GPU) {
    type = TOTEM_MEM_DEVICE;
  } 
  totem_free(ps->rank, type);
  totem_free(ps->rank_s[partition->id], type);
  totem_free(ps, TOTEM_MEM_HOST);
  partition->algo_state = NULL;
}

error_t page_rank_incoming_hybrid(rank_t* rank_i, rank_t* rank) {
  // check for special cases
  bool finished = false;
  error_t rc = check_special_cases(rank, &finished);
  if (finished) return rc;

  // initialize global state
  rank_final = rank;
  c1 = ((1 - PAGE_RANK_DAMPING_FACTOR) / ((double)engine_vertex_count()));

  // initialize the engine
  engine_config_t config = {
    NULL, page_rank_incoming, NULL, page_rank_incoming_gather, 
    page_rank_incoming_init, page_rank_incoming_finalize, 
    page_rank_incoming_aggr, GROOVES_PULL
  };
  engine_config(&config);
  if (engine_largest_gpu_partition()) {
    CALL_SAFE(totem_malloc(engine_largest_gpu_partition() * sizeof(rank_t), 
                           TOTEM_MEM_HOST_PINNED, (void**)&rank_host));
  }
  engine_execute();

  // clean up and return
  if (engine_largest_gpu_partition()) {
    totem_free(rank_host, TOTEM_MEM_HOST_PINNED);
  }
  return SUCCESS;
}
