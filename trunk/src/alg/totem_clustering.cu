#include "hip/hip_runtime.h"
/**
 *
 *  Implements Clustering Coefficient algorithm for CPU and GPU.
 *
 *  Created on: 2014-02-03
 *  Author: Tahsin Arafat Reza 
 */

// totem includes
#include "totem_alg.h"

/**
 * Checks for input parameters and special cases. This is invoked at the
 * beginning of public interfaces (CPU and GPU).
 *
 * @param[in] graph
 * @param[in] finished
 * @param[in] coefficients
 */
PRIVATE
error_t check_special_cases(const graph_t* graph, bool* finished,
                            weight_t** coefficients) {
  // Check whether the graph is null or vertex set is empty
  if ((graph == NULL) || (graph->vertex_count == 0) ||
      (coefficients == NULL)) {
    return FAILURE;
  }

  // Check whether the edge set is empty
  if (graph->edge_count == 0) {
    totem_malloc(graph->vertex_count * sizeof(weight_t), TOTEM_MEM_HOST_PINNED,
                 reinterpret_cast<void**>(coefficients));
    memset(*coefficients, (weight_t)0.0, graph->vertex_count
           * sizeof(weight_t));
    return SUCCESS;
  }

  *finished = false;
  return SUCCESS;
}

/**
 * Initialize GPU.
 *
 * @param[in] graph
 * @param[in] distance_length
 * @param[out] graph_d
 * @param[out] coefficients_d
 */
PRIVATE
error_t initialize_gpu(const graph_t* graph, graph_t** graph_d,
                       weight_t** coefficients_d) {
  totem_mem_t type = TOTEM_MEM_DEVICE;

  // Transfer the graph to the device memory
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);

  // Allocate memory for the coefficients array
  CHK_SUCCESS(totem_malloc(graph->vertex_count * sizeof(weight_t), type,
                           reinterpret_cast<void**>(coefficients_d)),
                           err_free_graph);

  // Set cofficients to zero
  totem_memset(*coefficients_d, (weight_t)0.0, graph->vertex_count, type);

  return SUCCESS;

  // Error handlers
  err_free_graph:
    graph_finalize_device(*graph_d);
  err:
    return FAILURE;
}

/**
 * Finalize GPU.
 *
 * @param[in] graph
 * @param[in] distance_length
 * @param[out] graph_d
 * @param[out] coefficients_d
 */
PRIVATE
error_t finalize_gpu(graph_t* graph_d, weight_t* coefficients_d,
                     weight_t* coefficients) {
  // Copy the pointer to the output paramenter
  CHK_CU_SUCCESS(hipMemcpy(coefficients, coefficients_d,
    graph_d->vertex_count * sizeof(weight_t), hipMemcpyDeviceToHost), err);

  // Release allocated memory for results
  totem_free(coefficients_d, TOTEM_MEM_DEVICE);
  graph_finalize_device(graph_d);
  return SUCCESS;

  // Error handlers
  err:
    return FAILURE;
}

/**
 * Implements the GPU kernel function.
 *
 * @param[in] graph the input graph
 * @param[out] clustering_coefficients array containing
 * computed coefficients
 */
__global__
void clustering_coefficient_kernel(graph_t graph,
                                   weight_t* clustering_coefficients) {
  const vid_t v = THREAD_GLOBAL_INDEX;
  if (v >= graph.vertex_count) return;

  vid_t triangle_count_v = 0;

  for (eid_t e = graph.vertices[v]; e < graph.vertices[v + 1]; e++) {
    vid_t e_v = graph.edges[e];  // v's neighbour
    for (eid_t f = graph.vertices[e_v]; f < graph.vertices[e_v + 1]; f++) {
      vid_t f_e_v = graph.edges[f];  // Neighbour of v's neighbour
      for (eid_t u = graph.vertices[v]; u < graph.vertices[v + 1]; u++) {
        vid_t u_v = graph.edges[u];  // v's neighbour
        if (u_v != e_v && u_v == f_e_v) {  // Common neighbour verification
          triangle_count_v++;
        }
      }  // for
    }  // for
  }  // for

  vid_t degree_v = graph.vertices[v + 1] - graph.vertices[v];
  weight_t cc = 0.0f;  // Clustering Coefficient
  weight_t triangles_v = (weight_t)triangle_count_v/2.0f;
  if (triangles_v > 0.0f) {
    cc = (2.0f * triangles_v) / (((weight_t)degree_v - 1.0f) *
                                 (weight_t)degree_v);
  }
  clustering_coefficients[v] = cc;
}

/**
 * Implements the GPU kernel function.
 *
 * @param[in] graph the input graph
 * @param[out] clustering_coefficients array containing
 * computed coefficients
 */

__global__
void clustering_coefficient_sorted_neighbours_kernel(graph_t graph,
  weight_t* clustering_coefficients) {
  const vid_t v = THREAD_GLOBAL_INDEX;
  if (v >= graph.vertex_count) return;

  uint64_t triangle_count_v = 0;
  eid_t degree_v = graph.vertices[v + 1] - graph.vertices[v];

  for (eid_t e = graph.vertices[v]; e < graph.vertices[v + 1]; e++) {
    vid_t e_v = graph.edges[e];  // v's neighbour
    eid_t degree_e_v = graph.vertices[e_v + 1] - graph.vertices[e_v];

    // Calculate intersection of v's neighbour list and e_v's neighbour list
    vid_t large_index, small_index, i, j;
    if (degree_v >= degree_e_v) {
      large_index = v;
      small_index = e_v;
      i = e;
      j = graph.vertices[small_index];
    } else {
      large_index = e_v;
      small_index = v;
      i = graph.vertices[large_index];
      j = e;
    }

    if (graph.edges[i] >
        graph.edges[graph.vertices[small_index + 1] - 1] ||
        graph.edges[j] > graph.edges[graph.vertices[large_index + 1] - 1])
        break;

    for (; j < graph.vertices[small_index + 1]; ) {
      if (graph.edges[i] == graph.edges[j]) {
        i++; j++; triangle_count_v++;
      } else if (graph.edges[i] > graph.edges[j]) {
        j++;
      } else if (graph.edges[i] < graph.edges[j]) {
        i++;
      }
      if (i == (graph.vertices[large_index + 1])) break;
    }  // for
  }  // for
  if (triangle_count_v > 0) {
    clustering_coefficients[v] =
    ((double)(2 * triangle_count_v)) / ((double)(degree_v - 1) * degree_v);
  }
}

/**
 * Implements the GPU-only clustering coefficient algorithm.
 *
 * Given a graph \f$G = (V, E)\f$, the clustering coefficient (\f$CC\f$) 
 * of a vertex \f$v\inV\f$ with degree \f$d\f$ is defined as 
 * \f$CC = 2*T / d(d-1)f$, where \f$Tf$ is the number of triangles 
 * incident on \f$v\f$.
 *
 * This algorithm utilizes GPU cores according to kernel launch
 * configuration (number of blocks, threads per block etc.). Each vertex
 * performs computation (defined by the kernel function) in a unique thread.
 * Computaions of triangle counting and clustering coeffienct involves
 * writing to thread local variables only. Therefore, vertices can carryout
 * computation in parallel without interrupting each other. Each vertex
 * iterates through the list of its neighbours (and the list of neighbours
 * of each of its neighbours) in a sequential manner, to verify presence of
 * a common neigbour; hence, count trainagles and calculate clustering
 * coefficeint.
 *
 * @param[in] graph the input graph
 * @param[out] coefficients array containing computed coefficients
 */
error_t clustering_coefficient_gpu(const graph_t* graph,
                                   weight_t** coefficients) {
  // Check inputs
  bool finished = true;
  error_t rc = check_special_cases(graph, &finished, coefficients);
  if (finished) return rc;

  // Initialize GPU states
  graph_t* graph_d;
  weight_t* coefficients_d;
  CHK_SUCCESS(initialize_gpu(graph, &graph_d, &coefficients_d), err);
  {
  dim3 block_count, threads_per_block;
  KERNEL_CONFIGURE(graph->vertex_count, block_count, threads_per_block);
  clustering_coefficient_kernel<<<block_count, threads_per_block>>>
    (*graph_d, coefficients_d);
  }

  // Finalize GPU states
  // Copy the calculated coefficients from the
  // device memory to the host memory
  CHK_SUCCESS(finalize_gpu(graph_d, coefficients_d, *coefficients),
    err_free_all);

  return SUCCESS;

  // Error handlers
  err_free_all:
    totem_free(coefficients_d, TOTEM_MEM_DEVICE);
    graph_finalize_device(graph_d);
  err:
    return FAILURE;
}

/**
 * Implements the GPU-only clustering coefficient algorithm.
 * The implementation assumes that the neighbour list is sorted in increasing
 * order with respect to vertex ID.
 * @param[in] graph the input graph
 * @param[out] coefficients array containing computed coefficients
 */

error_t clustering_coefficient_sorted_neighbours_gpu(const graph_t* graph,
                                                     weight_t** coefficients) {
  // Check inputs
  bool finished = true;
  error_t rc = check_special_cases(graph, &finished, coefficients);
  if (finished) return rc;

  // Initialize GPU states
  graph_t* graph_d;
  weight_t* coefficients_d;
  CHK_SUCCESS(initialize_gpu(graph, &graph_d, &coefficients_d), err);
  {
  dim3 block_count, threads_per_block;
  KERNEL_CONFIGURE(graph->vertex_count, block_count, threads_per_block);
  clustering_coefficient_sorted_neighbours_kernel
    <<<block_count, threads_per_block>>>(*graph_d, coefficients_d);
  }

  // Finalize GPU states
  // Copy the calculated coefficients from the
  // device memory to the host memory
  CHK_SUCCESS(finalize_gpu(graph_d, coefficients_d, *coefficients),
    err_free_all);

  return SUCCESS;

  // Error handlers
  err_free_all:
    totem_free(coefficients_d, TOTEM_MEM_DEVICE);
    graph_finalize_device(graph_d);
  err:
    return FAILURE;
}

/**
 * Implements the CPU-only clustering coefficient algorithm.
 *
 * Given a graph \f$G = (V, E)\f$, the clustering coefficient (\f$CC\f$)
 * of a vertex \f$v\inV\f$ with degree \f$d\f$ is defined as
 * \f$CC = 2*T / d(d-1)f$, where \f$Tf$ is the number of triangles
 * incident on \f$v\f$.
 *
 * The outmost loop is parallelized with OpenMP. Each vertex performs
 * computation in a unique thread. Computaions of triangle
 * counting and clustering coeffienct involves writing to thread
 * local variables only. Therefore, vertices can carryout computation in
 * parallel without interrupting each other. Each vertex iterates through
 * the list of its neighbours (and the list of neighbours of each of its
 * neighbours) in a sequential manner, to verify presence of a common neigbour;
 * hence, count trainagles and calculate clustering coefficeint.
 *
 * @param[in] graph the input graph
 * @param[out] coefficients array containing computed coefficients
 */
error_t clustering_coefficient_cpu(const graph_t* graph,
                                   weight_t** coefficients) {
  // Check inputs
  bool finished = true;
  error_t rc = check_special_cases(graph, &finished, coefficients);
  if (finished) return rc;

  // Allocate memory for the results
  weight_t* clustering_coefficients = NULL;
  totem_malloc(graph->vertex_count * sizeof(weight_t), TOTEM_MEM_HOST_PINNED,
               reinterpret_cast<void**>(&clustering_coefficients));

  memset(clustering_coefficients, 0.0, graph->vertex_count * sizeof(weight_t));

  OMP(omp parallel for)
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    vid_t triangle_count_v = 0;
    for (eid_t e = graph->vertices[v]; e < graph->vertices[v + 1]; e++) {
      vid_t e_v = graph->edges[e];  // v's neighbour
      for (eid_t f = graph->vertices[e_v]; f < graph->vertices[e_v + 1]; f++) {
        vid_t f_e_v = graph->edges[f];  // Neighbour of v's neighbour
        for (eid_t u = graph->vertices[v]; u < graph->vertices[v + 1]; u++) {
          vid_t u_v = graph->edges[u];  // v's neighbour
          if (u_v != e_v && u_v == f_e_v) {  // Common neighbour verification
            triangle_count_v++;
          }
        }  // for
      }  // for
    }  // for

    vid_t degree_v = graph->vertices[v + 1] - graph->vertices[v];

    clustering_coefficients[v] = 0.0f;

    weight_t triangles_v = (weight_t)triangle_count_v/2.0f;

    if (triangles_v > 0.0f) {
      clustering_coefficients[v] =
        (2.0f * triangles_v) / (((weight_t)degree_v - 1.0f) *
                                (weight_t)degree_v);
    }
  }  // parallel for

  *coefficients = clustering_coefficients;
  return SUCCESS;
}

/**
 * Implements the CPU-only clustering coefficient algorithm.
 * The implementation assumes that the neighbour list is sorted in increasing
 * order with respect to vertex ID.
 * @param[in] graph the input graph
 * @param[out] coefficients array containing computed coefficients
 */

error_t clustering_coefficient_sorted_neighbours_cpu(const graph_t* graph,
                                                     weight_t** coefficients) {
  // Check inputs
  bool finished = true;
  error_t rc = check_special_cases(graph, &finished, coefficients);
  if (finished) return rc;

  // Allocate memory for the results
  weight_t* clustering_coefficients = NULL;
  totem_malloc(graph->vertex_count * sizeof(weight_t), TOTEM_MEM_HOST_PINNED,
               reinterpret_cast<void**>(&clustering_coefficients));

  memset(clustering_coefficients, (weight_t)0.0,
         graph->vertex_count * sizeof(weight_t));
  OMP(omp parallel for schedule(runtime))
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    uint64_t triangle_count_v = 0;
    eid_t degree_v = graph->vertices[v + 1] - graph->vertices[v];

    for (eid_t e = graph->vertices[v]; e < graph->vertices[v + 1]; e++) {
      vid_t e_v = graph->edges[e];  // v's neighbour
      eid_t degree_e_v = graph->vertices[e_v + 1] - graph->vertices[e_v];

      // Calculate intersection of v's neighbour list and e_v's neighbour list
      vid_t large_index, small_index, i, j;
      if (degree_v >= degree_e_v) {
        large_index = v;
        small_index = e_v;
        i = e;
        j = graph->vertices[small_index];
      } else {
        large_index = e_v;
        small_index = v;
        i = graph->vertices[large_index];
        j = e;
      }

      if (graph->edges[i] >
          graph->edges[graph->vertices[small_index + 1] - 1] ||
          graph->edges[j] > graph->edges[graph->vertices[large_index + 1] - 1])
          break;

      for (; j < graph->vertices[small_index + 1]; ) {
        if (graph->edges[i] == graph->edges[j]) {
          i++; j++; triangle_count_v++;
        } else if (graph->edges[i] > graph->edges[j]) {
          j++;
        } else if (graph->edges[i] < graph->edges[j]) {
          i++;
        }
        if (i == (graph->vertices[large_index + 1])) break;
      }  // for
    }  // for

    if (triangle_count_v > 0) {
      clustering_coefficients[v] =
        ((double)(2 * triangle_count_v)) / ((double)(degree_v - 1) *
        degree_v);
    }
  }  // parallel for

  *coefficients = clustering_coefficients;
  return SUCCESS;
}

/**
 * Summary of what works and what does not work:
 *
 * "clustering_coefficient_cpu" and "clustering_coefficient_gpu" do not require
 * graphs with sorted neighbour list but show very poor performnace for large
 * graphs.
 * 
 * "clustering_coefficient_sorted_neighbours_cpu" and
 * "clustering_coefficient_sorted_neighbours_gpu" require graphs with sorted
 * neighbour list.
 *
 * Architecture   Algorithm     Graph-type     Graph-scale       Status
 * CPU                cc          random            20        does not work
 * CPU          cc-sorted-nbrs  sorted-nbrs         20            works
 * GPU                cc          random            20        does not work
 * GPU          cc-sorted-nbrs  sorted-nbrs         20            works
 * CPU                cc          random            21        does not work
 * CPU          cc-sorted-nbrs  sorted-nbrs         21            works
 * GPU                cc          random            21        does not work
 * GPU          cc-sorted-nbrs  sorted-nbrs         21            works
 * CPU                cc          random            22        does not work
 * CPU          cc-sorted-nbrs  sorted-nbrs         22            works
 * GPU                cc          random            22        does not work
 * GPU          cc-sorted-nbrs  sorted-nbrs         22            works
 * CPU                cc          random            23        does not work
 * CPU          cc-sorted-nbrs  sorted-nbrs         23            works
 * GPU                cc          random            23        does not work
 * GPU          cc-sorted-nbrs  sorted-nbrs         23         intermittent
 * CPU                cc          random            24        does not work
 * CPU          cc-sorted-nbrs  sorted-nbrs         24            works
 * GPU                cc          random            24        does not work
 * GPU          cc-sorted-nbrs  sorted-nbrs         24         intermittent
 */
