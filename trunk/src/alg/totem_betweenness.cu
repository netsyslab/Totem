#include "hip/hip_runtime.h"
/**
 *  Defines Betweenness Centrality functions for both CPU and GPU.
 *
 *  Created on: 2012-05-24
 *  Author: Greg Redekop
 */

// totem includes
#include "totem_alg.h"
#include "totem_centrality.h"

/**
 * Allocates and initializes memory on the GPU for the successors implementation
 * of betweenness centrality.
 */
PRIVATE
error_t initialize_succs_gpu(const graph_t* graph, uint64_t vertex_count,
                             graph_t** graph_d, vid_t** sigma_d,
                             int32_t** dists_d, vid_t** succ_d,
                             uint32_t** succ_count_d, vid_t** stack_d,
                             uint32_t** stack_count_d, score_t** delta_d,
                             bool** finished_d,
                             score_t** betweenness_centrality_d) {
  // Allocate space on GPU
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);
  CHK_CU_SUCCESS(hipMalloc((void**)sigma_d, vertex_count * sizeof(vid_t)),
                 err_free_graph_d);
  CHK_CU_SUCCESS(hipMalloc((void**)dists_d, vertex_count * sizeof(int32_t)),
                 err_free_sigma_d);
  CHK_CU_SUCCESS(hipMalloc((void**)succ_d, graph->edge_count * sizeof(vid_t)),
                 err_free_dists_d);
  CHK_CU_SUCCESS(hipMalloc((void**)succ_count_d, vertex_count
                            * sizeof(uint32_t)), err_free_succ_d);
  CHK_CU_SUCCESS(hipMalloc((void**)stack_d, vertex_count * vertex_count
                            * sizeof(vid_t)), err_free_succ_count_d);
  CHK_CU_SUCCESS(hipMalloc((void**)stack_count_d, vertex_count
                            * sizeof(uint32_t)), err_free_stack_d);
  CHK_CU_SUCCESS(hipMalloc((void**)delta_d, vertex_count * sizeof(score_t)),
                 err_free_stack_count_d);
  CHK_CU_SUCCESS(hipMalloc((void**)finished_d, sizeof(bool)),
                 err_free_delta_d);
  CHK_CU_SUCCESS(hipMalloc((void**)betweenness_centrality_d, vertex_count
                            * sizeof(score_t)), err_free_finished_d);

  // Setup initial parameters
  CHK_CU_SUCCESS(hipMemset(*betweenness_centrality_d, (score_t)0.0,
                            vertex_count * sizeof(score_t)), err_free_all);
  return SUCCESS;

 err_free_all:
  hipFree(betweenness_centrality_d);
 err_free_finished_d:
  hipFree(finished_d);
 err_free_delta_d:
  hipFree(delta_d);
 err_free_stack_count_d:
  hipFree(stack_count_d);
 err_free_stack_d:
  hipFree(stack_d);
 err_free_succ_count_d:
  hipFree(succ_count_d);
 err_free_succ_d:
  hipFree(succ_d);
 err_free_dists_d:
  hipFree(dists_d);
 err_free_sigma_d:
  hipFree(sigma_d);
 err_free_graph_d:
  graph_finalize_device(*graph_d);
 err:
  return FAILURE;
}

/**
 * Allocates and initializes memory on the GPU for the predecessors
 * implementation of betweenness centrality.
 */
PRIVATE
error_t initialize_preds_gpu(const graph_t* graph, uint64_t vertex_count,
                             vid_t* r_edges, graph_t** graph_d, 
                             vid_t** r_edges_d, bool** preds_d, 
                             vid_t** sigma_d, int32_t** dist_d,
                             score_t** delta_d, bool** finished_d,
                             score_t** betweenness_centrality_d) {
  // Allocate space on GPU
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);
  CHK_CU_SUCCESS(hipMalloc((void**)r_edges_d, graph->edge_count
                            * sizeof(vid_t)), err_free_graph_d);
  CHK_CU_SUCCESS(hipMalloc((void**)preds_d, graph->edge_count * sizeof(vid_t)),
                 err_free_r_edges_d);
  CHK_CU_SUCCESS(hipMalloc((void**)sigma_d, vertex_count * sizeof(vid_t)),
                 err_free_preds_d);
  CHK_CU_SUCCESS(hipMalloc((void**)dist_d, vertex_count * sizeof(int32_t)),
                 err_free_sigma_d);
  CHK_CU_SUCCESS(hipMalloc((void**)delta_d, vertex_count * sizeof(score_t)),
                 err_free_dist_d);
  CHK_CU_SUCCESS(hipMalloc((void**)finished_d, sizeof(bool)),
                 err_free_delta_d);
  CHK_CU_SUCCESS(hipMalloc((void**)betweenness_centrality_d,
                             graph->vertex_count * sizeof(score_t)),
                 err_free_finished_d);

  // Setup initial parameters
  CHK_CU_SUCCESS(hipMemcpy(*r_edges_d, r_edges, graph->edge_count
                            * sizeof(vid_t), hipMemcpyHostToDevice),
                 err_free_all);
  CHK_CU_SUCCESS(hipMemset(*betweenness_centrality_d, 0, graph->vertex_count
                            * sizeof(score_t)), err_free_all);
  return SUCCESS;

 err_free_all:
  hipFree(betweenness_centrality_d);
 err_free_finished_d:
  hipFree(finished_d);
 err_free_delta_d:
  hipFree(delta_d);
 err_free_dist_d:
  hipFree(dist_d);
 err_free_sigma_d:
  hipFree(sigma_d);
 err_free_preds_d:
  hipFree(preds_d);
 err_free_r_edges_d:
  hipFree(r_edges_d);
 err_free_graph_d:
  graph_finalize_device(*graph_d);
 err:
  return FAILURE;
}

/**
 * Finalize function for the successor stack GPU implementation. It allocates
 * the host output buffer, moves the final results from GPU to the host buffers
 * and frees up GPU resources.
 */
PRIVATE
error_t finalize_succs_gpu(graph_t* graph_d, vid_t* sigma_d, int32_t* dist_d,
                           vid_t* succ_d, uint32_t* succ_count_d, 
                           vid_t* stack_d, uint32_t* stack_count_d, 
                           score_t* delta_d, bool* finished_d, 
                           score_t* betweenness_centrality_d,
                           score_t* betweenness_centrality) {
  // Copy back the centrality scores
  CHK_CU_SUCCESS(hipMemcpy(betweenness_centrality, betweenness_centrality_d,
                            graph_d->vertex_count * sizeof(score_t),
                            hipMemcpyDeviceToHost), err);
  graph_finalize_device(graph_d);
  hipFree(sigma_d);
  hipFree(dist_d);
  hipFree(succ_d);
  hipFree(succ_count_d);
  hipFree(stack_d);
  hipFree(stack_count_d);
  hipFree(delta_d);
  hipFree(finished_d);
  hipFree(betweenness_centrality_d);
  return SUCCESS;

 err:
  return FAILURE;
}

/**
 * Finalize function for the predecessor map GPU implementation. It allocates
 * the host output buffer, moves the final results from GPU to the host buffers
 * and frees up GPU resources.
 */
PRIVATE
error_t finalize_preds_gpu(graph_t* graph_d, vid_t* r_edges_d, bool* preds_d,
                           vid_t* sigma_d, int32_t* dist_d, score_t* delta_d,
                           bool* finished_d, score_t* betweenness_centrality_d,
                           score_t* betweenness_centrality) {
  // Copy back the centrality scores
  CHK_CU_SUCCESS(hipMemcpy(betweenness_centrality, betweenness_centrality_d,
                            graph_d->vertex_count * sizeof(score_t),
                            hipMemcpyDeviceToHost), err);
  graph_finalize_device(graph_d);
  hipFree(r_edges_d);
  hipFree(preds_d);
  hipFree(sigma_d);
  hipFree(dist_d);
  hipFree(delta_d);
  hipFree(finished_d);
  hipFree(betweenness_centrality_d);
  return SUCCESS;

 err:
  return FAILURE;
}

/**
 * This kernel is invoked for each iteration of the successors GPU betweenness
 * algorithm. It re-initializes variables for the SSSP problem using a different
 * source vertex.
 */
__global__
void unweighted_bc_succs_init_kernel(vid_t source, vid_t* sigma, int32_t* dist,
                                     uint32_t* stack_count, vid_t* stack) {
  sigma[source] = 1;
  dist[source] = 0;
  stack_count[0] = 1;
  stack[0] = source;
}

/**
 * This kernel is invoked for each iteration of the predecessors GPU betweenness
 * algorithm. It re-initializes variables for the SSSP problem using a different
 * source vertex.
 */
__global__
void unweighted_bc_preds_init_kernel(vid_t source, int32_t* dist, 
                                     vid_t* sigma) {
  dist[source] = 0;
  sigma[source] = 1;
}

/**
 * For each iteration of the successors GPU betweenness algorithm, we have to
 * reset all the variables and setup the initial parameters for the SSSP problem
 * using the new source vertex.
 */
PRIVATE
error_t unweighted_succs_init(const graph_t* graph, vid_t source, vid_t* sigma,
                              int32_t* dist, vid_t* succ, uint32_t* succ_count,
                              vid_t* stack, uint32_t* stack_count,
                              score_t* delta) {
  // Perform the memsets directly on the GPU
  dim3 blocks;
  dim3 threads_per_block;
  CHK_CU_SUCCESS(hipMemset(succ, 0, graph->edge_count * sizeof(vid_t)), err);
  CHK_CU_SUCCESS(hipMemset(stack, 0, graph->vertex_count * graph->vertex_count
                            * sizeof(vid_t)), err);
  CHK_CU_SUCCESS(hipMemset(succ_count, 0, graph->vertex_count
                            * sizeof(uint32_t)), err);
  CHK_CU_SUCCESS(hipMemset(stack_count, 0, graph->vertex_count
                            * sizeof(uint32_t)), err);
  CHK_CU_SUCCESS(hipMemset(sigma, 0, graph->vertex_count * sizeof(vid_t)), 
                 err);
  CHK_CU_SUCCESS(hipMemset(dist, -1, graph->vertex_count * sizeof(int32_t)),
                 err);
  CHK_CU_SUCCESS(hipDeviceSynchronize(), err);

  // Initialize the appropriate counts for the source vertex
  KERNEL_CONFIGURE(1, blocks, threads_per_block);
  unweighted_bc_succs_init_kernel<<<blocks, threads_per_block>>>
    (source, sigma, dist, stack_count, stack);
  CALL_CU_SAFE(hipGetLastError());
  return SUCCESS;

 err:
  return FAILURE;
}

/**
 * Unweighted betweenness centrality dependence accumulation kernel for the
 * successors stack GPU implementation. After finding the APSP solution and
 * counts of shortest paths through each node, this function calculates the
 * dependence for each node.
 */
__global__ void
unweighted_dep_acc_kernel(graph_t graph, int64_t phase, uint32_t* stack_count, 
                          vid_t* sigma, vid_t* stack, vid_t* succ, 
                          uint32_t* succ_count, score_t* delta, 
                          score_t* betweenness_centrality) {
  const vid_t thread_id = THREAD_GLOBAL_INDEX;

  if (thread_id < stack_count[phase]) {
    vid_t w = stack[graph.vertex_count * phase + thread_id];
    score_t dsw = 0.0;
    score_t sw = sigma[w];
    for (vid_t i = 0; i < succ_count[w]; i++) {
      vid_t v = succ[graph.vertices[w] + i];
      dsw = dsw + (sw / sigma[v]) * (1.0 + delta[v]);
    }
    delta[w] = dsw;
    atomicAdd(&betweenness_centrality[w], dsw);
  }
}

/**
 * Unweighted betweenness centrality back sum kernel for predecessor map
 * implementation. This function calculates the actual betweenness centrality
 * score by summing dependences for each vertex.
 */
__global__
void unweighted_back_sum_kernel(graph_t graph, vid_t source, int32_t dist,
                                int32_t* dists, score_t* delta,
                                score_t* betweenness_centrality) {
  const vid_t thread_id = THREAD_GLOBAL_INDEX;
  if (thread_id < graph.vertex_count) {
    if (thread_id != source && dists[thread_id] == (dist - 1)) {
      betweenness_centrality[thread_id] += delta[thread_id];
    }
  }
}

/**
 * Implements the parallel Brandes betweenness centrality algorithm using a
 * successor stack, as described in "A Faster Parallel Algorithm and Efficient
 * Multithreaded Implementations for Evaluating Betweenness Centrality on
 * Massive Datasets" [Madduri09]
 */
error_t betweenness_unweighted_gpu(const graph_t* graph,
                                   score_t* betweenness_centrality) {
  // Sanity check on input
  bool finished = true;
  error_t rc = betweenness_check_special_cases(graph, &finished, 
                                               betweenness_centrality);
  if (finished) return rc;

  // Allocate memory and initialize state on the GPU
  graph_t* graph_d;
  vid_t* sigma_d;
  int32_t* dist_d;
  vid_t* succ_d;
  uint32_t* succ_count_d;
  vid_t* stack_d;
  uint32_t* stack_count_d;
  score_t* delta_d;
  bool* finished_d;
  score_t* betweenness_centrality_d;

  // Initialization stage
  CHK_SUCCESS(initialize_succs_gpu(graph, graph->vertex_count, &graph_d,
                                   &sigma_d, &dist_d, &succ_d, &succ_count_d,
                                   &stack_d, &stack_count_d, &delta_d,
                                   &finished_d, &betweenness_centrality_d),
              err);

  // {} used to limit scope and avoid problems with error handles.
  {
  dim3 blocks;
  dim3 threads_per_block;

  // Find and count all shortest paths from every source vertex to every other
  // vertex in the graph. These paths and counts are used to determine the
  // betweenness centrality for each vertex
  for (vid_t source = 0; source < graph->vertex_count; source++) {
    // Initializations for this iteration
    CHK_SUCCESS(unweighted_succs_init(graph, source, sigma_d, dist_d, succ_d,
                                      succ_count_d, stack_d, stack_count_d,
                                      delta_d), err_free_all);
    CHK_CU_SUCCESS(hipGetLastError(), err_free_all);

    // SSSP and path counting stage
    KERNEL_CONFIGURE(graph->edge_count, blocks, threads_per_block);
    bool finished = false;
    int64_t phase = 0;
    // Keep counting distances until the BFS kernel completes
    while (!finished) {
      CHK_CU_SUCCESS(hipMemset(finished_d, true, sizeof(bool)), err_free_all);
      CHK_CU_SUCCESS(hipDeviceSynchronize(), err_free_all);
      unweighted_sssp_succs_kernel<<<blocks, threads_per_block>>>
        (*graph_d, phase, sigma_d, dist_d, succ_d, succ_count_d, stack_d,
         stack_count_d, finished_d);
      CHK_CU_SUCCESS(hipMemcpy(&finished, finished_d, sizeof(bool),
                                hipMemcpyDeviceToHost), err_free_all);
      phase++;
    }

    // Dependency accumulation stage
    phase -= 2;
    CHK_CU_SUCCESS(hipMemset(delta_d, (score_t)0.0,
                              graph->vertex_count * sizeof(vid_t)),
                   err_free_all);
    KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);
    while (phase > 0) {
      unweighted_dep_acc_kernel<<<blocks, threads_per_block>>>
        (*graph_d, phase, stack_count_d, sigma_d, stack_d, succ_d, succ_count_d,
         delta_d, betweenness_centrality_d);
      CHK_CU_SUCCESS(hipDeviceSynchronize(), err_free_all);
      CHK_CU_SUCCESS(hipGetLastError(), err_free_all);
      phase--;
    }
  }}

  // Cleanup phase
  CHK_SUCCESS(finalize_succs_gpu(graph_d, sigma_d, dist_d, succ_d, succ_count_d,
                                 stack_d, stack_count_d, delta_d, finished_d,
                                 betweenness_centrality_d,
                                 betweenness_centrality), err_free_all);

  return SUCCESS;

 err_free_all:
  graph_finalize_device(graph_d);
  hipFree(sigma_d);
  hipFree(dist_d);
  hipFree(succ_d);
  hipFree(succ_count_d);
  hipFree(stack_d);
  hipFree(stack_count_d);
  hipFree(delta_d);
  hipFree(finished_d);
  hipFree(betweenness_centrality_d);
 err:
  return FAILURE;
}

/**
 * Implements the parallel Brandes betweenness centrality algorithm using
 * predecessor maps as described in "Fast Network Centrality Analysis Using
 * GPUs" [Shi11]
 */
error_t betweenness_unweighted_shi_gpu(const graph_t* graph,
                                       score_t* betweenness_centrality) {
  // Sanity check on input
  bool finished = true;
  error_t rc = betweenness_check_special_cases(graph, &finished, 
                                               betweenness_centrality);
  if (finished) return rc;

  // Construct the reverse edges list (graph->edges is a list of destination
  // vertices, r_edges is a list of source vertices, indexed by edge id)
  vid_t* r_edges;
  totem_malloc(graph->edge_count * sizeof(vid_t), 
               TOTEM_MEM_HOST_PINNED, (void**)&r_edges);
  vid_t v = 0;
  for (eid_t e = 0; e < graph->edge_count; e++) {
    while (v <= graph->vertex_count &&
           !(e >= graph->vertices[v] && e < graph->vertices[v+1])) {
      v++;
    }
    r_edges[e] = v;
  }

  // Allocate memory and initialize state on the GPU
  graph_t* graph_d;
  vid_t* r_edges_d;
  bool* preds_d;
  vid_t* sigma_d;
  int32_t* dist_d;
  score_t* delta_d;
  bool* finished_d;
  score_t* betweenness_centrality_d;


  CHK_SUCCESS(initialize_preds_gpu(graph, graph->vertex_count, r_edges,
                                   &graph_d, &r_edges_d, &preds_d, &sigma_d,
                                   &dist_d, & delta_d, &finished_d,
                                   &betweenness_centrality_d),
              err_free_betweenness);

  // {} used to limit scope and avoid problems with error handles.
  {
  dim3 blocks;
  dim3 threads_per_block;

  // Find and count all shortest paths from every source vertex to every other
  // vertex in the graph. These paths and counts are used to determine the
  // betweenness centrality for each vertex
  for (vid_t source = 0; source < graph->vertex_count; source++) {
    // APSP
    int32_t dist = 0;
    CHK_CU_SUCCESS(hipMemset(dist_d, -1, graph->vertex_count
                                          * sizeof(int32_t)), err_free_all);
    CHK_CU_SUCCESS(hipMemset(preds_d, false, graph->edge_count * sizeof(bool)),
                   err_free_all);
    CHK_CU_SUCCESS(hipMemset(sigma_d, 0, graph->vertex_count * sizeof(vid_t)),
                   err_free_all);
    CHK_CU_SUCCESS(hipMemset(delta_d, 0, graph->vertex_count
                              * sizeof(score_t)), err_free_all);
    KERNEL_CONFIGURE(1, blocks, threads_per_block);
    unweighted_bc_preds_init_kernel<<<blocks, threads_per_block>>>
      (source, dist_d, sigma_d);
    CHK_CU_SUCCESS(hipDeviceSynchronize(), err_free_all);

    KERNEL_CONFIGURE(graph->edge_count, blocks, threads_per_block);
    bool finished = false;
    while (!finished) {
      CHK_CU_SUCCESS(hipMemset(finished_d, true, sizeof(bool)), err_free_all);
      unweighted_sssp_preds_kernel<<<blocks, threads_per_block>>>
        (*graph_d, r_edges_d, dist, dist_d, sigma_d, preds_d, finished_d);
      CHK_CU_SUCCESS(hipDeviceSynchronize(), err_free_all);
      CHK_CU_SUCCESS(hipMemcpy(&finished, finished_d, sizeof(bool),
                                hipMemcpyDeviceToHost), err_free_all);
      dist++;
    }
    // Back Propogation
    while (dist > 1) {
      KERNEL_CONFIGURE(graph->edge_count, blocks, threads_per_block);
      unweighted_back_prop_kernel<<<blocks, threads_per_block>>>
        (*graph_d, r_edges_d, dist_d, sigma_d, preds_d, dist, delta_d);
      KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);
      unweighted_back_sum_kernel<<<blocks, threads_per_block>>>
        (*graph_d, source, dist, dist_d, delta_d, betweenness_centrality_d);
      dist--;
    }
  }}

  CHK_SUCCESS(finalize_preds_gpu(graph_d, r_edges_d, preds_d, sigma_d, dist_d,
                                 delta_d, finished_d, betweenness_centrality_d,
                                 betweenness_centrality), err_free_all);
  totem_free(r_edges, TOTEM_MEM_HOST_PINNED);

  return SUCCESS;

 err_free_all:
  graph_finalize_device(graph_d);
  hipFree(r_edges_d);
  hipFree(preds_d);
  hipFree(sigma_d);
  hipFree(dist_d);
  hipFree(delta_d);
  hipFree(finished_d);
  hipFree(betweenness_centrality_d);
 err_free_betweenness:
  totem_free(r_edges, TOTEM_MEM_HOST_PINNED);
  return FAILURE;
}

/**
 * Implements the parallel Brandes betweenness centrality algorithm using a
 * successor stack, as described in "A Faster Parallel Algorithm and Efficient
 * Multithreaded Implementations for Evaluating Betweenness Centrality on
 * Massive Datasets" [Madduri09]
 */
error_t betweenness_unweighted_cpu(const graph_t* graph, 
                                   score_t* betweenness_centrality) {
  // Sanity check on input
  bool finished = true;
  error_t rc = betweenness_check_special_cases(graph, &finished, 
                                               betweenness_centrality);
  if (finished) return rc;

  // Allocate memory for the shortest paths problem
  uint32_t* sigma = 
    (uint32_t*)malloc(graph->vertex_count * sizeof(uint32_t));
  int32_t* dist = (int32_t*)malloc(graph->vertex_count * sizeof(int32_t));
  vid_t* succ = (vid_t*)malloc(graph->edge_count * sizeof(vid_t));
  vid_t* succ_count = (vid_t*)malloc(graph->vertex_count * sizeof(vid_t));
  vid_t* stack = (vid_t*)malloc(graph->vertex_count * graph->vertex_count
                                 * sizeof(vid_t));
  vid_t* stack_count = (vid_t*)malloc(graph->vertex_count * sizeof(vid_t));
  score_t* delta =
    (score_t*)malloc(graph->vertex_count * sizeof(score_t));
  int64_t phase = 0;

  // Initialization stage
  OMP(omp parallel for)
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    betweenness_centrality[v] = (score_t)0.0;
  }

  // Find and count all shortest paths from every source vertex to every other
  // vertex in the graph. These paths and counts are used to determine the
  // betweenness centrality for each vertex
  for (vid_t source = 0; source < graph->vertex_count; source++) {
    // Initializations for this iteration
    memset(succ, 0, graph->edge_count * sizeof(vid_t));
    memset(succ_count, 0, graph->vertex_count * sizeof(vid_t));
    memset(stack, 0, graph->vertex_count * graph->vertex_count * sizeof(vid_t));
    memset(stack_count, 0,  graph->vertex_count * sizeof(vid_t));
    OMP(omp parallel for)
    for (vid_t t = 0; t < graph->vertex_count; t++) {
      sigma[t] = 0;
      dist[t] = -1;
    }
    sigma[source] = 1;
    dist[source] = 0;
    phase = 0;
    stack_count[phase] = 1;
    stack[graph->vertex_count * phase] = source;

    // SSSP and path counting
    bool finished = false;
    while (!finished) {
      finished = true;
      for (vid_t v_index = 0; v_index < stack_count[phase]; v_index++) {
        vid_t v = stack[graph->vertex_count * phase + v_index];
        // For all neighbors of v in parallel, iterate over paths
        OMP(omp parallel for)
        for (eid_t e = graph->vertices[v]; e < graph->vertices[v + 1]; e++) {
          vid_t w = graph->edges[e];
          int32_t dw = __sync_val_compare_and_swap(&dist[w], (uint32_t)-1,
                                                   phase + 1);
          if (dw == -1) {
            finished = false;
            vid_t p = __sync_fetch_and_add(&stack_count[phase + 1], 1);
            stack[graph->vertex_count * (phase + 1) + p] = w;
            dw = phase + 1;
          }
          if (dw == phase + 1) {
            vid_t p = (vid_t)__sync_fetch_and_add(&succ_count[v], 1);
            succ[graph->vertices[v] + p] = w;
            __sync_fetch_and_add(&sigma[w], sigma[v]);
          }
        }
      }
      phase++;
    }
    phase--;

    // Dependency accumulation stage
    memset(delta, (score_t)0.0, graph->vertex_count * sizeof(vid_t));
    phase--;
    while (phase > 0) {
      OMP(omp parallel for)
      for (vid_t p = 0; p < stack_count[phase]; p++) {
        vid_t w = stack[graph->vertex_count * phase + p];
        score_t dsw = 0.0;
        score_t sw = sigma[w];
        for (vid_t i = 0; i < succ_count[w]; i++) {
          vid_t v = succ[graph->vertices[w] + i];
          dsw = dsw + (sw / sigma[v]) * (1.0 + delta[v]);
        }
        delta[w] = dsw;
        betweenness_centrality[w] = betweenness_centrality[w] + dsw;
      }
      phase--;
    }
  }

  // Cleanup phase
  free(sigma);
  free(dist);
  free(delta);
  free(stack_count);
  free(succ);
  free(succ_count);
  free(stack);
  return SUCCESS;
}

/**
 * Implements the forward propagation phase of the Betweenness Centrality
 * Algorithm described in Chapter 2 of GPU Computing Gems
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[in] source the source node for the shortest paths
 * @param[in] level the shared level variable between backward and forward
 *            propagations
 * @param[in] numSPs an array which counts the number of shortest paths in
 *            which each node is involved 
 * @param[in] distance an array which stores the distance of the shortest
 *            path for each node
 * @return void
 */
inline PRIVATE 
void betweenness_cpu_forward_propagation(const graph_t* graph, 
                                         vid_t source, cost_t& level,
                                         uint32_t* numSPs, cost_t* distance) {
  // Initialize the shortest path count to 0 and distance to infinity given
  // this source node
  totem_memset(numSPs, (uint32_t)0, graph->vertex_count, TOTEM_MEM_HOST);
  totem_memset(distance, (cost_t)INF_COST, graph->vertex_count, TOTEM_MEM_HOST);
  // Set the distance from source to itself to 0
  distance[source] = 0;
  // Set the shortest path count to 1 (from source to itself)
  numSPs[source] = 1;

  bool done = false;
  while (!done) {
    done = true;
    // In parallel, iterate over vertices which are at the current level
    OMP(omp parallel for schedule(runtime) reduction(& : done))
    for (vid_t v = 0; v < graph->vertex_count; v++) {
      if (distance[v] == level) {
        // For all neighbors of v, iterate over paths
        for (eid_t e = graph->vertices[v]; e < graph->vertices[v + 1]; e++) {
          vid_t w = graph->edges[e];
          if (distance[w] == INF_COST) {
            distance[w] = level + 1;
            done = false;
          }
          if (distance[w] == level + 1) {
            __sync_fetch_and_add(&numSPs[w], numSPs[v]);
          }
        }
      }
    }
    level++;
  }
}

/**
 * Implements the backward propagation phase of the Betweenness Centrality
 * Algorithm described in Chapter 2 of GPU Computing Gems
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[in] level the shared level variable between backward and forward
 *            propagations
 * @param[in] numSPs an array which counts the number of shortest paths in
 *            which each node is involved 
 * @param[in] distance an array which stores the distance of the shortest
 *            path for each node
 * @param[in] delta an array of the dependencies for each node, which are used
 *            to compute the betweenness centrality measure
 * @param[out] betweenness_centrality the output list which contains the
 *             betweenness centrality values computed for each node
 * @return void
 */
inline PRIVATE 
void betweenness_cpu_backward_propagation(const graph_t* graph,
                                          cost_t& level, uint32_t* numSPs, 
                                          cost_t* distance, score_t* delta, 
                                          score_t* betweenness_centrality) {
  // Set deltas to 0 for every input node
  memset(delta, 0, graph->vertex_count * sizeof(score_t));
  while (level > 1) {
    level--;
    // In parallel, iterate over vertices which are at the current level
    OMP(omp parallel for  schedule(runtime))
    for (vid_t v = 0; v < graph->vertex_count; v++) {
      if (distance[v] == level) {
        // For all neighbors of v, iterate over paths
        for (eid_t e = graph->vertices[v]; e < graph->vertices[v + 1]; e++) {
          vid_t w = graph->edges[e];
          if (distance[w] == level + 1) {
            delta[v] = (delta[v] + ((((score_t)numSPs[v]) /
                       ((score_t)numSPs[w]))*(delta[w] + 1)));
          }
        }
        // Add the dependency to the BC sum
        betweenness_centrality[v] = betweenness_centrality[v] + delta[v];
      }
    }
  }
}

/**
 * Implements the core functionality for computing Betweenness Centrality
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[in] source the source node for the shortest paths
 * @param[in] numSPs an array which counts the number of shortest paths in
 *            which each node is involved 
 * @param[in] distance an array which stores the distance of the shortest
 *            path for each node
 * @param[in] delta an array of the dependencies for each node, which are used
 *            to compute the betweenness centrality measure
 * @param[out] betweenness_centrality the output list which contains the
 *             betweenness centrality values computed for each node
 * @return void
 */
inline PRIVATE void betweenness_cpu_core(const graph_t* graph, vid_t source, 
                                         uint32_t* numSPs, cost_t* distance, 
                                         score_t* delta, 
                                         score_t* betweenness_score) {
  // Initialize variable to keep track of level
  cost_t level = 0;
  // Perform the forward propagation phase for this source node
  betweenness_cpu_forward_propagation(graph, source, level, numSPs, distance);
  // Perform the backward propagation phase for this source node
  betweenness_cpu_backward_propagation(graph, level, numSPs, distance, delta,
                                       betweenness_score);
}

/**
 * Parallel CPU implementation of  Bewteenness Centrality algorithm described
 * in Chapter 2 of GPU Computing Gems (Algorithm 1 - Sequential BC Computation)
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[in] epsilon determines how precise the results of the algorithm will
 *            be, and thus also how long it will take to compute
 * @param[out] betweenness_score the output list of betweenness centrality
 *             scores per vertex
 * @return generic success or failure
 */
error_t betweenness_cpu(const graph_t* graph, double epsilon,
                        score_t* betweenness_score) {
  // Sanity check on input
  bool finished = true;
  error_t rc = betweenness_check_special_cases(graph, &finished, 
                                               betweenness_score);
  if (finished) return rc;

  // Allocate memory for the shortest paths problem
  cost_t* distance = (cost_t*)malloc(graph->vertex_count * sizeof(cost_t));
  uint32_t* numSPs = (uint32_t*)malloc(graph->vertex_count * sizeof(uint32_t));
  score_t* delta = (score_t*)malloc(graph->vertex_count * sizeof(score_t));

  // Initialization stage
  // Set BC(v) to 0 for every input node
  memset(betweenness_score, 0, graph->vertex_count * sizeof(vid_t));

 // determine whether we will compute exact or approximate BC values
  if (epsilon == CENTRALITY_EXACT) {
    // Compute exact values for Betweenness Centrality
    for (vid_t source = 0; source < graph->vertex_count; source++) { 
      // Perform forward and backward propagation with source node
      betweenness_cpu_core(graph, source, numSPs, distance, delta,
                           betweenness_score);  
    }
  } else {
    // Compute approximate values based on the value of epsilon provided
    // Select a subset of source nodes to make the computation faster
    int num_samples = centrality_get_number_sample_nodes(graph->vertex_count,
                                                         epsilon);
    // Populate the array of indices to sample
    vid_t* sample_nodes = centrality_select_sampling_nodes(graph,
                                                           num_samples);
 
    for (int source_index = 0; source_index < num_samples; source_index++) {
      // Get the next sample node in the array to use as a source
      vid_t source = sample_nodes[source_index];
      // Perform forward and backward propagation with source node
      betweenness_cpu_core(graph, source, numSPs, distance, delta,
                           betweenness_score);   
    }
    
    // Scale the computed Betweenness Centrality metrics since they were
    // computed using a subset of the total nodes within the graph
    // The scaling value is: (Total Number of Nodes / Subset of Nodes Used)
    OMP(omp parallel for) 
    for (vid_t v = 0; v < graph->vertex_count; v++) {
      betweenness_score[v] *= (score_t)((double)(graph->vertex_count) /
                                        (double)num_samples);
    }
 
    // Clean up the allocated memory
    free(sample_nodes);
  }

  // Clean up the allocated memory
  free(numSPs);
  free(distance);
  free(delta);

  return SUCCESS;
}

/**
 * Scales the computed betweenness centrality scores
 * when computing approximate values
 */
__global__
void betweenness_gpu_scale_scores_kernel(vid_t vertex_count, int num_samples,
                                         score_t* betweenness_score_d) { 
  const vid_t vid = THREAD_GLOBAL_INDEX; 
  if (vid < vertex_count) {
    betweenness_score_d[vid] = (score_t)(((double)(vertex_count) / 
                                          num_samples) * 
                                         betweenness_score_d[vid]);
  }
}

/**
 * Initializes the distance and number of shortest paths for each
 * source node before forward propagation, as well as to initialize 
 * the distance and number of shortest paths  for the specified
 * source node.
 */
__global__
void betweenness_gpu_forward_init_kernel(vid_t source, bool* done_d, 
                                         vid_t vertex_count, cost_t* distance_d,
                                         uint32_t* numSPs_d) {
  const vid_t vid = THREAD_GLOBAL_INDEX;
  if (vid >= vertex_count) return;
  if (vid == source) {
    distance_d[vid] = 0;
    numSPs_d[vid] = 1;
    *done_d = false;
  } else {
    distance_d[vid] = INF_COST;
    numSPs_d[vid] = 0;
  } 
}

/**
 * This structure is used by the virtual warp-based implementation. It stores a
 * batch of work. It is allocated on shared memory and is processed by a single
 * virtual warp. Basically it caches the state of the vertices to be processed.
 */
typedef struct {
  eid_t    vertices[VWARP_DEFAULT_BATCH_SIZE + 1];
  cost_t   distance[VWARP_DEFAULT_BATCH_SIZE];
  uint32_t numSPs[VWARP_DEFAULT_BATCH_SIZE];
} batch_mem_t;

/**
 * The neighbors forward propagation processing function. This function sets 
 * the level of the neighbors' vertex to one level more than the parent vertex.
 * The assumption is that the threads of a warp invoke this function to process
 * the warp's batch of work. In each iteration of the for loop, each thread 
 * processes a neighbor. For example, thread 0 in the warp processes neighbors 
 * at indices 0, VWARP_DEFAULT_WARP_WIDTH, (2 * VWARP_DEFAULT_WARP_WIDTH) etc. 
 * in the edges array, while thread 1 in the warp processes neighbors 1, 
 * (1 + VWARP_DEFAULT_WARP_WIDTH), (1 + 2 * VWARP_DEFAULT_WARP_WIDTH) and so on.
*/
__device__
void forward_process_neighbors(vid_t warp_offset, vid_t* nbrs, vid_t nbr_count, 
                               uint32_t my_numSPs, uint32_t* numSPs_d,
                               cost_t* distance_d, cost_t level, bool* done_d) {
  for(vid_t i = warp_offset; i < nbr_count; i += VWARP_DEFAULT_WARP_WIDTH) {
    vid_t nbr = nbrs[i];
    if (distance_d[nbr] == INF_COST) {
      distance_d[nbr] = level + 1;
      *done_d = false;
    }
    if (distance_d[nbr] == level + 1) {
      atomicAdd(&numSPs_d[nbr], my_numSPs);
    }
  }
}

/**
 * Performs forward propagation
 */
__global__
void betweenness_gpu_forward_kernel(const graph_t graph_d, bool* done_d,
                                    cost_t level, uint32_t* numSPs_d, 
                                    cost_t* distance_d, uint32_t thread_count) {
  if (THREAD_GLOBAL_INDEX >= thread_count) return;
  vid_t warp_offset = THREAD_GLOBAL_INDEX % VWARP_DEFAULT_WARP_WIDTH;
  vid_t warp_id     = THREAD_GLOBAL_INDEX / VWARP_DEFAULT_WARP_WIDTH;

  __shared__ batch_mem_t batch_s[(MAX_THREADS_PER_BLOCK / 
                                  VWARP_DEFAULT_WARP_WIDTH)];
  batch_mem_t* vwarp_batch_s = &batch_s[THREAD_BLOCK_INDEX / 
                                        VWARP_DEFAULT_WARP_WIDTH];
  vid_t base_v = warp_id * VWARP_DEFAULT_BATCH_SIZE;
  vwarp_memcpy(vwarp_batch_s->vertices, &(graph_d.vertices[base_v]), 
               VWARP_DEFAULT_BATCH_SIZE + 1, warp_offset);
  vwarp_memcpy(vwarp_batch_s->distance, &distance_d[base_v],
               VWARP_DEFAULT_BATCH_SIZE, warp_offset);
  vwarp_memcpy(vwarp_batch_s->numSPs, &numSPs_d[base_v], 
               VWARP_DEFAULT_BATCH_SIZE, warp_offset);

  // iterate over my work
  for(vid_t v = 0; v < VWARP_DEFAULT_BATCH_SIZE; v++) {
    if (vwarp_batch_s->distance[v] == level) {
      vid_t* nbrs = &(graph_d.edges[vwarp_batch_s->vertices[v]]);
      vid_t nbr_count = vwarp_batch_s->vertices[v + 1] - 
        vwarp_batch_s->vertices[v];
      forward_process_neighbors(warp_offset, nbrs, nbr_count, 
                                vwarp_batch_s->numSPs[v], numSPs_d, 
                                distance_d, level, done_d);
    }
  }
}

/**
 * The neighbors backward propagation processing function. This function 
 * computes the delta of a vertex.
 */
__device__
void backward_process_neighbors(vid_t warp_offset, vid_t* nbrs, vid_t nbr_count,
                                uint32_t my_numSPs, score_t* vwarp_delta_s, 
                                uint32_t* numSPs_d, cost_t* distance_d, 
                                score_t* delta_d, cost_t level,
                                score_t* my_delta_d, score_t* my_bc_d) {
  vwarp_delta_s[warp_offset] = 0;
  for(vid_t i = warp_offset; i < nbr_count; i += VWARP_DEFAULT_WARP_WIDTH) {
    vid_t nbr = nbrs[i];
    if (distance_d[nbr] == level + 1) {
      // Compute an intermediary delta value in shared memory
      vwarp_delta_s[warp_offset] += 
        (((score_t)my_numSPs) / ((score_t)numSPs_d[nbr])) * (delta_d[nbr] + 1);
    }
  }

  // Only one thread in the warp aggregates the final value of delta
  if (warp_offset == 0) {
    score_t delta = 0;
    for (vid_t i = 0; i < VWARP_DEFAULT_WARP_WIDTH; i++) {
      delta += vwarp_delta_s[i];
    }
    // Add the dependency to the BC sum
    if (delta) {
      *my_delta_d = delta;
      *my_bc_d += delta;
    }
  }
}

/**
 * Performs backward propagation
 */
__global__
void betweenness_gpu_backward_kernel(const graph_t graph_d, cost_t level, 
                                     uint32_t* numSPs_d, cost_t* distance_d,
                                     score_t* delta_d, 
                                     score_t* betweenness_scores_d, 
                                     uint32_t thread_count) {
  if (THREAD_GLOBAL_INDEX >= thread_count) return;
  vid_t warp_offset = THREAD_GLOBAL_INDEX % VWARP_DEFAULT_WARP_WIDTH;
  vid_t warp_id     = THREAD_GLOBAL_INDEX / VWARP_DEFAULT_WARP_WIDTH;

  // Each warp has a single entry in the following shared memory array.
  // The entry corresponds to a batch of work which will be processed
  // in parallel by a warp of threads.
  __shared__ batch_mem_t batch_s[(MAX_THREADS_PER_BLOCK / 
                                  VWARP_DEFAULT_WARP_WIDTH)];

  // Get a reference to the batch of work of the warp this thread belongs to
  batch_mem_t* vwarp_batch_s = &batch_s[THREAD_BLOCK_INDEX / 
                                        VWARP_DEFAULT_WARP_WIDTH];

  // Calculate the starting vertex of the batch
  vid_t base_v = warp_id * VWARP_DEFAULT_BATCH_SIZE;

  // Cache the state of my warp's batch in the shared memory space
  vwarp_memcpy(vwarp_batch_s->vertices, &(graph_d.vertices[base_v]),
               VWARP_DEFAULT_BATCH_SIZE + 1, warp_offset);
  vwarp_memcpy(vwarp_batch_s->distance, &distance_d[base_v], 
               VWARP_DEFAULT_BATCH_SIZE, warp_offset);
  vwarp_memcpy(vwarp_batch_s->numSPs, &numSPs_d[base_v], 
               VWARP_DEFAULT_BATCH_SIZE, warp_offset);

  // Each thread in every warp has an entry in the following array which will be
  // used to calcule intermediary delta values in shared memory
  __shared__ score_t delta_s[MAX_THREADS_PER_BLOCK];

  // Get a reference to the entry of the first thread in the warp. This will be
  // indexed later using warp_offset
  int index = THREAD_BLOCK_INDEX / VWARP_DEFAULT_WARP_WIDTH;
  score_t* vwarp_delta_s = &delta_s[index * VWARP_DEFAULT_WARP_WIDTH];

  // Iterate over the warp's batch of work
  for(vid_t v = 0; v < VWARP_DEFAULT_BATCH_SIZE; v++) {
    if (vwarp_batch_s->distance[v] == level) {
      vid_t* nbrs = &(graph_d.edges[vwarp_batch_s->vertices[v]]);
      vid_t nbr_count = vwarp_batch_s->vertices[v + 1] - 
        vwarp_batch_s->vertices[v];
      backward_process_neighbors(warp_offset, nbrs, nbr_count, 
                                 vwarp_batch_s->numSPs[v], vwarp_delta_s,
                                 numSPs_d, distance_d, delta_d, level, 
                                 &delta_d[base_v + v], 
                                 &betweenness_scores_d[base_v + v]);
    }
  }
}

/**
 * Implements the core functionality for computing Betweenness Centrality
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[in] source the source node for the shortest paths
 * @param[in] numSPs an array which counts the number of shortest paths in
 *            which each node is involved 
 * @param[in] distance an array which stores the distance of the shortest
 *            path for each node
 * @param[in] delta an array of the dependencies for each node, which are used
 *            to compute the betweenness centrality measure
 * @param[out] betweenness_centrality the output list which contains the
 *             betweenness centrality values computed for each node
 * @return void
 */
inline PRIVATE 
error_t betweenness_gpu_core(graph_t* graph_d, vid_t vertex_count, bool* done_d,
                             vid_t source, uint32_t* numSPs_d, 
                             cost_t* distance_d, score_t* delta_d, 
                             score_t* betweenness_scores_d) {
  // Initialize variables for both forward and backward propagation
  dim3 blocks;
  dim3 threads_per_block;
  bool done = false;
  cost_t level = 0;

  // FORWARD PROPAGATION PHASE
  // Initialize the shortest path count to 0 and distance to infinity given
  // this source node, and also set the distance from source to itself to 0
  // and set the shortest path count to 1 (from source to itself), along 
  // with setting done_d to false
  KERNEL_CONFIGURE(vertex_count, blocks, threads_per_block);
  betweenness_gpu_forward_init_kernel<<<blocks, threads_per_block>>>
    (source, done_d, vwarp_default_state_length(vertex_count), distance_d, 
     numSPs_d);
  CHK_CU_SUCCESS(hipDeviceSynchronize(), err);
  KERNEL_CONFIGURE(vwarp_default_thread_count(vertex_count), blocks, 
                   threads_per_block);
  while (!done) {
    CHK_CU_SUCCESS(hipMemset(done_d, true, sizeof(bool)), err);
    // In parallel, iterate over vertices which are at the current level
    betweenness_gpu_forward_kernel<<<blocks, threads_per_block>>>
      (*graph_d, done_d, level, numSPs_d, distance_d, 
       vwarp_default_thread_count(vertex_count));
    CHK_CU_SUCCESS(hipMemcpy(&done, done_d, sizeof(bool), 
                              hipMemcpyDeviceToHost), err);
    level++;
  }

  // BACKWARD PROPAGATION PHASE
  // Set deltas to 0 for every input node
  CHK_CU_SUCCESS(hipMemset(delta_d, 0, 
                            vwarp_default_state_length(vertex_count) * 
                            sizeof(score_t)), err);
  while (level > 1) {
    level--;
    CHK_CU_SUCCESS(hipDeviceSynchronize(), err);
    // In parallel, iterate over vertices which are at the current level
    betweenness_gpu_backward_kernel<<<blocks, threads_per_block>>>
      (*graph_d, level, numSPs_d, distance_d, delta_d, betweenness_scores_d, 
       vwarp_default_thread_count(vertex_count));
  }

  return SUCCESS;

 err:
  return FAILURE;
}

/**
 * Allocates and initializes memory on the GPU for betweenness_gpu
 */
PRIVATE
error_t initialize_betweenness_gpu(const graph_t* graph, graph_t** graph_d, 
                                   cost_t** distance_d, uint32_t** numSPs_d, 
                                   score_t** delta_d, bool** done_d, 
                                   score_t** betweenness_scores_d) {
  vid_t state_length = vwarp_default_state_length(graph->vertex_count);
  // Allocate space on GPU
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);
  CHK_CU_SUCCESS(hipMalloc((void**)distance_d, state_length * sizeof(cost_t)),
                 err_free_graph_d);
  CHK_CU_SUCCESS(hipMalloc((void**)numSPs_d, state_length * sizeof(uint32_t)),
                 err_free_distance_d);
  CHK_CU_SUCCESS(hipMalloc((void**)delta_d, state_length * sizeof(score_t)),
                 err_free_numSPs_d);
  CHK_CU_SUCCESS(hipMalloc((void**)done_d, sizeof(bool)), err_free_delta_d);
  CHK_CU_SUCCESS(hipMalloc((void**)betweenness_scores_d, state_length
                            * sizeof(score_t)), err_free_done_d);

  // Setup initial parameters
  CHK_CU_SUCCESS(hipMemset(*betweenness_scores_d, (score_t)0.0,
                            state_length * sizeof(score_t)), err_free_all);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(betweenness_gpu_forward_kernel), 
                         hipFuncCachePreferShared);
  return SUCCESS;

  // Failure cases for freeing memory
 err_free_all:
  hipFree(betweenness_scores_d);
 err_free_done_d:
  hipFree(done_d);
 err_free_delta_d:
  hipFree(delta_d);
 err_free_numSPs_d:
  hipFree(numSPs_d);
 err_free_distance_d:
  hipFree(distance_d);
 err_free_graph_d:
  graph_finalize_device(*graph_d);
 err:
  return FAILURE;
}

/**
 * GPU implementation of  Bewteenness Centrality algorithm described in
 * Chapter 2 of GPU Computing Gems (Algorithm 1 - Sequential BC Computation)
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[in] epsilon determines how precise the results of the algorithm will
 *            be, and thus also how long it will take to compute
 * @param[out] betweenness_score the output list of betweenness centrality
 *             scores per vertex
 * @return generic success or failure
 */
error_t betweenness_gpu(const graph_t* graph, double epsilon,
                        score_t* betweenness_score) {
  // Sanity check on input
  bool finished = true;
  error_t rc = betweenness_check_special_cases(graph, &finished, 
                                               betweenness_score);
  if (finished) return rc;

  // Initialization stage
  // Set BC(v) to 0 for every input node
  memset(betweenness_score, 0, graph->vertex_count * sizeof(score_t));

  // Create pointers for use with the GPU's memory
  graph_t*  graph_d;
  cost_t*   distance_d;
  uint32_t* numSPs_d;
  score_t*  delta_d;
  bool*     done_d;
  score_t*  betweenness_scores_d;

  // Initialization stage
  CHK_SUCCESS(initialize_betweenness_gpu(graph, &graph_d, &distance_d, 
                                         &numSPs_d, &delta_d, &done_d, 
                                         &betweenness_scores_d), err);
  // determine whether we will compute exact or approximate BC values
  if (epsilon == CENTRALITY_EXACT) {
    // Compute exact values for Betweenness Centrality
    for (vid_t source = 0; source < graph->vertex_count; source++) { 
      // Perform forward and backward propagation with source node
      CHK_SUCCESS(betweenness_gpu_core(graph_d, graph->vertex_count, done_d, 
                                       source, numSPs_d, distance_d, delta_d, 
                                       betweenness_scores_d), err_free_all);
    }
  } else {
    // Compute approximate values based on the value of epsilon provided
    // Select a subset of source nodes to make the computation faster
    int num_samples = centrality_get_number_sample_nodes(graph->vertex_count,
                                                         epsilon);
    // Populate the array of indices to sample
    vid_t* sample_nodes = centrality_select_sampling_nodes(graph,
                                                           num_samples);
 
    for (vid_t source_index = 0; source_index < num_samples; source_index++) {
      // Get the next sample node in the array to use as a source
      vid_t source = sample_nodes[source_index];
      // Perform forward and backward propagation with source node
      CHK_SUCCESS(betweenness_gpu_core(graph_d, graph->vertex_count, done_d, 
                                       source, numSPs_d, distance_d, delta_d,
                                       betweenness_scores_d), err_free_all);
    }
    
    // Scale the computed Betweenness Centrality metrics since they were
    // computed using a subset of the total nodes within the graph
    // The scaling value is: (Total Number of Nodes / Subset of Nodes Used)
    dim3 blocks;
    dim3 threads_per_block;
    KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);
    betweenness_gpu_scale_scores_kernel<<<blocks, threads_per_block>>>
      (graph->vertex_count, num_samples, betweenness_scores_d);

    // Clean up the allocated memory
    free(sample_nodes);
  }

  // Copy the calculated betweenness centrality scores back to host memory
  CHK_CU_SUCCESS(hipMemcpy(betweenness_score, betweenness_scores_d,
                            graph->vertex_count * sizeof(score_t),
                            hipMemcpyDeviceToHost), err_free_all);

  // Clean up the memory allocated on the GPU
  graph_finalize_device(graph_d);
  hipFree(distance_d);
  hipFree(numSPs_d);
  hipFree(delta_d);
  hipFree(done_d);
  hipFree(betweenness_scores_d);
  return SUCCESS;

 err_free_all:
  graph_finalize_device(graph_d);
  hipFree(distance_d);
  hipFree(numSPs_d);
  hipFree(delta_d);
  hipFree(done_d);
  hipFree(betweenness_scores_d);
 err:
  return FAILURE;
}
