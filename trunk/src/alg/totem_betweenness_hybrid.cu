#include "hip/hip_runtime.h"
/**
 * This file contains a hybrid implementation of the Betweenness Centrality
 * algorithm using the Totem framework
 *
 *  Created on: 2013-03-10
 *  Author: Abdullah Gharaibeh
 *          Robert Woff
 */

// Totem includes
#include "totem_alg.h"
#include "totem_centrality.h"
#include "totem_engine.cuh"
#include <thrust/sort.h>

/**
 * Per-partition specific state
 */
typedef struct betweenness_state_s {
  cost_t*   distance[MAX_PARTITION_COUNT]; // a list of distances state, one per
                                           // partition
  uint32_t* numSPs[MAX_PARTITION_COUNT]; // a list of number of shortest paths
                                           // state, one per partition
  uint32_t* numSPs_f[MAX_PARTITION_COUNT]; // a list of number of shortest paths
                                           // state, one per partition
  score_t*  delta[MAX_PARTITION_COUNT];    // delta BC score for a vertex
  bool*     done;        // pointer to global finish flag
  cost_t    level;       // current level being processed by the partition
  score_t*  betweenness; // betweenness score
  frontier_state_t frontier;
  bool* comm;            // flags that indicates whether to instruct the engine 
                         // to perform communication or not. This array is
                         // populated during the forward phase, and used during
                         // the backward propagation phase
} betweenness_state_t;

/**
 * State shared between all partitions
 */
typedef struct betweenness_global_state_s {
  score_t*   betweenness_score;   // final output buffer
  score_t*   betweenness_score_h; // used as a temporary buffer
  vid_t      src;                 // source vertex id (id after partitioning)
  double     epsilon;             // determines accuracy of BC computation
  int        num_samples;         // number of samples for approximate BC
} betweenness_global_state_t;
PRIVATE betweenness_global_state_t bc_g;

/**
 * The neighbors forward propagation processing function. This function sets 
 * the level of the neighbors' vertex to one level more than the parent vertex.
 * The assumption is that the threads of a warp invoke this function to process
 * the warp's batch of work. In each iteration of the for loop, each thread 
 * processes a neighbor. For example, thread 0 in the warp processes neighbors 
 * at indices 0, VWARP_WIDTH, (2 * VWARP_WIDTH) etc. in the edges array, while
 * thread 1 in the warp processes neighbors 1, (1 + VWARP_WIDTH), 
 * (1 + 2 * VWARP_WIDTH) and so on.
 */
template<int VWARP_WIDTH>
__device__ inline void
forward_process_neighbors(int warp_offset, const vid_t* __restrict nbrs, 
                          const vid_t nbr_count, uint32_t v_numSPs, 
                          betweenness_state_t* state, bool& done_d) {
  // Iterate through the portion of work
  for(vid_t i = warp_offset; i < nbr_count; i += VWARP_WIDTH) {
    vid_t nbr   = GET_VERTEX_ID(nbrs[i]);
    int nbr_pid = GET_PARTITION_ID(nbrs[i]);
    cost_t* nbr_distance = state->distance[nbr_pid];
    if (nbr_distance[nbr] == INF_COST) {
      nbr_distance[nbr] = state->level + 1;
      done_d = false;
    }
    if (nbr_distance[nbr] == state->level + 1) {
      uint32_t* numSPs = state->numSPs_f[nbr_pid];
      atomicAdd(&(numSPs[nbr]), v_numSPs);
    }
  }
}

template<int VWARP_WIDTH, int VWARP_BATCH>
__global__ void
forward_kernel(partition_t par, betweenness_state_t state, 
               const vid_t* __restrict frontier, vid_t count) {
  if (THREAD_GLOBAL_INDEX >= 
      vwarp_thread_count(count, VWARP_WIDTH, VWARP_BATCH)) return;

  const eid_t* __restrict vertices = par.subgraph.vertices;
  const uint32_t* __restrict numSPs = state.numSPs[par.id];

  // This flag is used to report the finish state of a block of threads. This
  // is useful to avoid having many threads writing to the global finished
  // flag, which can hurt performance (since "finished" is actually allocated
  // on the host, and each write will cause a transfer over the PCI-E bus)
  __shared__ bool finished_block;
  finished_block = true;
  __syncthreads();

  vid_t start_vertex = vwarp_block_start_vertex(VWARP_WIDTH, VWARP_BATCH) + 
    vwarp_warp_start_vertex(VWARP_WIDTH, VWARP_BATCH);
  vid_t end_vertex = start_vertex +
    vwarp_warp_batch_size(count, VWARP_WIDTH, VWARP_BATCH);
  int warp_offset = vwarp_thread_index(VWARP_WIDTH);
  
  // Iterate over my work
  for(vid_t i = start_vertex; i < end_vertex; i++) {
    vid_t v = frontier[i];
    // If the distance for this node is equal to the current level, then
    // forward process its neighbours to determine its contribution to
    // the number of shortest paths
    const eid_t nbr_count = vertices[v + 1] - vertices[v];
    vid_t* nbrs = par.subgraph.edges + vertices[v];
    if (v >= par.subgraph.vertex_ext) {
      nbrs = par.subgraph.edges_ext + 
        (vertices[v] - par.subgraph.edge_count_ext);
    }
    forward_process_neighbors<VWARP_WIDTH>
      (warp_offset, nbrs, nbr_count, numSPs[v], &state, finished_block);    
  }

  __syncthreads();
  // If there is remaining work to do, set the done flag to false
  if (!finished_block && THREAD_BLOCK_INDEX == 0) *(state.done) = false;
}

typedef void(*bc_gpu_func_t)(partition_t*, betweenness_state_t*, vid_t*, vid_t, 
                             hipStream_t);
template<int VWARP_WIDTH, int VWARP_BATCH>
#ifdef FEATURE_SM35
PRIVATE __host__ __device__ 
#else
PRIVATE __host__
#endif /* FEATURE_SM35  */
void forward_launch_gpu(partition_t* par, betweenness_state_t* state,
                        vid_t* frontier, vid_t count, hipStream_t stream) {
  if (count == 0) return;
  dim3 blocks; 
  const int threads = MAX_THREADS_PER_BLOCK;
  kernel_configure(vwarp_thread_count(count, VWARP_WIDTH, VWARP_BATCH),
                   blocks, threads);
  forward_kernel<VWARP_WIDTH, VWARP_BATCH><<<blocks, threads, 0, stream>>>
    (*par, *state, frontier, count);
}

#ifdef FEATURE_SM35
PRIVATE __global__
void forward_launch_at_boundary_kernel(partition_t par, 
                                       betweenness_state_t state) {
  if (THREAD_GLOBAL_INDEX > 0 || (*state.frontier.count == 0)) {
    return;
  }
  const bc_gpu_func_t FORWARD_GPU_FUNC[] = {
    forward_launch_gpu<1,   2>,   // (0) < 8
    forward_launch_gpu<8,   8>,   // (1) > 8    && < 32
    forward_launch_gpu<32,  32>,   // (2) > 32   && < 128
    forward_launch_gpu<128, 32>,   // (3) > 128  && < 256
    forward_launch_gpu<256, 32>,   // (4) > 256  && < 1K
    forward_launch_gpu<512, 32>,   // (5) > 1K   && < 2K
    forward_launch_gpu<MAX_THREADS_PER_BLOCK, 8>  // (6) > 2k
  };

  int64_t end = *(state.frontier.count);
  for (int i = FRONTIER_BOUNDARY_COUNT; i >= 0; i--) {
    int64_t start = state.frontier.boundaries[i];
    int64_t count = end - start;
    if (count > 0) {
      hipStream_t s;
      hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
      FORWARD_GPU_FUNC[i](&par, &state, state.frontier.list + start, count, s);
      end = start;
    }
  }
}
#endif /* FEATURE_SM35  */

PRIVATE const bc_gpu_func_t FORWARD_GPU_FUNC[] = {
  // RANDOM partitioning
  forward_launch_gpu<VWARP_MEDIUM_WARP_WIDTH,  VWARP_MEDIUM_BATCH_SIZE>,
  // HIGH partitioning
  forward_launch_gpu<VWARP_MEDIUM_WARP_WIDTH,  VWARP_MEDIUM_BATCH_SIZE>,
  // LOW partitioning
  forward_launch_gpu<MAX_THREADS_PER_BLOCK,  VWARP_MEDIUM_BATCH_SIZE>
};

/**
 * Entry point for forward propagation on the GPU
 */
PRIVATE inline void betweenness_forward_gpu(partition_t* par) {
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
  frontier_update_list_gpu(&state->frontier, state->level, 
                           state->distance[par->id], par->streams[1]);

  if (engine_partition_algorithm() == PAR_SORTED_DSC) {
    vid_t count;
    CALL_CU_SAFE(hipMemcpyAsync(&count, state->frontier.count,
                                 sizeof(vid_t), hipMemcpyDefault,
                                 par->streams[1]));
    CALL_CU_SAFE(hipStreamSynchronize(par->streams[1]));
    if (count == 0) {
      state->comm[state->level] = false;
      engine_report_no_comm(par->id);
      return;
    }
  }

  // clear out the outbox buffers
  for (int rmt_pid = 0; rmt_pid < context.pset->partition_count; rmt_pid++) {
    grooves_box_table_t* outbox =  &par->outbox[rmt_pid];
    if (rmt_pid == par->id || !outbox->count) continue;
    hipMemsetAsync(outbox->push_values, 0, outbox->count * sizeof(uint32_t), 
                    par->streams[1]);
  }

  // If the vertices are sorted by degree, call a kernel that takes 
  // advantage of that
#ifdef FEATURE_SM35
  if (engine_sorted()) {
    frontier_update_boundaries_gpu(&state->frontier, &par->subgraph,
                                   par->streams[1]);
    forward_launch_at_boundary_kernel<<<1, 1, 0, par->streams[1]>>>
      (*par, *state);
    CALL_CU_SAFE(hipGetLastError());
    return;
  }
#endif /* FEATURE_SM35 */

  // Call the corresponding cuda kernel to perform forward propagation
  // given the current state of the algorithm
  vid_t count;
  CALL_CU_SAFE(hipMemcpyAsync(&count, state->frontier.count, 
                               sizeof(vid_t), hipMemcpyDefault, 
                               par->streams[1]));
  CALL_CU_SAFE(hipStreamSynchronize(par->streams[1]));
  int par_alg = engine_partition_algorithm();
  FORWARD_GPU_FUNC[par_alg](par, state, state->frontier.list, count, 
                            par->streams[1]);
}

/**
 * Entry point for forward propagation on the CPU
 */
void betweenness_forward_cpu(partition_t* par) {
  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
  graph_t* subgraph = &par->subgraph;  
  cost_t* distance = state->distance[par->id];
  uint32_t* numSPs = state->numSPs[par->id];
  bool done = true;
  bool comm = false;
  // In parallel, iterate over vertices which are at the current level
  OMP(omp parallel for schedule(runtime) reduction(& : done) 
      reduction(| : comm))
  for (vid_t v = 0; v < subgraph->vertex_count; v++) {
    if (distance[v] == state->level) {
      for (eid_t e = subgraph->vertices[v]; e < subgraph->vertices[v + 1]; 
           e++) {
        vid_t nbr = GET_VERTEX_ID(subgraph->edges[e]);
        int nbr_pid = GET_PARTITION_ID(subgraph->edges[e]);
        cost_t* nbr_distance = state->distance[nbr_pid];
        if (nbr_distance[nbr] == INF_COST) {
          nbr_distance[nbr] = state->level + 1;
          done = false;
          if (nbr_pid != par->id) comm = true;
        }
        if (nbr_distance[nbr] == state->level + 1) {
          uint32_t* nbr_numSPs = state->numSPs_f[nbr_pid];
          __sync_fetch_and_add(&nbr_numSPs[nbr], numSPs[v]); 
        }
      }
    }
  }
  if (!comm) {
    engine_report_no_comm(par->id);
    state->comm[state->level] = false;
  }

  // If there is remaining work to do, set the done flag to false
  if (!done) {    
    *(state->done) = false;
  }
}

/**
 * Distributes work to either the CPU or GPU
 */
PRIVATE void betweenness_forward(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;

  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;

  for (int pid = 0; pid < engine_partition_count(); pid++) {
    if (pid != par->id) {
      state->numSPs_f[pid] = (uint32_t*)par->outbox[pid].push_values;
    }
  }

  // Check which kind of processor this partition corresponds to and
  // call the appropriate function to perform forward propagation
  if (par->processor.type == PROCESSOR_CPU) {
    betweenness_forward_cpu(par);
  } else if (par->processor.type == PROCESSOR_GPU) {
    betweenness_forward_gpu(par);
  } else {
    assert(false);
  }
  // Increment the level for the next round of forward propagation
  state->level++;
}

/**
 * The neighbors backward propagation processing function. This function 
 * computes the delta of a vertex.
 */
template<int VWARP_WIDTH>
__device__ void
backward_process_neighbors(partition_t* par, betweenness_state_t* state,
                           const vid_t* __restrict nbrs, vid_t nbr_count,
                           uint32_t v_numSPs, score_t* vwarp_delta_s, vid_t v) {
  int warp_offset = vwarp_thread_index(VWARP_WIDTH);
  score_t sum = 0;
  // Iterate through the portion of work
  for(vid_t i = warp_offset; i < nbr_count; i += VWARP_WIDTH) {
    vid_t nbr = GET_VERTEX_ID(nbrs[i]);
    int nbr_pid = GET_PARTITION_ID(nbrs[i]);
    cost_t* nbr_distance = state->distance[nbr_pid];
    if (nbr_distance[nbr] == state->level + 1) {
      // Compute an intermediary delta value in shared memory
      score_t* nbr_delta = state->delta[nbr_pid];
      uint32_t* nbr_numSPs = state->numSPs[nbr_pid];
      sum += ((((score_t)v_numSPs) / ((score_t)nbr_numSPs[nbr])) *
              (nbr_delta[nbr] + 1));
    }
  }
  vwarp_delta_s[warp_offset] = sum;

  if (VWARP_WIDTH > 32) __syncthreads();
  for (uint32_t s = VWARP_WIDTH / 2; s > 0; s >>= 1) {
    if (warp_offset < s) {
      vwarp_delta_s[warp_offset] += vwarp_delta_s[warp_offset + s];
    }
    __syncthreads();
  }
  if ((warp_offset == 0) && vwarp_delta_s[0]) {
    (state->delta[par->id])[v] = vwarp_delta_s[0];
    state->betweenness[v] += vwarp_delta_s[0];
  }
}

/**
 * CUDA kernel which performs backward propagation
 */
template<int VWARP_WIDTH, int VWARP_BATCH>
__global__ void
betweenness_backward_kernel(partition_t par, betweenness_state_t state,
                            const vid_t* __restrict frontier, vid_t count) {
  if (THREAD_GLOBAL_INDEX >= 
      vwarp_thread_count(count, VWARP_WIDTH, VWARP_BATCH)) return;

  const eid_t* __restrict vertices = par.subgraph.vertices;
  const uint32_t* __restrict numSPs = state.numSPs[par.id];

  // Each thread in every warp has an entry in the following array which will be
  // used to calculate intermediary delta values in shared memory
  __shared__ score_t delta_s[MAX_THREADS_PER_BLOCK];
  const int index = THREAD_BLOCK_INDEX / VWARP_WIDTH;
  score_t* vwarp_delta_s = &delta_s[index * VWARP_WIDTH];

  vid_t start_vertex = vwarp_block_start_vertex(VWARP_WIDTH, VWARP_BATCH) + 
    vwarp_warp_start_vertex(VWARP_WIDTH, VWARP_BATCH);
  vid_t end_vertex = start_vertex +
    vwarp_warp_batch_size(count, VWARP_WIDTH, VWARP_BATCH);
  int warp_offset = vwarp_thread_index(VWARP_WIDTH);
  
  // Iterate over my work
  for(vid_t i = start_vertex; i < end_vertex; i++) {
    vid_t v = frontier[i];
    // If the vertex is at the current level, determine its contribution
    // to the source vertex's delta value
    const eid_t nbr_count = vertices[v + 1] - vertices[v];
    vid_t* nbrs = par.subgraph.edges + vertices[v];
    if (v >= par.subgraph.vertex_ext) {
      nbrs = par.subgraph.edges_ext + 
        (vertices[v] - par.subgraph.edge_count_ext);
    }
    backward_process_neighbors<VWARP_WIDTH>
      (&par, &state, nbrs, nbr_count, numSPs[v], vwarp_delta_s, v);
  }
}

template<int VWARP_WIDTH, int VWARP_BATCH>
#ifdef FEATURE_SM35
PRIVATE __host__ __device__ 
#else
PRIVATE __host__
#endif /* FEATURE_SM35  */
void backward_launch_gpu(partition_t* par, betweenness_state_t* state,
                         vid_t* frontier, vid_t count, hipStream_t stream) {
  if (count == 0) return;
  dim3 blocks; const int threads = MAX_THREADS_PER_BLOCK;
  kernel_configure(vwarp_thread_count(count, VWARP_WIDTH, VWARP_BATCH), 
                   blocks, threads);
  betweenness_backward_kernel<VWARP_WIDTH, VWARP_BATCH>
    <<<blocks, threads, 0, stream>>>(*par, *state, frontier, count);
}

#ifdef FEATURE_SM35
PRIVATE __global__
void backward_launch_at_boundary_kernel(partition_t par, 
                                        betweenness_state_t state) {
  if (THREAD_GLOBAL_INDEX > 0 || (*state.frontier.count == 0)) {
    return;
  }
  const bc_gpu_func_t BACKWARD_GPU_FUNC[] = {
    backward_launch_gpu<1,   2>,   // (0) < 8
    backward_launch_gpu<8,   8>,   // (1) > 8    && < 32
    backward_launch_gpu<32,  32>,   // (2) > 32   && < 128
    backward_launch_gpu<128, 32>,   // (3) > 128  && < 256
    backward_launch_gpu<256, 32>,   // (4) > 256  && < 1K
    backward_launch_gpu<512, 32>,   // (5) > 1K   && < 2K
    backward_launch_gpu<MAX_THREADS_PER_BLOCK, 8>  // (6) > 2k
  };

  int64_t end = *(state.frontier.count);
  for (int i = FRONTIER_BOUNDARY_COUNT; i >= 0; i--) {
    int64_t start = state.frontier.boundaries[i];
    int64_t count = end - start;
    if (count > 0) {
      hipStream_t s;
      hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
      BACKWARD_GPU_FUNC[i](&par, &state, state.frontier.list + start, count, s);
      end = start;
    }
  }
}
#endif /* FEATURE_SM35  */

PRIVATE const bc_gpu_func_t BACKWARD_GPU_FUNC[] = {
  // RANDOM algorithm
  backward_launch_gpu<VWARP_MEDIUM_WARP_WIDTH, VWARP_MEDIUM_BATCH_SIZE>,
  // HIGH partitioning
  backward_launch_gpu<VWARP_MEDIUM_WARP_WIDTH, VWARP_MEDIUM_BATCH_SIZE>,
  // LOW partitioning
  backward_launch_gpu<MAX_THREADS_PER_BLOCK,  VWARP_MEDIUM_BATCH_SIZE>
};

/**
 * Entry point for backward propagation on GPU
 */
PRIVATE inline void betweenness_backward_gpu(partition_t* par) {
  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;

  // If the vertices are sorted by degree, call a kernel that takes 
  // advantage of that
#ifdef FEATURE_SM35
  if (engine_sorted()) {
    frontier_update_boundaries_gpu(&state->frontier, &par->subgraph,
                                   par->streams[1]);
    backward_launch_at_boundary_kernel<<<1, 1, 0, par->streams[1]>>>
      (*par, *state);
    CALL_CU_SAFE(hipGetLastError());
    return;
  }
#endif /* FEATURE_SM35 */

  vid_t count;
  CALL_CU_SAFE(hipMemcpyAsync(&count, state->frontier.count, 
                               sizeof(vid_t), hipMemcpyDefault, 
                               par->streams[1]));
  CALL_CU_SAFE(hipStreamSynchronize(par->streams[1]));
  int par_alg = engine_partition_algorithm();
  BACKWARD_GPU_FUNC[par_alg](par, state, state->frontier.list, 
                             count, par->streams[1]);
}

/**
 * Entry point for backward propagation on CPU
 */
void betweenness_backward_cpu(partition_t* par) {
  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
  graph_t* subgraph = &par->subgraph;
  cost_t* distance = state->distance[par->id];
  uint32_t* numSPs = state->numSPs[par->id];
  score_t* delta = state->delta[par->id];

  // In parallel, iterate over vertices which are at the current level
  OMP(omp parallel for schedule(runtime))
  for (vid_t v = 0; v < subgraph->vertex_count; v++) {
    cost_t v_distance = distance[v];
    if (v_distance == state->level) {
      // For all neighbors of v, iterate over paths
      score_t delta_v = 0;
      for (eid_t e = subgraph->vertices[v]; e < subgraph->vertices[v + 1];
           e++) {
        vid_t nbr = GET_VERTEX_ID(subgraph->edges[e]);
        int nbr_pid = GET_PARTITION_ID(subgraph->edges[e]);
        cost_t* nbr_distance = state->distance[nbr_pid];

        // Check whether the neighbour is local or remote and update accordingly
        if (nbr_distance[nbr] == state->level + 1) {
          score_t* nbr_delta = state->delta[nbr_pid];
          uint32_t* nbr_numSPs = state->numSPs[nbr_pid];
          delta_v += ((((score_t)(numSPs[v])) / ((score_t)(nbr_numSPs[nbr]))) *
                      (nbr_delta[nbr] + 1));
        }
      }
      // Add the dependency to the BC sum
      delta[v] += delta_v;
      state->betweenness[v] += delta[v];
    }
  }
}

/**
 * Distributes work for backward propagation to either the CPU or GPU
 */
PRIVATE void betweenness_backward(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;

  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;

  for (int pid = 0; pid < engine_partition_count(); pid++) {
    if (pid != par->id) {
      state->delta[pid] = (score_t*)par->outbox[pid].pull_values;
    }
  }

  if (engine_superstep() > 1) {
    // Check what kind of processing unit corresponds to this partition and
    // then call the appropriate function to perform backward propagation
    if (par->processor.type == PROCESSOR_CPU) {
      betweenness_backward_cpu(par);
    } else if (par->processor.type == PROCESSOR_GPU) {
      betweenness_backward_gpu(par);
    } else {
      assert(false);
    }
  }
  // Decrement the level for the next round of backward propagation
  state->level--;

  // Check whether backward propagation is finished
  if (state->level > 0) {
    engine_report_not_finished();
    if (!state->comm[state->level]) {
      engine_report_no_comm(par->id);
    }
  }
}

/*
 * Parallel CPU implementation of betweenness scatter function
 */
PRIVATE inline void betweenness_scatter_cpu(int pid, grooves_box_table_t* inbox,
                                            betweenness_state_t* state) {
  cost_t* distance = state->distance[pid];
  uint32_t* numSPs = state->numSPs[pid];
  // Get the values that have been pushed to this vertex
  uint32_t* inbox_values = (uint32_t*)inbox->push_values;
  OMP(omp parallel for schedule(runtime))
  for (vid_t index = 0; index < inbox->count; index++) {
    if (inbox_values[index] != 0) {
      vid_t vid = inbox->rmt_nbrs[index];
      // If the distance was previously infinity, initialize it to the
      // current level 
      if (distance[vid] == INF_COST) {
        distance[vid] = state->level;
      }
      // If the distance is equal to the current level, update the 
      // nodes number of shortest paths with the pushed value
      if (distance[vid] == state->level) {
        numSPs[vid] += inbox_values[index];
      }
    }
  }
}

/*
 * Kernel for betweenness_scatter_gpu
 */
__global__ void betweenness_scatter_kernel(grooves_box_table_t inbox, 
                                           cost_t* distance, uint32_t* numSPs,
                                           cost_t level) {
  vid_t index = THREAD_GLOBAL_INDEX;
  if (index >= inbox.count) return;

  // Get the values that have been pushed to this vertex
  uint32_t* inbox_values = (uint32_t*)inbox.push_values;
  if (inbox_values[index] != 0) {
    vid_t vid = inbox.rmt_nbrs[index];
    // If the distance was previously infinity, initialize it to the
    // current level   
    if (distance[vid] == INF_COST) {
      distance[vid] = level;
    }
    // If the distance is equal to the current level, update the 
    // nodes number of shortest paths with the pushed value
    if (distance[vid] == level) {
      numSPs[vid] += inbox_values[index];
    }
  }
}

/*
 * Parallel GPU implementation of betweenness scatter function
 */
PRIVATE inline void betweenness_scatter_gpu(partition_t* par, 
                                            grooves_box_table_t* inbox,
                                            betweenness_state_t* state) {
  dim3 blocks, threads;
  KERNEL_CONFIGURE(inbox->count, blocks, threads);
  // Invoke the appropriate CUDA kernel to perform the scatter functionality
  betweenness_scatter_kernel<<<blocks, threads, 0, par->streams[1]>>>
    (*inbox, state->distance[par->id], state->numSPs[par->id], state->level);
  CALL_CU_SAFE(hipGetLastError());
}

/**
 * Update the number of shortest paths from remote vertices
 * Also update distance if it has yet to be initialized
 */
PRIVATE void betweenness_scatter_forward(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;

  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;

  for (int rmt_pid = 0; rmt_pid < engine_partition_count(); rmt_pid++) {
    if (rmt_pid == par->id) continue;
    // For all remote partitions, get the corresponding inbox
    grooves_box_table_t* inbox = &par->inbox[rmt_pid];
    if (!inbox->count) continue;
    // If the inbox has some values, determine which type of processing unit
    // corresponds to this partition and call the appropriate scatter function
    if (!engine_get_comm_prev(rmt_pid)) continue;
    if (par->processor.type == PROCESSOR_CPU) {
      betweenness_scatter_cpu(par->id, inbox, state);
    } else if (par->processor.type == PROCESSOR_GPU) {
      betweenness_scatter_gpu(par, inbox, state);
    } else {
      assert(false);
    }
  }
}

/*
 * Parallel CPU implementation of betweenness gather function
 */
PRIVATE inline void betweenness_gather_cpu(int pid, grooves_box_table_t* inbox, 
                                           betweenness_state_t* state,
                                           score_t* values) {
  cost_t* distance = state->distance[pid];
  score_t* delta = state->delta[pid];
  OMP(omp parallel for schedule(runtime))
  for (vid_t index = 0; index < inbox->count; index++) {
    vid_t vid = inbox->rmt_nbrs[index];
    // Check whether the vertex's distance is equal to level + 1
    if (distance[vid] == (state->level + 1)) {
      // If it is, we'll pass the vertex's current delta value to neighbouring
      // nodes to be used during their next backward propagation phase
      values[index]  = delta[vid];
    }
  }
}

/*
 * Kernel for betweenness_gather_gpu
 */
__global__ 
void betweenness_gather_kernel(const vid_t* __restrict rmt_nbrs, 
                               const vid_t rmt_nbrs_count,
                               const cost_t* __restrict distance, 
                               const cost_t level, 
                               const score_t* __restrict delta, 
                               score_t* values) {
  vid_t index = THREAD_GLOBAL_INDEX;
  if (index >= rmt_nbrs_count) return;
  vid_t vid = rmt_nbrs[index];
  // Check whether the vertex's distance is equal to level + 1
  if (distance[vid] == level + 1) {
      // If it is, we'll pass the vertex's current delta value to neighbouring
      // nodes to be used during their next backward propagation phase
    values[index]  = delta[vid];
  }
}

/*
 * Parallel GPU implementation of betweenness gather function
 */
PRIVATE inline 
void betweenness_gather_gpu(partition_t* par, grooves_box_table_t* inbox,
                            betweenness_state_t* state, score_t* values) {
  dim3 blocks, threads;
  KERNEL_CONFIGURE(inbox->count, blocks, threads); 
  // Invoke the appropriate CUDA kernel to perform the gather functionality
  betweenness_gather_kernel<<<blocks, threads, 0, par->streams[1]>>>
    (inbox->rmt_nbrs, inbox->count, state->distance[par->id], 
     state->level, state->delta[par->id], values);
  CALL_CU_SAFE(hipGetLastError());
}

/**
 * Pass the number of shortest paths and delta values to neighbouring
 * vertices to be used in the backwards propagation phase
 */
PRIVATE void betweenness_gather_backward(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;

  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;

  if (par->processor.type == PROCESSOR_GPU) {
    if (!state->comm[state->level]) {
      hipMemsetAsync(state->frontier.count, 0, sizeof(vid_t), par->streams[1]);
    } else {
      frontier_update_list_gpu(&state->frontier, state->level, 
                               state->distance[par->id], par->streams[1]);
    }
  }
  
  for (int rmt_pid = 0; rmt_pid < engine_partition_count(); rmt_pid++) {
    if (rmt_pid == par->id) continue;
    grooves_box_table_t* inbox = &par->inbox[rmt_pid]; 
    // For all remote partitions, get the corresponding inbox
    if (!inbox->count) continue;
    score_t* values = (score_t*)inbox->pull_values;

    if (!engine_get_comm_curr(rmt_pid)) continue;
    // If the inbox has some values, determine which type of processing unit
    // corresponds to this partition and call the appropriate gather function
    if (par->processor.type == PROCESSOR_CPU) {
      betweenness_gather_cpu(par->id, inbox, state, values);
    } else if (par->processor.type == PROCESSOR_GPU) {
      betweenness_gather_gpu(par, inbox, state, values);
    } else {
      assert(false);
    }   
  }
}

/**
 * Initializes the state for a round of backward propagation
 */
PRIVATE void betweenness_init_backward(partition_t* par) {
  if (!par->subgraph.vertex_count) return;
  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
  assert(state);
  vid_t vcount = par->subgraph.vertex_count;

  // Determine which type of memory this partition corresponds to
  totem_mem_t type = TOTEM_MEM_HOST; 
  if (par->processor.type == PROCESSOR_GPU) { 
    type = TOTEM_MEM_DEVICE;
  }

  // Initialize the delta values to 0
  CALL_SAFE(totem_memset(state->delta[par->id], (score_t)0, vcount, type, 
                         par->streams[1]));

  state->level--;
}

/**
 * Initializes the state for a round of forward propagation
 */
PRIVATE void betweenness_init_forward(partition_t* par) {
  if (!par->subgraph.vertex_count) return;
  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
  assert(state);
  // Get the source partition and source vertex values
  id_t src_pid = GET_PARTITION_ID(bc_g.src);
  id_t src_vid = GET_VERTEX_ID(bc_g.src);
  vid_t vcount = par->subgraph.vertex_count;

  // Determine which type of memory this partition corresponds to
  totem_mem_t type = TOTEM_MEM_HOST; 
  if (par->processor.type == PROCESSOR_GPU) { 
    type = TOTEM_MEM_DEVICE;
  }

  // Initialize the distances to infinity and numSPs to 0
  for (int pid = 0; pid < engine_partition_count(); pid++) {
    vid_t count = (pid != par->id) ? par->outbox[pid].count : vcount;
    if (count) {
      CALL_SAFE(totem_memset((state->distance[pid]), INF_COST, count, type, 
                             par->streams[1]));
      CALL_SAFE(totem_memset((state->numSPs[pid]), (uint32_t)0, count, type, 
                             par->streams[1]));
    }
  }
  if (src_pid == par->id) {
    // For the source vertex, initialize its own distance and numSPs
    CALL_SAFE(totem_memset(&((state->distance[par->id])[src_vid]), (cost_t)0,
                           1, type, par->streams[1]));
    CALL_SAFE(totem_memset(&((state->numSPs[par->id])[src_vid]), (uint32_t)1,
                           1, type, par->streams[1]));
  }
  
  // Initialize the outbox to 0 and set the level to 0
  engine_set_outbox(par->id, 0); 
  state->level = 0;
  totem_memset(state->comm, true, engine_vertex_count(), TOTEM_MEM_HOST);
}

/**
 * Allocates and initializes the state for Betweenness Centrality
 */
PRIVATE void betweenness_init(partition_t* par) {
  if (!par->subgraph.vertex_count) return;
  // Allocate memory for the per-partition state
  betweenness_state_t* state = (betweenness_state_t*)
                               calloc(1, sizeof(betweenness_state_t));
  assert(state); 
  // Set the partition's state variable to the previously allocated state
  par->algo_state = state;
  vid_t vcount = par->subgraph.vertex_count;

  // Determine which type of memory this partition corresponds to
  totem_mem_t type = TOTEM_MEM_HOST; 
  if (par->processor.type == PROCESSOR_GPU) { 
    type = TOTEM_MEM_DEVICE;
    frontier_init_gpu(&state->frontier, par->subgraph.vertex_count);
  }
  
  CALL_SAFE(totem_calloc(vcount * sizeof(score_t), type,
                         (void**)&(state->delta[par->id])));
  CALL_SAFE(totem_calloc(vcount * sizeof(score_t), type,
                         (void**)&(state->betweenness)));

  // Allocate memory for the various pieces of data required for the
  // Betweenness Centrality algorithm
  for (int pid = 0; pid < engine_partition_count(); pid++) {
    vid_t count = (pid != par->id) ? par->outbox[pid].count : vcount;
    if (count) {
      CALL_SAFE(totem_malloc(count * sizeof(cost_t), type, 
                             (void**)&(state->distance[pid])));
      CALL_SAFE(totem_calloc(count * sizeof(uint32_t), type, 
                             (void**)&(state->numSPs[pid])));
    }    
    state->numSPs_f[pid] = state->numSPs[pid];
  }

  // Initialize the state's done flag
  state->done = engine_get_finished_ptr(par->id);

  // Initialize the comm array
  totem_calloc(engine_vertex_count(), TOTEM_MEM_HOST, (void**)&state->comm);

  // Initialize the state
  betweenness_init_forward(par);
}

/**
 * Cleans up allocated memory on the CPU and GPU
 */
PRIVATE void betweenness_finalize(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;  

  // Free the allocated memory
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
 
  // Determine which type of memory this partition corresponds to
  totem_mem_t type = TOTEM_MEM_HOST; 
  if (par->processor.type == PROCESSOR_GPU) { 
    type = TOTEM_MEM_DEVICE; 
    frontier_finalize_gpu(&state->frontier);
  }

  // Free the memory allocated for the algorithm
  for (int pid = 0; pid < engine_partition_count(); pid++) {
    totem_free(state->distance[pid], type);
    totem_free(state->numSPs[pid], type);
  }
  totem_free(state->delta[par->id], type);
  totem_free(state->betweenness, type);
  totem_free(state->comm, TOTEM_MEM_HOST);

  // Free the per-partition state and set it to NULL
  free(state);
  par->algo_state = NULL;
}

/**
 * Aggregates the final result to be returned at the end
 */
PRIVATE void betweenness_aggr(partition_t* par) {  
  if (!par->subgraph.vertex_count) return;
  // Get the current state of the algorithm
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
  graph_t* subgraph = &par->subgraph;
  score_t* betweenness_values = NULL;
  // Determine which type of processor this partition corresponds to
  if (par->processor.type == PROCESSOR_CPU) {
    // If it is a CPU partition, grab the computed betweenness value directly
    betweenness_values = state->betweenness;
  } else if (par->processor.type == PROCESSOR_GPU) {
    // If it is a GPU partition, copy the computed score back to the host
    assert(bc_g.betweenness_score_h);
    CALL_CU_SAFE(hipMemcpy(bc_g.betweenness_score_h, state->betweenness, 
                            subgraph->vertex_count * sizeof(score_t),
                            hipMemcpyDefault));
    betweenness_values = bc_g.betweenness_score_h;
  } else {
    assert(false);
  }
  // Aggregate the results
  assert(bc_g.betweenness_score);
  OMP(omp parallel for schedule(static))
  for (vid_t v = 0; v < subgraph->vertex_count; v++) {
    // Check whether we are computing exact centrality values
    if (bc_g.epsilon == CENTRALITY_EXACT) {
      // Return the exact values computed
      bc_g.betweenness_score[par->map[v]] = betweenness_values[v];
    } else {
      // Scale the computed Betweenness Centrality metrics since they were
      // computed using a subset of the total nodes within the graph
      // The scaling value is: (Total Number of Nodes / Subset of Nodes Used)
      bc_g.betweenness_score[par->map[v]] = betweenness_values[v] *
        (score_t)(((double)(engine_vertex_count())) / bc_g.num_samples); 
    }
  }
}

/**
 * The following two functions are the kernel and gather callbacks of a single 
 * BSP cycle that synchronizes the distance of remote vertices
 */
PRIVATE void betweenness_gather_distance(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;
  if (engine_superstep() == 1) {
    betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
    assert(state);
    engine_gather_inbox(par->id, state->distance[par->id]);
  }
}
PRIVATE void betweenness_synch_distance(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
  assert(state);
  if (engine_superstep() == 1) {
    engine_report_not_finished();
  } else {
    for (int rmt_pid = 0; rmt_pid < engine_partition_count(); rmt_pid++) {
      if (par->id == rmt_pid) continue;
      if (par->processor.type == PROCESSOR_GPU) {
        CALL_CU_SAFE(hipMemcpyAsync(state->distance[rmt_pid], 
                                     par->outbox[rmt_pid].pull_values,
                                     par->outbox[rmt_pid].count * 
                                     sizeof(cost_t),
                                     hipMemcpyDefault, par->streams[1]));
      } else {
        cost_t* src = (cost_t*)par->outbox[rmt_pid].pull_values;
        cost_t* dst = state->distance[rmt_pid];
        OMP(omp parallel for schedule(static))
        for (int i = 0; i < par->outbox[rmt_pid].count; i++) {
          dst[i] = src[i];
        }
      }
    }
  }
}

/**
 * The following two functions are the kernel and gather callbacks of a single 
 * BSP cycle that synchronizes the numSPs of remote vertices
 */
PRIVATE void betweenness_gather_numSPs(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;
  if (engine_superstep() == 1) {
    betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
    assert(state);
    engine_gather_inbox(par->id, state->numSPs[par->id]);
  }
}
PRIVATE void betweenness_synch_numSPs(partition_t* par) {
  // Check if there is no work to be done
  if (!par->subgraph.vertex_count) return;
  betweenness_state_t* state = (betweenness_state_t*)par->algo_state;
  assert(state);
  if (engine_superstep() == 1) {
    engine_report_not_finished();
  } else {
    for (int rmt_pid = 0; rmt_pid < engine_partition_count(); rmt_pid++) {
      if (par->id == rmt_pid) continue;
      if (par->processor.type == PROCESSOR_GPU) {
        CALL_CU_SAFE(hipMemcpyAsync(state->numSPs[rmt_pid], 
                                     par->outbox[rmt_pid].pull_values,
                                     par->outbox[rmt_pid].count * 
                                     sizeof(uint32_t),
                                     hipMemcpyDefault, par->streams[1]));
      } else {
        uint32_t* src = (uint32_t*)par->outbox[rmt_pid].pull_values;
        uint32_t* dst = state->numSPs[rmt_pid];
        OMP(omp parallel for schedule(static))
        for (int i = 0; i < par->outbox[rmt_pid].count; i++) {
          dst[i] = src[i];
        }
      }
    }
  }
}

/**
 * Core functionality for main for loop within the BC computation
 */
void betweenness_hybrid_core(vid_t source, bool is_first_iteration,
                             bool is_last_iteration) {
  // Set the source node for this iteration
  bc_g.src  = engine_vertex_id_in_partition(source);

  // Forward propagation
  engine_par_init_func_t init_forward = betweenness_init_forward;
  if (is_first_iteration) {
    init_forward = betweenness_init;
  }
  // Configure the parameters for forward propagation given the current
  // iteration of the overall computation
  engine_config_t config_forward = {
    NULL, betweenness_forward, betweenness_scatter_forward, NULL, 
    init_forward, NULL, NULL, GROOVES_PUSH
  };
  // Call Totem to begin the computation phase given the specified 
  // configuration
  engine_config(&config_forward);
  engine_execute();

  // Synchronize the distance and numSPs state, which have been calculated in
  // the forward phase, across all partitions. This state will be used in the
  // backward propagation phase
  engine_config_t config_distance_state = {
    NULL, betweenness_synch_distance, NULL, betweenness_gather_distance,
    NULL, NULL, NULL, GROOVES_PULL
  };
  engine_config(&config_distance_state);
  engine_execute();
  engine_config_t config_numSPs_state = {
    NULL, betweenness_synch_numSPs, NULL, betweenness_gather_numSPs,
    NULL, NULL, NULL, GROOVES_PULL
  };
  engine_config(&config_numSPs_state);
  engine_execute();

  // Backward propagation
  engine_par_finalize_func_t finalize_backward = NULL;
  engine_par_aggr_func_t aggr_backward = NULL;
  if (is_last_iteration) {
    finalize_backward = betweenness_finalize;
    aggr_backward = betweenness_aggr;
  }
  // Configure the parameters for backward propagation given the current
  // iteration of the overall computation
  engine_config_t config_backward = {
    NULL, betweenness_backward, NULL, betweenness_gather_backward,
    betweenness_init_backward, finalize_backward, aggr_backward, GROOVES_PULL
  };
  // Call Totem to begin the computation phase given the specified
  // configuration
  engine_config(&config_backward);
  engine_execute();
}

/**
 * Main function for hybrid betweenness centrality
 */
error_t betweenness_hybrid(double epsilon, score_t* betweenness_score) {
  // Sanity check on input
  bool finished = false;
  error_t rc = betweenness_check_special_cases(engine_get_graph(), 
                                               &finished, betweenness_score);
  if (finished) return rc;

  // Initialize the global state
  memset(&bc_g, 0, sizeof(bc_g));
  bc_g.betweenness_score = betweenness_score;
  CALL_SAFE(totem_memset(bc_g.betweenness_score, (score_t)0, 
                         engine_vertex_count(), TOTEM_MEM_HOST));
  bc_g.epsilon = epsilon;

  if (engine_largest_gpu_partition()) {
    CALL_SAFE(totem_malloc(engine_largest_gpu_partition() * sizeof(score_t),
                           TOTEM_MEM_HOST_PINNED, 
                           (void**)&bc_g.betweenness_score_h));
  }

  // Determine whether we will compute exact or approximate BC values
  if (epsilon == CENTRALITY_EXACT) {
    // Compute exact values for Betweenness Centrality
    vid_t vcount = engine_vertex_count();
    for (vid_t source = 0; source < vcount; source++) { 
      betweenness_hybrid_core(source, (source == 0), (source == (vcount-1)));  
    }
  } else {
    // Compute approximate values based on the value of epsilon provided
    // Select a subset of source nodes to make the computation faster
    int num_samples = centrality_get_number_sample_nodes(engine_vertex_count(),
                                                         epsilon);
    // Store the number of samples used in the global state to be used for 
    // scaling the computed metric during aggregation
    bc_g.num_samples = num_samples;
    // Populate the array of indices to sample
    vid_t* sample_nodes = centrality_select_sampling_nodes(
                          engine_get_graph(), num_samples);
 
    for (int source_index = 0; source_index < num_samples; source_index++) {
      // Get the next sample node in the array to use as a source
      vid_t source = sample_nodes[source_index];    
      betweenness_hybrid_core(source, (source_index == 0), 
                              (source_index == (num_samples-1)));  
    } 
 
    // Clean up the allocated memory
    free(sample_nodes);
  }
 
  // Clean up and return
  if (engine_largest_gpu_partition()) { 
    totem_free(bc_g.betweenness_score_h, TOTEM_MEM_HOST_PINNED);
  }
  memset(&bc_g, 0, sizeof(betweenness_global_state_t));
  return SUCCESS;
}
