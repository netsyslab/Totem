#include "hip/hip_runtime.h"
/*
 * This file contains an implementation of the breadth-first search (BFS) graph
 * search algorithm based on the algorithms in [Hong2011PPoPP, Hong2011PACT].
 * [Hong2011PPoPP] S. Hong,  S.K. Kim, T. Oguntebi and K. Olukotun, 
 *   "Accelerating CUDA graph algorithms at maximum warp" in PPoPP 2011.
 * [Hong2011PACT] S. Hong, T. Oguntebi and K. Olukotun, "Efficient parallel 
 *   graph exploration on multi-core cpu and gpu" in PACT 2011.
 *
 *  Created on: 2011-02-28
 *      Author: Lauro Beltrão Costa
 *              Abdullah Gharaibeh
 */

// totem includes
#include "totem_alg.h"

/**
 * This structure is used by the virtual warp-based implementation. It stores a
 * batch of work. It is allocated on shared memory and is processed by a single
 * virtual warp.
 */
typedef struct {
  // One is added to make it easy to calculate the number of neighbors of the
  // last vertex. Another one is added to ensure 8Bytes alignment irrespective
  // whether sizeof(eid_t) is 4 or 8. Alignment is enforced for performance
  // reasons.
  eid_t vertices[VWARP_DEFAULT_BATCH_SIZE + 2];
  cost_t cost[VWARP_DEFAULT_BATCH_SIZE];
} vwarp_mem_t;

PRIVATE error_t check_special_cases(graph_t* graph, vid_t src_id,
                                    cost_t* cost, bool* finished) {
  *finished = true;
  if ((graph == NULL) || (src_id >= graph->vertex_count) || (cost == NULL)) {
    return FAILURE;
  } else if (graph->vertex_count == 1) {
    cost[0] = 0;
    return SUCCESS;
  } else if (graph->edge_count == 0) {
    // Initialize cost to INFINITE and zero to the source node
    totem_memset(cost, INF_COST, graph->vertex_count, TOTEM_MEM_HOST);
    cost[src_id] = 0;
    return SUCCESS;
  }
  *finished = false;
  return SUCCESS;
}

/**
 * A common initialization function for GPU implementations. It allocates and
 * initalizes state on the GPU
*/
PRIVATE
error_t initialize_gpu(const graph_t* graph, vid_t source_id, vid_t cost_len,
                       graph_t** graph_d, cost_t** cost_d, bool** finished_d) {
  // Allocate space on GPU
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);
  CHK_SUCCESS(totem_malloc(cost_len * sizeof(cost_t), TOTEM_MEM_DEVICE,
                          reinterpret_cast<void**>(cost_d)), err_free_graph_d);
  // Initialize cost to INFINITE.
  totem_memset(*cost_d, INF_COST, cost_len, TOTEM_MEM_DEVICE);
  // For the source vertex, initialize cost.
  totem_memset(&((*cost_d)[source_id]), (cost_t)0, 1, TOTEM_MEM_DEVICE);
  // Allocate the termination flag
  CHK_SUCCESS(totem_malloc(sizeof(bool), TOTEM_MEM_DEVICE,
                           reinterpret_cast<void**>(finished_d)),
              err_free_cost_d_graph_d);
  return SUCCESS;

  err_free_cost_d_graph_d:
    totem_free(cost_d, TOTEM_MEM_DEVICE);
  err_free_graph_d:
    graph_finalize_device(*graph_d);
  err:
    return FAILURE;
}

/**
 * A common finalize function for GPU implementations. It allocates the host
 * output buffer, moves the final results from GPU to the host buffers and
 * frees up some resources.
*/
PRIVATE
error_t finalize_gpu(graph_t* graph_d, bool* finished_d,
                     cost_t* cost_d, cost_t* cost) {
  CHK_CU_SUCCESS(hipMemcpy(cost, cost_d, graph_d->vertex_count *
                            sizeof(cost_t), hipMemcpyDeviceToHost), err);
  graph_finalize_device(graph_d);
  totem_free(finished_d, TOTEM_MEM_DEVICE);
  totem_free(cost_d, TOTEM_MEM_DEVICE);
  return SUCCESS;
 err:
  return FAILURE;
}

/* This comment describes implementation details of the next two functions.
 * Modified from [Harish07].
 * Breadth First Search
 * This implementation uses level synchronization. BFS traverses the graph
 * in levels; once a level is visited it is not visited again. The BFs frontier
 * corresponds to all the nodes being processed at the current level.
 * Each thread processes a vertex (in the following text these terms are used
 * interchangeably). An integer array, cost_d, stores the minimal number of 
 * edges from the source vertex to each vertex. The cost for vertices that have
 * not been visited yet is INFINITE. In each iteration, each vertex checks if it
 * belongs to the current level by verifying its own cost. If it does, it
 * updates its not yet visited neighbors. If the cost of, at least, one neighbor
 * is updated, the variable finished_d is set to false and there will be another
 * iteration.
 */
__global__
void bfs_kernel(graph_t graph, cost_t level, bool* finished, cost_t* cost) {
  const vid_t vertex_id = THREAD_GLOBAL_INDEX;
  if (vertex_id >= graph.vertex_count) return;
  if (cost[vertex_id] != level) return;
  for (eid_t i = graph.vertices[vertex_id];
       i < graph.vertices[vertex_id + 1]; i++) {
    const vid_t neighbor_id = graph.edges[i];
    if (cost[neighbor_id] == INF_COST) {
      // Threads may update finished and the same position in the cost array
      // concurrently. It does not affect correctness since all
      // threads would update with the same value.
      *finished = false;
      cost[neighbor_id] = level + 1;
    }
  }  // for
}

/**
 * The neighbors processing function. This function sets the level of the
 * neighbors' vertex to one level more than the vertex. The assumption is that
 * the threads of a warp invoke this function to process the warp's batch of
 * work. In each iteration of the for loop, each thread processes a neighbor.
 * For example, thread 0 in the warp processes neighbors at indices 0,
 * VWARP_DEFAULT_WARP_WIDTH, (2 * VWARP_DEFAULT_WARP_WIDTH) etc. in the edges
 * array, while thread 1 in the warp processes neighbors 1,
 * (1 + VWARP_DEFAULT_WARP_WIDTH), (1 + 2 * VWARP_DEFAULT_WARP_WIDTH) and so on.
*/
__device__
void vwarp_process_neighbors(vid_t warp_offset, vid_t neighbor_count,
                             vid_t* neighbors, cost_t* cost, cost_t level,
                             bool* finished) {
  for (vid_t i = warp_offset; i < neighbor_count;
      i += VWARP_DEFAULT_WARP_WIDTH) {
    vid_t neighbor_id = neighbors[i];
    if (cost[neighbor_id] == INF_COST) {
      cost[neighbor_id] = level + 1;
      *finished = false;
    }
  }
}

/**
 * A warp-based implementation of the BFS kernel. Please refer to the
 * description of the warp technique for details. Also, please refer to
 * bfs_kernel for details on the BFS implementation.
 */
__global__
void vwarp_bfs_kernel(graph_t graph, cost_t level, bool* finished,
                      cost_t* cost, uint32_t thread_count) {
  if (THREAD_GLOBAL_INDEX >= thread_count) return;
  vid_t warp_offset = THREAD_GLOBAL_INDEX % VWARP_DEFAULT_WARP_WIDTH;
  vid_t warp_id     = THREAD_GLOBAL_INDEX / VWARP_DEFAULT_WARP_WIDTH;

  __shared__ vwarp_mem_t shared_memory[(MAX_THREADS_PER_BLOCK /
                                        VWARP_DEFAULT_WARP_WIDTH)];
  vwarp_mem_t* my_space = &shared_memory[THREAD_BLOCK_INDEX /
                                         VWARP_DEFAULT_WARP_WIDTH];

  // copy my work to local space
  vid_t v_ = warp_id * VWARP_DEFAULT_BATCH_SIZE;
  vwarp_memcpy(my_space->cost, &cost[v_], VWARP_DEFAULT_BATCH_SIZE,
               warp_offset);
  vwarp_memcpy(my_space->vertices, &(graph.vertices[v_]),
               VWARP_DEFAULT_BATCH_SIZE + 1, warp_offset);

  // iterate over my work
  for (vid_t v = 0; v < VWARP_DEFAULT_BATCH_SIZE; v++) {
    if (my_space->cost[v] == level) {
      vid_t neighbor_count = my_space->vertices[v + 1] - my_space->vertices[v];
      vid_t* neighbors = &(graph.edges[my_space->vertices[v]]);
      vwarp_process_neighbors(warp_offset, neighbor_count, neighbors, cost,
                              level, finished);
    }
  }
}

__host__
error_t bfs_vwarp_gpu(graph_t* graph, vid_t source_id, cost_t* cost) {
  // Check for special cases
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, cost, &finished);
  if (finished) return rc;

  // Create and initialize state on GPU
  graph_t* graph_d;
  cost_t* cost_d;
  bool* finished_d;
  CHK_SUCCESS(initialize_gpu(graph, source_id,
                             vwarp_default_state_length(graph->vertex_count),
                             &graph_d, &cost_d, &finished_d), err_free_all);

  // {} used to limit scope and avoid problems with error handles.
  {
  // Configure the kernel's threads and on-chip memory. On-ship memory is
  // configured as shared memory rather than L1 cache.
  dim3 blocks;
  dim3 threads_per_block;
  KERNEL_CONFIGURE(vwarp_default_thread_count(graph->vertex_count), blocks,
                   threads_per_block);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(vwarp_bfs_kernel), hipFuncCachePreferShared);
  bool finished = false;
  // while the current level has vertices to be processed.
  for (cost_t level = 0; !finished; level++) {
    CHK_CU_SUCCESS(hipMemset(finished_d, true, 1), err_free_all);
    vwarp_bfs_kernel<<<blocks, threads_per_block>>>
      (*graph_d, level, finished_d, cost_d,
       vwarp_default_thread_count(graph->vertex_count));
    CHK_CU_SUCCESS(hipGetLastError(), err_free_all);
    CHK_CU_SUCCESS(hipMemcpy(&finished, finished_d, sizeof(bool),
                              hipMemcpyDeviceToHost), err_free_all);
  }
  }

  CHK_SUCCESS(finalize_gpu(graph_d, finished_d, cost_d, cost), err_free_all);
  return SUCCESS;

  // error handlers
  err_free_all:
    totem_free(finished_d, TOTEM_MEM_DEVICE);
    totem_free(cost_d, TOTEM_MEM_DEVICE);
    graph_finalize_device(graph_d);
    return FAILURE;
}

__host__
error_t bfs_gpu(graph_t* graph, vid_t source_id, cost_t* cost) {
  // Check for special cases.
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, cost, &finished);
  if (finished) return rc;

  // Create and initialize state on GPU.
  graph_t* graph_d;
  cost_t* cost_d;
  bool* finished_d;
  CHK_SUCCESS(initialize_gpu(graph, source_id, graph->vertex_count,
                             &graph_d, &cost_d, &finished_d), err_free_all);

  // {} used to limit scope and avoid problems with error handles.
  {
  dim3 blocks;
  dim3 threads_per_block;
  KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);
  bool finished = false;
  // while the current level has vertices to be processed.
  for (cost_t level = 0; !finished; level++) {
    CHK_CU_SUCCESS(hipMemset(finished_d, true, 1), err_free_all);
    // for each vertex V in parallel do
    bfs_kernel<<<blocks, threads_per_block>>>(*graph_d, level, finished_d,
                                              cost_d);
    CHK_CU_SUCCESS(hipGetLastError(), err_free_all);
    CHK_CU_SUCCESS(hipMemcpy(&finished, finished_d, sizeof(bool),
                              hipMemcpyDeviceToHost), err_free_all);
  }}

  // We are done, get the results back and clean up state.
  CHK_SUCCESS(finalize_gpu(graph_d, finished_d, cost_d, cost), err_free_all);
  return SUCCESS;

  // error handlers
  err_free_all:
    totem_free(finished_d, TOTEM_MEM_DEVICE);
    totem_free(cost_d, TOTEM_MEM_DEVICE);
    graph_finalize_device(graph_d);
    return FAILURE;
}

PRIVATE bitmap_t initialize_cpu(graph_t* graph, vid_t source_id,
                                cost_t* cost) {
  // Initialize cost to INFINITE and create the vertices bitmap.
  totem_memset(cost, INF_COST, graph->vertex_count, TOTEM_MEM_HOST);
  bitmap_t visited = bitmap_init_cpu(graph->vertex_count);

  // Initialize the state of the source vertex.
  cost[source_id] = 0;
  bitmap_set_cpu(visited, source_id);
  return visited;
}

__host__
error_t bfs_cpu(graph_t* graph, vid_t source_id, cost_t* cost) {
  // Check for special cases
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, cost, &finished);
  if (finished) return rc;

  bitmap_t visited = initialize_cpu(graph, source_id, cost);

  finished = false;
  // Within the following code segment, all threads execute in parallel the
  // same code (similar to a cuda kernel)
  OMP(omp parallel) {
    // level is a local variable to each thread, having a separate copy per
    // thread reduces the overhead of cache coherency protocol compared to
    // the case where level is shared
    cost_t level = 0;
    // while the current level has vertices to be processed.
    while (!finished) {
      // The following barrier is necessary to ensure that all threads have
      // checked the while condition above using the same "finished" value
      // that resulted from the previous iteration before it is initialized
      // again for the next one.
      OMP(omp barrier)

      // This "single" clause ensures that only one thread sets the variable.
      // Note that this close has an implicit barrier (i.e., all threads will
      // block until the variable is set by the responsible thread)
      OMP(omp single)
      finished = true;

      // The "for" clause instructs openmp to run the loop in parallel. Each
      // thread will be assigned a chunk of work depending on the chosen OMP
      // scheduling algorithm. The reduction clause tells openmp to define a
      // private temporary variable for each thread, and reduce them in the
      // end using an "and" operator and store the value in "finished". Similar
      // to the argument above, this improves performance by reducing cache
      // coherency overhead. The "runtime" scheduling clause defer the choice
      // of thread scheduling algorithm to the choice of the client, either
      // via OS environment variable or omp_set_schedule interface.
      OMP(omp for schedule(runtime) reduction(& : finished))
      for (vid_t vertex_id = 0; vertex_id < graph->vertex_count; vertex_id++) {
        if (cost[vertex_id] != level) continue;
        for (eid_t i = graph->vertices[vertex_id];
             i < graph->vertices[vertex_id + 1]; i++) {
          const vid_t neighbor_id = graph->edges[i];
          if (!bitmap_is_set(visited, neighbor_id)) {
            if (bitmap_set_cpu(visited, neighbor_id)) {
              finished = false;
              cost[neighbor_id] = level + 1;
            }
          }
        }
      }
      level++;
    }
  }  // omp parallel
  bitmap_finalize_cpu(visited);
  return SUCCESS;
}

PRIVATE void allocate_frontiers(graph_t* graph, vid_t** currF, vid_t** nextF,
                                vid_t*** localFs) {
  int thread_count = omp_get_max_threads();
  // Allocate a local queue for each thread.
  *localFs = reinterpret_cast<vid_t**>(malloc(thread_count * sizeof(vid_t*)));
  for (int tid = 0; tid < thread_count; tid++) {
    // allocate space assuming the worst case: all the vertices are
    // pushed to the local queue of a thread.
    // TODO(abdullah): reduce the memory footprint of the local stacks
    //                 (e.g., coarse-grained dynamic expansion of stack size)
    (*localFs)[tid] = reinterpret_cast<vid_t*>(malloc(graph->vertex_count *
                                               sizeof(vid_t)));
    assert((*localFs)[tid]);
  }
  *currF = reinterpret_cast<vid_t*>(malloc(graph->vertex_count *
                                    sizeof(vid_t)));
  *nextF = reinterpret_cast<vid_t*>(malloc(graph->vertex_count *
                                    sizeof(vid_t)));
  assert(*currF && *nextF);
}

PRIVATE void free_frontiers(vid_t* currF, vid_t* nextF, vid_t** localFs) {
  int thread_count = omp_get_max_threads();
  for (int tid = 0; tid < thread_count; tid++) {
    free(localFs[tid]);
  }
  free(localFs);
  free(currF);
  free(nextF);
}

/* Based on the implementation by Agarwal et al.
 * The implementation uses two arrays that maintains the current and next 
 * frontier. The current frontier array contains the vertices that are being 
 * visited in the current level. While the vertices in the current frontier
 * are being processed, their not-visited neighbors are stored in the next 
 * frontier array. Once the current level is done, the next frontier array 
 * becomes the current frontier array, and the processing of the next level
 * starts. To improve performance, this implementation uses what is called local
 * next frontier arrays: each thread has a local next array that in the end 
 * merged into the global next array. This improves performance by getting rid
 * of the required synchronization if the threads were to access the global next
 * array directly.
 */
__host__
error_t bfs_queue_cpu(graph_t* graph, vid_t source_id, cost_t* cost) {
  // Check for special cases
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, cost, &finished);
  if (finished) return rc;
  bitmap_t visited = initialize_cpu(graph, source_id, cost);

  // Initialize queues
  vid_t currF_index = 0;
  vid_t nextF_index = 0;
  vid_t* currF = NULL;
  vid_t* nextF = NULL;
  vid_t** localFs = NULL;
  allocate_frontiers(graph, &currF, &nextF, &localFs);

  // Do level 0 separately and parallelize across neighbours.
  // Only the source node is active in level 0
  OMP(omp parallel for schedule(static))
  for (vid_t v = graph->vertices[source_id];
       v < graph->vertices[source_id + 1]; v++) {
    vid_t nbr = graph->edges[v];
    cost[nbr] = 1;
    bitmap_set_cpu(visited, nbr);
    nextF[__sync_fetch_and_add(&nextF_index, 1)] = nbr;
  }


  OMP(omp parallel) {
    // thread-local variables
    cost_t level        = 1;
    vid_t  localF_index = 0;
    vid_t* localF       = localFs[omp_get_thread_num()];

    // while the current level has vertices to be processed.
    while (nextF_index > 0) {
      // The following barrier is necessary to ensure that all threads have
      // checked the while condition before nextF_index is cleared for the next
      // round.
      OMP(omp barrier)

      // This "single" clause ensures that only one thread enters the
      // following block of code. Note that this close has an implicit
      // barrier.
      OMP(omp single) {
        // Swap the current with the next queue.
        vid_t* tmp = currF;
        currF = nextF;
        nextF = tmp;
        currF_index = nextF_index;
        nextF_index = 0;
      }
      localF_index = 0;

      // The "for" clause instructs openmp to run the loop in parallel. Each
      // thread will be assigned a chunk of work depending on the chosen
      // OMP scheduling algorithm. The "runtime" scheduling clause defer the
      // choice of thread scheduling algorithm to the choice of the client,
      // either via OS environment variable or omp_set_schedule interface.
      OMP(omp for schedule(runtime))
      for (vid_t q = 0; q < currF_index; q++) {
        vid_t v = currF[q];
        for (eid_t i = graph->vertices[v]; i < graph->vertices[v + 1]; i++) {
          const vid_t nbr = graph->edges[i];
          if (!bitmap_is_set(visited, nbr)) {
            if (bitmap_set_cpu(visited, nbr)) {
              cost[nbr] = level + 1;
              localF[localF_index++] = nbr;
            }
          }
        }
      }
      if (localF_index > 0) {
        vid_t idx = __sync_fetch_and_add(&nextF_index, localF_index);
        memcpy(&(nextF[idx]), localF, localF_index * sizeof(vid_t));
      }
      level++;

      // The following barrier is necessary to ensure that all threads see the
      // same nextF_index value that is being incremented by the localF_indices
      OMP(omp barrier)
    }
  }  // omp parallel
  bitmap_finalize_cpu(visited);
  free_frontiers(currF, nextF, localFs);
  return SUCCESS;
}

// A classic top down step that iterates over vertices in the frontier
// and tries to add their neighbours to the next frontier.
bool top_down_step(graph_t* graph, cost_t* cost, bitmap_t* visited,
                   frontier_state_t* state, cost_t level) {
  bool finished = true;

  // Iterate across all vertices in frontier.
  OMP(omp parallel for schedule(runtime) reduction(& : finished))
  for (vid_t vertex_id = 0; vertex_id < graph->vertex_count; vertex_id++) {
    if (!bitmap_is_set(state->current, vertex_id)) continue;

    // Iterate across all neighbours of this vertex.
    for (eid_t i = graph->vertices[vertex_id];
         i < graph->vertices[vertex_id + 1]; i++) {
      const vid_t neighbor_id = graph->edges[i];

      // If already visited, ignore neighbour.
      if (!bitmap_is_set(*visited, neighbor_id)) {
        if (bitmap_set_cpu(*visited, neighbor_id)) {
          // If a new vertex is now visited, we have a new level
          // of frontier - we are not finished.
          finished = false;
          // Increment the level of this neighbour.
          cost[neighbor_id] = level + 1;
        }
      }
    }
  }  // End of omp for

  return finished;
}

// A step that iterates across unvisited vertices and determines
// their status in the next frontier.
bool bottom_up_step(graph_t* graph, cost_t* cost, bitmap_t* visited,
                    frontier_state_t* state, cost_t level) {
  bool finished = true;

  // Iterate across all vertices.
  OMP(omp parallel for schedule(runtime) reduction(& : finished))
  for (vid_t vertex_id = 0; vertex_id < graph->vertex_count; vertex_id++) {
    // Ignore vertices that have been visited.
    if (bitmap_is_set(*visited, vertex_id)) { continue; }

    // Iterate across the neighbours of this vertex.
    for (eid_t i = graph->vertices[vertex_id];
         i < graph->vertices[vertex_id + 1]; i++) {
      if (bitmap_is_set(state->current, graph->edges[i])) {
        // Add the vertex we are exploring to the next frontier.
        bitmap_set_cpu(*visited, vertex_id);
        finished = false;
        // Increment the level of this vertex.
        cost[vertex_id] = level + 1;
        break;
      }
    }  // End of neighbour check - vertex examined.
  }  // All vertices examined in level.
  return finished;
}

/* Similar to the regular BFS for cpu, the difference being choosing
 * a step for each level is now possible.
 * Based off of the work by Scott Beamer et al.
 * Searching for a Parent Instead of Fighting Over Children: A Fast 
 * Breadth-First Search Implementation for Graph500.
 * http://www.eecs.berkeley.edu/Pubs/TechRpts/2011/EECS-2011-117.pdf
 */
__host__
error_t bfs_bu_cpu(graph_t* graph, vid_t source_id, cost_t* cost) {
  // TODO(scott): Make this a heuristic instead of a constant.
  const vid_t SMALL_THRESHOLD = graph->vertex_count/(16*16*16*16);

  // Check for special cases.
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, cost, &finished);
  if (finished) return rc;

  // Initialize frontier set-up.
  frontier_state_t state;
  frontier_init_cpu(&state, graph->vertex_count);

  // Initialize the bitmap.
  bitmap_t visited = initialize_cpu(graph, source_id, cost);

  // Start by assuming we are not done yet.
  finished = false;
  cost_t level = 0;

  // Complete a step for each level, while not finished.
  while (!finished) {
    // Update the frontier.
    frontier_update_bitmap_cpu(&state, visited);

    // Choose a top-down step if the frontier count is small.
    if (bitmap_count_cpu(state.current, graph->vertex_count)
                         < SMALL_THRESHOLD) {
      finished = top_down_step(graph, cost, &visited, &state, level);
    } else {
      // Choose a bottom-up step if the frontier count is large.
      finished = bottom_up_step(graph, cost, &visited, &state, level);
    }
    level++;
  }

  bitmap_finalize_cpu(visited);
  return SUCCESS;
}

// A gpu version of the Top-down step as a kernel.
__global__
void bfs_td_kernel(graph_t graph, cost_t level, cost_t* cost,
                   bitmap_t visited, frontier_state_t state) {
  const vid_t vertex_id = THREAD_GLOBAL_INDEX;
  if (vertex_id >= graph.vertex_count) { return; }

  // Ignore vertices not in frontier.
  if (!bitmap_is_set(state.current, vertex_id)) { return; }

  // Iterate across all neighbours of the vertex.
  for (eid_t i = graph.vertices[vertex_id];
       i < graph.vertices[vertex_id + 1]; i++) {
    const vid_t neighbor_id = graph.edges[i];

    // If already visited, ignore neighbour.
    if (!bitmap_is_set(visited, neighbor_id)) {
      if (bitmap_set_gpu(visited, neighbor_id)) {
        // Increment the level of this neighbour.
        cost[neighbor_id] = level + 1;
      }
    }
  }
}

// A gpu version of the Bottom-up step as a kernel.
__global__
void bfs_bu_kernel(graph_t graph, cost_t level, cost_t* cost,
                   bitmap_t visited, frontier_state_t state) {
  const vid_t vertex_id = THREAD_GLOBAL_INDEX;
  if (vertex_id >= graph.vertex_count) { return; }

  // Ignore vertices that have been visited.
  if (bitmap_is_set(visited, vertex_id) ) { return; }

  // Iterate across all neighbours of the vertex.
  for (eid_t i = graph.vertices[vertex_id];
       i < graph.vertices[vertex_id + 1]; i++) {
    const vid_t neighbor_id = graph.edges[i];

    // Check if neighbour is in the current frontier.
    if (bitmap_is_set(state.current, neighbor_id)) {
      bitmap_set_gpu(visited, vertex_id);
      // Increment the level of this vertex.
      cost[vertex_id] = level + 1;
      break;
    }
  }
}

// A simple kernel to set the source vertex as visited.
__global__
void bfs_bu_init_kernel(bitmap_t visited, vid_t vertex_id) {
  if (THREAD_GLOBAL_INDEX != 0) { return; }
  bitmap_set_gpu(visited, vertex_id);
}

// This is a GPU only version of the above Bottom-up/Top-down BFS algorithm.
// See bfs_bu_cpu for full details.
__host__
error_t bfs_bu_gpu(graph_t* graph, vid_t source_id, cost_t* cost) {
  // TODO(scott): Make this a heuristic instead of a constant.
  const vid_t SMALL_THRESHOLD = graph->vertex_count/(16*16*16*16);

  // Check for special cases.
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, cost, &finished);
  if (finished) return rc;

  // Create and initialize state on GPU.
  graph_t* graph_d;
  cost_t* cost_d;
  bool* finished_d;
  CHK_SUCCESS(initialize_gpu(graph, source_id, graph->vertex_count,
                             &graph_d, &cost_d, &finished_d), err_free_all);

  // Initialize the visited bitmap on the GPU.
  bitmap_t visited;
  visited = bitmap_init_gpu(graph->vertex_count);
  bfs_bu_init_kernel<<<1, 1>>>(visited, source_id);
  CALL_CU_SAFE(hipGetLastError());

  // Initialize the frontier state on the GPU.
  frontier_state_t state;
  frontier_init_gpu(&state, graph->vertex_count);

  // {} used to limit scope and avoid problems with error handles.
  {
  dim3 blocks;
  dim3 threads_per_block;
  KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);

  // Complete a step for each level, while not finished.
  for (cost_t level = 0; ; level++) {
    // Update the frontier.
    frontier_update_bitmap_gpu(&state, visited, 0);

    if (bitmap_count_gpu(state.current, graph->vertex_count) <
                         SMALL_THRESHOLD) {
      // Choose a top-down step if the frontier count is small.
      bfs_td_kernel<<<blocks, threads_per_block>>>(*graph_d, level,
                                                   cost_d, visited, state);
    } else {
      // Choose a bottom-up step if the frontier count is large.
      bfs_bu_kernel<<<blocks, threads_per_block>>>(*graph_d, level,
                                                   cost_d, visited, state);
    }
    CHK_CU_SUCCESS(hipGetLastError(), err_free_all);

    // We are done if the frontier is empty.
    if (bitmap_count_gpu(state.current, graph->vertex_count) == 0) { break; }
  }}

  // We are done, get the results back and clean up state.
  CHK_SUCCESS(finalize_gpu(graph_d, finished_d, cost_d, cost), err_free_all);
  return SUCCESS;

  // error handlers
  err_free_all:
    totem_free(finished_d, TOTEM_MEM_DEVICE);
    totem_free(cost_d, TOTEM_MEM_DEVICE);
    graph_finalize_device(graph_d);
    return FAILURE;
}
