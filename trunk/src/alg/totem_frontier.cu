#include "hip/hip_runtime.h"
/**
 * This file contains an implementation of the frontier interface which is used
 * by some traversal-based algorithms such as BFS, Graph500 and Betweenness 
 * Centrality
 *
 *  Created on: 2013-08-3
 *  Author: Abdullah Gharaibeh
 */

#include "totem_alg.h"
#include "totem_engine.cuh"

template<int THREADS_PER_BLOCK>
PRIVATE __global__ void
frontier_build_kernel(frontier_state_t state, const cost_t level,
                      const cost_t* __restrict cost) {
  const vid_t v = THREAD_GLOBAL_INDEX;
  if (v >= state.len) return;

  __shared__ vid_t queue_l[THREADS_PER_BLOCK];
  __shared__ vid_t count_l;
  count_l = 0;
  __syncthreads();

  if (cost[v] == level) {
    int index = atomicAdd(&count_l, 1);
    queue_l[index] = v;
  }
  __syncthreads();

  if (THREAD_BLOCK_INDEX >= count_l) return;

  __shared__ int index;
  if (THREAD_BLOCK_INDEX == 0) {
    index = atomicAdd(state.count, count_l);
  }
  __syncthreads();

  state.list[index + THREAD_BLOCK_INDEX] = queue_l[THREAD_BLOCK_INDEX];  
}

#ifdef FEATURE_SM35
PRIVATE __global__ void
frontier_update_boundaries_kernel(frontier_state_t state,
                                  const eid_t* __restrict vertices) {
  const vid_t index = THREAD_GLOBAL_INDEX;
  if (index >= *state.count) return;

  const vid_t* __restrict frontier = state.list;
  vid_t* boundaries = &state.boundaries[1];

  vid_t v = frontier[index];
  vid_t nbr_count = vertices[v + 1] - vertices[v];

  if (nbr_count > 7 && nbr_count < 32) {
    atomicMin(&boundaries[0], index);
  }
  if (nbr_count > 31 && nbr_count < 128) {
    atomicMin(&boundaries[1], index);
  }
  if (nbr_count > 127 && nbr_count < 256) {
    atomicMin(&boundaries[2], index);
  }
  if (nbr_count > 255 && nbr_count < 1024) {
    atomicMin(&boundaries[3], index);
  }
  if (nbr_count > 1023 && nbr_count < (2 * 1024)) {
    atomicMin(&boundaries[4], index);
  }
  if (nbr_count >= (2 * 1024)) {
    atomicMin(&boundaries[5], index);
  }
}

PRIVATE __global__ void
frontier_update_boundaries_launch_kernel(frontier_state_t state,
                                         const eid_t* __restrict vertices) {
  if (THREAD_GLOBAL_INDEX > 0 || (*state.count == 0)) return;
  dim3 blocks;
  kernel_configure(*state.count, blocks);
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  frontier_update_boundaries_kernel
    <<<blocks, DEFAULT_THREADS_PER_BLOCK, 0, stream>>>(state, vertices);
}

PRIVATE __global__ void
frontier_init_boundaries_kernel(frontier_state_t state) {
  const vid_t index = THREAD_GLOBAL_INDEX;
  if (index >= FRONTIER_BOUNDARY_COUNT) return;
  state.boundaries[index + 1] = *state.count;
}

#endif /* FEATURE_SM35  */

void frontier_update_list_gpu(frontier_state_t* state,
                              vid_t level, const cost_t* cost, 
                              const hipStream_t stream) {
  hipMemsetAsync(state->count, 0, sizeof(vid_t), stream);
  dim3 blocks;
  kernel_configure(state->len, blocks);
  frontier_build_kernel<DEFAULT_THREADS_PER_BLOCK>
    <<<blocks, DEFAULT_THREADS_PER_BLOCK, 0, stream>>>(*state, level, cost);
  CALL_CU_SAFE(hipGetLastError());
}

#ifdef FEATURE_SM35
void frontier_update_boundaries_gpu(frontier_state_t* state, 
                                    const graph_t* graph, 
                                    const hipStream_t stream) {
  if (engine_sorted()) {
    // If the vertices are sorted by degree, build the boundaries array
    // to optimize thread scheduling when launching the traversal kernel
    dim3 blocks;
    kernel_configure(FRONTIER_BOUNDARY_COUNT, blocks);
    frontier_init_boundaries_kernel
      <<<blocks, DEFAULT_THREADS_PER_BLOCK, 0, stream>>>(*state);
    CALL_CU_SAFE(hipGetLastError());

    frontier_update_boundaries_launch_kernel
      <<<1, 1, 0, stream>>>(*state, graph->vertices);
    CALL_CU_SAFE(hipGetLastError());
  }
}
#endif /* FEATURE_SM35 */

vid_t frontier_update_bitmap_gpu(frontier_state_t* state, bitmap_t visited,
                                 hipStream_t stream) {
  bitmap_t tmp = state->current;
  state->current = state->visited_last;
  state->visited_last = tmp;
    bitmap_diff_copy_gpu(visited, state->current, state->visited_last, 
                         state->len, stream);
  return 0;
}
vid_t frontier_update_bitmap_cpu(frontier_state_t* state, bitmap_t visited) {
  // Build the frontier bitmap
  bitmap_t tmp = state->current;
  state->current = state->visited_last;
  state->visited_last = tmp;
  bitmap_diff_copy_cpu(visited, state->current, state->visited_last, 
                       state->len);
  return 0;
}

void frontier_reset_gpu(frontier_state_t* state) {
  bitmap_reset_gpu(state->current, state->len);
  bitmap_reset_gpu(state->visited_last, state->len);
  totem_memset(state->count, (vid_t)0, 1, TOTEM_MEM_DEVICE);  
}
void frontier_reset_cpu(frontier_state_t* state) {
  bitmap_reset_cpu(state->current, state->len);
  bitmap_reset_cpu(state->visited_last, state->len);
}

void frontier_init_gpu(frontier_state_t* state, vid_t vertex_count) {
  assert(state);
  state->len = vertex_count;
  state->current = bitmap_init_gpu(vertex_count);
  state->visited_last = bitmap_init_gpu(vertex_count);
  CALL_SAFE(totem_calloc(sizeof(vid_t), TOTEM_MEM_DEVICE, 
                         (void **)&state->count));
  CALL_SAFE(totem_calloc(vertex_count * sizeof(vid_t), TOTEM_MEM_DEVICE,
                         (void **)&state->list));
  state->list_len = vertex_count;
  if (engine_partition_algorithm() == PAR_SORTED_ASC) {
    // LOW-degree vertices were placed on the GPU. Since there is typically
    // many of them, and the GPU has limited memory, we restrict the frontier
    // array size. If the frontier in a specific level was longer, then the 
    // algorithm will not build a frontier array, and will iterate over all
    // the vertices.
    state->list_len = vertex_count * TRV_MAX_FRONTIER_LEN;
  }
#ifdef FEATURE_SM35
  if (engine_sorted()) {
    CALL_SAFE(totem_calloc((FRONTIER_BOUNDARY_COUNT + 1) * sizeof(vid_t), 
                           TOTEM_MEM_DEVICE, (void**)&state->boundaries));
  }
#endif /* FEATURE_SM35 */
}
void frontier_init_cpu(frontier_state_t* state, vid_t vertex_count) {
  state->len = vertex_count;
  state->current = bitmap_init_cpu(vertex_count);
  state->visited_last = bitmap_init_cpu(vertex_count);
}

void frontier_finalize_gpu(frontier_state_t* state) {
  assert(state);
#ifdef FEATURE_SM35
    if (engine_sorted()) totem_free(state->boundaries, TOTEM_MEM_DEVICE);
#endif /* FEATURE_SM35  */
  totem_free(state->list, TOTEM_MEM_DEVICE);
  totem_free(state->count, TOTEM_MEM_DEVICE);
  bitmap_finalize_gpu(state->visited_last);
  bitmap_finalize_gpu(state->current);
}
void frontier_finalize_cpu(frontier_state_t* state) {
  bitmap_finalize_cpu(state->visited_last);
  bitmap_finalize_cpu(state->current);
}
