#include "hip/hip_runtime.h"
/**
 * Contains unit tests for the execution engine.
 *
 *  Created on: 2012-02-09
 *      Author: Abdullah
 */

// totem includes
#include "totem_common_unittest.h"
#include "totem_engine.cuh"

int* degree_g;
int* degree_h;

__global__ void degree_kernel(partition_t par, int pcount) {
  id_t v = THREAD_GLOBAL_INDEX;
  if (v >= par.subgraph.vertex_count) return;
  for (id_t i = par.subgraph.vertices[v]; 
       i < par.subgraph.vertices[v + 1]; i++) {
    int* dst;
    id_t nbr = par.subgraph.edges[i];
    ENGINE_FETCH_DST(par.id, nbr, par.outbox_d, (int*)par.algo_state, 
                     pcount, dst, int);
    atomicAdd(dst, 1);
  }
}

void degree_gpu(partition_t* par) {
  int pcount = engine_partition_count();
  dim3 blocks, threads;
  KERNEL_CONFIGURE(par->subgraph.vertex_count, blocks, threads);
  degree_kernel<<<blocks, threads, 1, par->streams[1]>>>(*par, pcount);
  CALL_CU_SAFE(hipGetLastError());
}

void degree_cpu(partition_t* par) {
  int pcount = engine_partition_count();
  #ifdef _OPENMP
  #pragma omp parallel for
  #endif
  for (id_t v = 0; v < par->subgraph.vertex_count; v++) {
    for (id_t i = par->subgraph.vertices[v]; 
         i < par->subgraph.vertices[v + 1]; i++) {
      int* dst;
      id_t nbr = par->subgraph.edges[i];
      ENGINE_FETCH_DST(par->id, nbr, par->outbox, (int*)par->algo_state, 
                       pcount, dst, int);
      __sync_fetch_and_add(dst, 1);
    }
  }
}

void degree(partition_t* par) {
  if (engine_superstep() == 1) {
    if (par->processor.type == PROCESSOR_GPU) { 
      degree_gpu(par);      
    } else {
      assert(par->processor.type == PROCESSOR_CPU);
      degree_cpu(par);
    }
  } else {
    engine_report_finished(par->id);
  }
}

void degree_scatter(partition_t* par) {
  int* pstate = (int*)par->algo_state;
  engine_scatter_inbox_add(par->id, pstate);
}

void degree_init(partition_t* par) {
  uint64_t vcount = par->subgraph.vertex_count;
  if (par->processor.type == PROCESSOR_GPU) {
    CALL_CU_SAFE(hipMalloc(&(par->algo_state), vcount * sizeof(int)));
    ASSERT_TRUE(par->algo_state);
    CALL_CU_SAFE(hipMemset(par->algo_state, 0, vcount * sizeof(int)));    
  } else {
    ASSERT_TRUE(par->processor.type == PROCESSOR_CPU);    
    par->algo_state = calloc(vcount, sizeof(int));
    ASSERT_TRUE(par->algo_state);
  }
  engine_set_outbox(par->id, 0);
}

void degree_finalize(partition_t* par) {
  int* pstate = (int*)par->algo_state;
  ASSERT_TRUE(pstate);
  if (par->processor.type == PROCESSOR_GPU) {
    CALL_CU_SAFE(hipFree(pstate));
  } else {
    ASSERT_EQ(PROCESSOR_CPU, par->processor.type);
    free(pstate);
  }
  par->algo_state = NULL;
}

void degree_aggr(partition_t* par) {
  int* src = NULL;
  if (par->processor.type == PROCESSOR_GPU) {
    CALL_CU_SAFE(hipMemcpy(degree_h, par->algo_state,
                            par->subgraph.vertex_count * sizeof(int),
                            hipMemcpyDefault));
    src = degree_h;
  } else {
    ASSERT_EQ(PROCESSOR_CPU, par->processor.type);
    src = (int*)par->algo_state;
  }
  // aggregate the results
  #ifdef _OPENMP
  #pragma omp parallel for
  #endif
  for (id_t v = 0; v < par->subgraph.vertex_count; v++) {
    degree_g[par->map[v]] = src[v];
  }
}

class GraphEngineTest : public ::testing::Test {
 protected:
  graph_t* graph_;
  engine_config_t config_;
  virtual void SetUp() {
    // Ensure the minimum CUDA architecture is supported
    CUDA_CHECK_VERSION();
    graph_ = NULL;
    engine_config_t config  = {
      NULL,
      PAR_RANDOM,
      sizeof(int),
      degree,
      degree_scatter,
      degree_init,
      degree_finalize,
      degree_aggr
    };
    config_ = config;
  }

  virtual void TearDown() {
    if (graph_ != NULL) {
      graph_finalize(graph_);
    }
  }
};

TEST_F(GraphEngineTest, ChainGraph) {
  graph_initialize(DATA_FOLDER("chain_1000_nodes.totem"), false, &graph_);
  EXPECT_FALSE(graph_->directed);
  config_.graph = graph_;
  engine_init(&config_);
  degree_g = (int*)calloc(graph_->vertex_count, sizeof(int));
  degree_h = (int*)mem_alloc(engine_largest_gpu_partition() * sizeof(int));
  engine_start();
  mem_free(degree_h);
  for (id_t v = 0; v < graph_->vertex_count; v++) {
    int nbr_count = graph_->vertices[v + 1] - graph_->vertices[v];
    EXPECT_EQ(nbr_count, degree_g[v]);
  }
  free(degree_g);
}

TEST_F(GraphEngineTest, StarGraph) {
  graph_initialize(DATA_FOLDER("star_1000_nodes.totem"), false, &graph_);
  EXPECT_FALSE(graph_->directed);
  config_.graph = graph_;
  engine_init(&config_);
  degree_g = (int*)calloc(graph_->vertex_count, sizeof(int));
  degree_h = (int*)mem_alloc(engine_largest_gpu_partition() * sizeof(int));
  engine_start();
  mem_free(degree_h);
  for (id_t v = 0; v < graph_->vertex_count; v++) {
    int nbr_count = graph_->vertices[v + 1] - graph_->vertices[v];
    EXPECT_EQ(nbr_count, degree_g[v]);
  }
  free(degree_g);
}

TEST_F(GraphEngineTest, CompleteGraph) {
  graph_initialize(DATA_FOLDER("complete_graph_300_nodes.totem"),
                   false, &graph_);
  EXPECT_FALSE(graph_->directed);
  config_.graph = graph_;
  engine_init(&config_);
  degree_g = (int*)calloc(graph_->vertex_count, sizeof(int));
  degree_h = (int*)mem_alloc(engine_largest_gpu_partition() * sizeof(int));
  engine_start();
  mem_free(degree_h);
  for (id_t v = 0; v < graph_->vertex_count; v++) {
    int nbr_count = graph_->vertices[v + 1] - graph_->vertices[v];
    EXPECT_EQ(nbr_count, degree_g[v]);
  }
  free(degree_g);
}
