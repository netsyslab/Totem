#include "hip/hip_runtime.h"
/**
 * Implements a simplified version of the PageRank algorithm based on the 
 * algorithm described by [Malewicz2010]
 * G. Malewicz, M. H. Austern, A. J. C. Bik, J. C. Dehnert, I. Horn, N. Leiser, 
 * and G. Czajkowski. Pregel: a system for large-scale graph processing. In
 * Proceedings of the 28th ACM symposium on Principles of distributed computing,
 * PODC 09, page 6, New York, NY, USA, 2009. ACM.
 *
 * Algorithm description [Malewicz2010]:
 * The graph is initialized so that in round 0, the PageRank of each vertex is 
 * set to 1 / vertex_count. For PAGE_RANK_ROUNDS rounds, each vertex sends along
 * each outgoing edge its tentative PageRank divided by the number of outgoing 
 * edges. The tentative PageRank is calculated as follows: the vertex sums up 
 * the values arriving into sum and sets its own tentative PageRank to 
 * ((1 - DAMPING_FACTOR) / vertex_count + DAMPING_FACTOR * sum).
 *
 *  Created on: 2011-03-06
 *  Author: Abdullah Gharaibeh
 */

#include "totem_comkernel.cuh"
#include "totem_graph.h"
#include "totem_mem.h"

/**
 * Used to define the number of rounds: a static convergance condition 
 * for PageRank
 */
#define PAGE_RANK_ROUNDS 30

/**
 * A probability used in the PageRank algorithm. A probability that models the 
 * behavior of the random surfer when she moves from one page to another 
 * without following the links on the current page.
 */
#define DAMPING_FACTOR 0.85

/**
 * The PageRank kernel. Based on the algorithm described in [Malewicz2010].
 * For each round, each vertex broadcasts along each outgoing edge its tentative
 * PageRank divided by the number of outgoing edges. The tentative PageRank of 
 * vertex is calculated as follows: the vertex sums up the values arriving into 
 * sum and sets its own tentative PageRank to 
 * ((1 - DAMPING_FACTOR) / vertex_count + DAMPING_FACTOR * sum).
 * Broadcasting messages over outgoing edges is done as follows: the value is 
 * placed in the outbox buffer. In the next round the inbox and outbox are 
 * swapped, and the message will be accessed in the next round via the
 * inbox buffer. This operation simulates a broadcast because all the neighbors 
 * of vertex v will access the same location (i.e., inbox[v]) to get the messege
 * (i.e., tentative_PageRank_of_v/neighbor_count). In the last round, outbox 
 * will contain the PageRank of each vertex.
 * @param[in] graph the graph to apply page rank on
 * @param[in] inbox messeges broadcasted to vertices
 * @param[in] outbox messeges to be broadcasted in the next round
 */
__global__ 
void page_rank_kernel(graph_t graph, float* inbox, float* outbox, 
                      bool last_round) {

  // get the thread's linear index
  uint32_t my_index = THREAD_GLOBAL_INDEX;
  
  // get direct access to graph members
  uint32_t  vertex_count = graph.vertex_count;
  uint32_t* vertices     = graph.vertices;
  uint32_t* edges        = graph.edges;

  if (my_index >= vertex_count) {
    return;
  }
                                                                               
  // get the neighbors
  uint32_t  neighbors_count = vertices[my_index + 1] - vertices[my_index];
  uint32_t* neighbors       = &(edges[vertices[my_index]]);

  // calculate the sum of all neighbors' rank
  double sum = 0;
  for (uint32_t i = 0; i < neighbors_count; i++) {
    uint32_t neighbor = neighbors[i];
    sum += inbox[neighbor];
  }
  
  // calculate my rank
  float my_rank = 
    ((1 - DAMPING_FACTOR) / (double)vertex_count) + (DAMPING_FACTOR * sum);
  outbox[my_index] = last_round? my_rank: my_rank / neighbors_count;
}


error_t page_rank_gpu(graph_t* graph, float** rank) {
  
  /* had to define them at the beginning to avoid a compilation problem with 
     goto-label error handling mechanism */
  dim3 blocks; 
  dim3 threads_per_block;

  // will be passed to the kernel
  graph_t graph_d;  
  memcpy(&graph_d, graph, sizeof(graph_t));

  uint32_t vertex_count = graph->vertex_count;
  uint32_t edge_count   = graph->edge_count;

  // allocate vertices and edges device buffers and move them to the device
  CHECK_ERR(hipMalloc((void**)&graph_d.vertices, (vertex_count + 1) *
                       sizeof(uint32_t)) == hipSuccess, err);
  CHECK_ERR(hipMalloc((void**)&graph_d.edges, edge_count * 
                       sizeof(uint32_t)) == hipSuccess, err_free_vertices);

  CHECK_ERR(hipMemcpy(graph_d.vertices, graph->vertices, 
                       (vertex_count + 1) * sizeof(uint32_t), 
                       hipMemcpyHostToDevice) == hipSuccess, 
            err_free_edges);
  CHECK_ERR(hipMemcpy(graph_d.edges, graph->edges, 
                       edge_count * sizeof(uint32_t),
                       hipMemcpyHostToDevice) == hipSuccess, 
            err_free_edges);

  // allocate inbox and outbox device buffers
  float *inbox_d;
  CHECK_ERR(hipMalloc((void**)&inbox_d, vertex_count * 
                       sizeof(float)) == hipSuccess, err_free_edges);
  float *outbox_d;
  CHECK_ERR(hipMalloc((void**)&outbox_d, vertex_count * 
                       sizeof(float)) == hipSuccess, err_free_inbox);

  /* set the number of blocks, TODO(abdullah) handle the case when 
     vertex_count is larger than number of threads. */
  assert(vertex_count <= MAX_THREAD_COUNT);
  KERNEL_CONFIGURE(vertex_count, blocks, threads_per_block);
  
  // initialize the rank of each vertex 
  float initial_value;
  initial_value = 1/(float)vertex_count;
  memset_device<<<blocks, threads_per_block>>>
    (outbox_d, initial_value, vertex_count);
  CHECK_ERR(hipGetLastError() == hipSuccess, err_free_outbox);

  uint32_t round;
  for (round = 0; round < PAGE_RANK_ROUNDS; round++) {
    // swap the inbox and outbox pointers (simulates passing messages)
    float* tmp = inbox_d;
    inbox_d = outbox_d;
    outbox_d = tmp;

    // call the kernel
    bool last_round = (round == (PAGE_RANK_ROUNDS - 1));
    page_rank_kernel<<<blocks, threads_per_block>>>
      (graph_d, inbox_d, outbox_d, last_round);
    CHECK_ERR(hipGetLastError() == hipSuccess, err_free_outbox);
    
    hipDeviceSynchronize();
    CHECK_ERR(hipGetLastError() == hipSuccess, err_free_outbox);
  }

  // copy back the final result from the outbox
  float* my_rank;
  my_rank = (float*)mem_alloc(vertex_count * sizeof(float));
  CHECK_ERR(hipMemcpy(my_rank, outbox_d, vertex_count * sizeof(float),
                       hipMemcpyDeviceToHost) == hipSuccess, err_free_all);

  // we are done! set the output and clean up
  *rank = my_rank;  
  hipFree(outbox_d);
  hipFree(inbox_d);
  hipFree(graph_d.edges);
  hipFree(graph_d.vertices);
  return SUCCESS;

  // error handlers
 err_free_all:
 err_free_outbox:
  hipFree(outbox_d);
 err_free_inbox:
  hipFree(inbox_d);
 err_free_edges:
  hipFree(graph_d.edges);
 err_free_vertices:
  hipFree(graph_d.vertices);
 err:
  printf("%d\n", hipGetLastError());
  return FAILURE;
}

error_t page_rank_cpu(graph_t* graph, float** rank) {

  // get direct access to graph members
  uint32_t  vertex_count = graph->vertex_count;
  uint32_t* vertices     = graph->vertices;
  uint32_t* edges        = graph->edges;

  // allocate buffers
  float* inbox = (float*)mem_alloc(vertex_count * sizeof(float));
  float* outbox = (float*)mem_alloc(vertex_count * sizeof(float));
  
  // initialize the rank of each vertex
  float initial_value;
  initial_value = 1/(float)vertex_count;
  for (uint32_t vid = 0; vid < vertex_count; vid++) {
    outbox[vid] = initial_value;
  }

  for (uint32_t round = 0; round < PAGE_RANK_ROUNDS; round++) {

    // swap the inbox and outbox pointers (simulates passing messages!)
    float* tmp = inbox;
    inbox = outbox;
    outbox = tmp;

    // iterate over all vertices to calculate the ranks for this round
    for(uint32_t vid = 0; vid < vertex_count; vid++) {
      // get the neighbors
      uint32_t   neighbors_count  = vertices[vid + 1] - vertices[vid];
      uint32_t*  neighbors        = &(edges[vertices[vid]]);

      // calculate the sum of all neighbors' rank
      double sum = 0;
      for (uint32_t i = 0; i < neighbors_count; i++) {
        uint32_t neighbor  = neighbors[i];
        sum               += inbox[neighbor];
      }

      // calculate my rank
      float my_rank = 
        ((1 - DAMPING_FACTOR) / vertex_count) + (DAMPING_FACTOR * sum);
      outbox[vid] = 
        (round == (PAGE_RANK_ROUNDS - 1)) ? my_rank : my_rank / neighbors_count;
    }   
  }

  // we are done! set the output and clean up.
  *rank = outbox;
  mem_free(inbox);
  return SUCCESS;
}
