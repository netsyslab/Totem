#include "hip/hip_runtime.h"
/**
 * // TODO(elizeu): Add license.
 *
 * Implements Dijkstra's single source shortest path algorithm. This
 * implementation is based on the algorithms presented by [Harish07]
 * P. Harish and P. Narayanan, "Accelerating large graph algorithms on the GPU
 * using CUDA," in High Performance Computing - HiPC 2007, LNCS v. 4873, ch. 21,
 * doi: http://dx.doi.org/10.1007/978-3-540-77220-0_21
 *
 *  Created on: 2011-03-04
 *      Author: Elizeu Santos-Neto (elizeus@ece.ubc.ca)
 */

// system includes
#include <hip/hip_runtime.h>

// totem includes
#include "totem_comdef.h"
#include "totem_comkernel.cuh"
#include "totem_graph.h"
#include "totem_mem.h"

/**
 * Tests whether any of the array elements is set to true.
 * @param[in] array a boolean array
 * @param[in] size the number of element in the array
 * @param[out] result indicates whether the kernel found a true element.
 */
__global__
void has_true_kernel(bool* array, uint32_t size, bool* result) {
  const id_t vertex_id = THREAD_GLOBAL_INDEX;
  *result = false;
  if (vertex_id >= size) {    
    return;
  }
  if (array[vertex_id]) {
    *result = true;
  }
}

/**
 * Computes the new distances for each neighbor in the graph.
 * @param[in] graph the input graph used to compute the distances
 * @param[in] to_update an array to indicate which nodes will update distances
 * @param[in] distances an array that contains the current state of distances
 * @param[in] mutex a mutex variable used to implement an atomicMin.
 * @param[out] new_distances an array with distances updated in this round
 */
__global__
void dijkstra_kernel(graph_t graph, bool* to_update, weight_t* distances,
                     weight_t* new_distances, uint32_t* mutex) {

  // get direct access to graph members
  id_t  vertex_count = graph.vertex_count;
  id_t* vertices     = graph.vertices;
  id_t* edges        = graph.edges;
  weight_t* weights  = graph.weights;

  // TODO(abdullah): May be there is an opportunity for optimization here.
  //                 Threads (vertices) that do not have the to_update set will
  //                 exit at this point, and they will stay idle until their
  //                 mates are done. I was wondering how we can keep them busy.
  //                 For BFS, the Stanford paper uses a work queue and they get
  //                 marginal improvement. It is not clear if SSSP will have the
  //                 same behavior.
  const id_t vertex_id = THREAD_GLOBAL_INDEX;
  if ((vertex_id >= vertex_count) || !to_update[vertex_id]) {
    return;
  }
  to_update[vertex_id] = false;

  id_t* neighbors = &(edges[vertices[vertex_id]]);
  weight_t* local_weights = &(weights[vertices[vertex_id]]);
  uint64_t neighbor_count = vertices[vertex_id + 1] - vertices[vertex_id];
  weight_t distance_to_vertex = distances[vertex_id];

  for (id_t i = 0; i < neighbor_count; i++) {
    id_t neighbor_id = neighbors[i];
    weight_t current_distance = distance_to_vertex + local_weights[i];
    // TODO(elizeu): This mutex is inefficient, as it serializes all threads.
    //               One approach to solve this is to have one mutex per vertex
    //               that indicates whether the position in the new_distance
    //               array regarding that vertex is locked or open.
    while(!atomicCAS(mutex, 1, 0));
    weight_t* new_distance = &(new_distances[neighbor_id]);
    if (current_distance < *new_distance) {
      *new_distance = current_distance;
    }
    atomicCAS(mutex, 0, 1);
  } // for
}

/**
 * Make the new distances permanent if the new distances are smaller than
 * current distances.
 * @param[in] graph the input graph used to compute the distances
 * @param[in] to_update an array to indicate which nodes will update distances
 * @param[in] distances an array that contains the current state of distances
 * @param[in] mutex a mutex variable used to implement an atomicMin.
 * @param[out] new_distances an array with distances updated in this round
 */
__global__
void dijkstra_final_kernel(graph_t graph, bool* to_update, weight_t* distances,
                           weight_t* new_distances) {
  const uint32_t vertex_id = THREAD_GLOBAL_INDEX;
  if (vertex_id >= graph.vertex_count) {
    return;
  }
  if (new_distances[vertex_id] < distances[vertex_id]) {
    distances[vertex_id] = new_distances[vertex_id];
    to_update[vertex_id] = true;
  }
  new_distances[vertex_id] = distances[vertex_id];
}

error_t dijkstra_gpu(graph_t* graph, id_t source_id,
                     weight_t** shortest_distances) {
  // TODO(elizeu): Move input validations to a common separate function.
  // Validate input parameters
  if ((graph == NULL) || !graph->weighted
      || (source_id >= graph->vertex_count)) {
    *shortest_distances = NULL;
    return FAILURE;
  } else if(graph->vertex_count == 1) {
    *shortest_distances = (weight_t*)mem_alloc(sizeof(weight_t));
    (*shortest_distances)[0] = 0;
    return SUCCESS;
  }

  // Check whether the graph has vertices, but an empty edge set.
  if ((graph->vertex_count > 0) && (graph->edge_count == 0)) {
    *shortest_distances =
      (weight_t*)mem_alloc(graph->vertex_count * sizeof(weight_t));
    for (id_t node_id = 0; node_id < graph->vertex_count; node_id++) {
      (*shortest_distances)[node_id] = WEIGHT_MAX;
    }
    (*shortest_distances)[source_id] =  (weight_t)0.0;
    return SUCCESS;
  }

  // Kernel configuration parameters.
  dim3 block_count;
  dim3 threads_per_block;

  graph_t* graph_d;
  // Allocate and transfer the vertex array to the device.
  CHK_SUCCESS(graph_initialize_device(graph, &graph_d), err);

  // The distance array from the source vertex to every nother in the graph.
  weight_t* distances_d;
  CHK_CU_SUCCESS(hipMalloc((void **)&distances_d,
                       graph_d->vertex_count * sizeof(weight_t)),
            err_free_graph);

  // An array that contains the newly computed array of distances.
  weight_t* new_distances_d;
  CHK_CU_SUCCESS(hipMalloc((void **)&new_distances_d,
                       graph_d->vertex_count * sizeof(weight_t)),
            err_free_distances);

  // An entry in this array indicate whether the corresponding vertex should
  // try to compute new distances.
  bool* changed_d;
  CHK_CU_SUCCESS(hipMalloc((void **)&changed_d, graph_d->vertex_count
                       * sizeof(bool)), err_free_new_distances);

  // Compute the number of blocks.
  KERNEL_CONFIGURE(graph_d->vertex_count, block_count, threads_per_block);

  // Initialize the mutex used in the kernel to avoid the race condition.
  // TODO(elizeu): We may want to move this feature into a separate file if
  //               atomic-*() functions that receive floating point arguments
  //               become common.
  uint32_t* mutex_d;
  CHK_CU_SUCCESS(hipMalloc((void **)&mutex_d, sizeof(uint32_t)),
                 err_free_new_distances);

  // Initialize the mutex.
  CHK_CU_SUCCESS(hipMemset(mutex_d, 1, sizeof(uint32_t)), err_free_mutex);

  // Set all distances to infinite.
  memset_device<<<block_count, threads_per_block>>>(distances_d, WEIGHT_MAX,
                                                    graph_d->vertex_count);
  memset_device<<<block_count, threads_per_block>>>(new_distances_d, WEIGHT_MAX,
                                                    graph_d->vertex_count);

  // Set the distance to the source to zero.
  CHK_CU_SUCCESS(hipMemset(&(distances_d[source_id]), (weight_t)0,
                 sizeof(weight_t)), err_free_mutex);

  // Activate the source vertex to compute distances.
  CHK_CU_SUCCESS(hipMemset(&(changed_d[source_id]), true, sizeof(bool)),
                 err_free_mutex);

  // Compute the distances update
  bool has_true;
  has_true = true;
  bool* has_true_d;
  CHK_CU_SUCCESS(hipMalloc((void **)&has_true_d, sizeof(bool)),
            err_free_has_true);
  CHK_CU_SUCCESS(hipMemset(has_true_d, false, sizeof(bool)),
            err_free_has_true);

  while (has_true) {
    dijkstra_kernel<<<block_count, threads_per_block>>>
      (*graph_d, changed_d, distances_d, new_distances_d, mutex_d);
    dijkstra_final_kernel<<<block_count, threads_per_block>>>
      (*graph_d, changed_d, distances_d, new_distances_d);
    has_true_kernel<<<block_count, threads_per_block>>>
      (changed_d, graph_d->vertex_count, has_true_d);
    CHK_CU_SUCCESS(hipMemcpy(&has_true, has_true_d, sizeof(bool),
                              hipMemcpyDeviceToHost), err_free_has_true);
  }

  // Copy the pointer to the output parameter
  weight_t* local_shortest_distances;
  local_shortest_distances =
    (weight_t*)mem_alloc(graph_d->vertex_count * sizeof(weight_t));
  CHK_CU_SUCCESS(hipMemcpy(local_shortest_distances, distances_d,
                       graph_d->vertex_count * sizeof(weight_t),
                       hipMemcpyDeviceToHost),
                       err_free_has_true);
  *shortest_distances = local_shortest_distances;

  // Release the allocated memory
  graph_finalize_device(graph_d);
  hipFree(distances_d);
  hipFree(changed_d);
  hipFree(mutex_d);
  hipFree(new_distances_d);

  return SUCCESS;

  // error handlers
  err_free_has_true:
    hipFree(has_true_d);
  err_free_mutex:
    hipFree(mutex_d);
  err_free_new_distances:
    hipFree(new_distances_d);
  err_free_distances:
    hipFree(distances_d);
  err_free_graph:
    graph_finalize_device(graph_d);
  err:
    return FAILURE;
}

__host__
error_t dijkstra_cpu(graph_t* graph, id_t source_id,
                     weight_t** shortest_distances) {
  // Validate input parameters
  if ((graph == NULL) || !graph->weighted
      || (source_id >= graph->vertex_count)) {
    *shortest_distances = NULL;
    return FAILURE;
  } else if (graph->vertex_count == 1) {
    *shortest_distances = (weight_t*)mem_alloc(sizeof(weight_t));
    (*shortest_distances)[0] = 0;
    return SUCCESS;
  }

  // Initialize the shortest_distances to infinite  
  *shortest_distances =
    (weight_t*)mem_alloc(graph->vertex_count * sizeof(weight_t));

  // An entry in this array indicate whether the corresponding vertex should
  // try to update the current distances.
  bool* to_update = (bool *)mem_alloc(graph->vertex_count * sizeof(bool));

  for (id_t vertex_id = 0; vertex_id < graph->vertex_count; vertex_id++) {
    (*shortest_distances)[vertex_id] = WEIGHT_MAX;
    to_update[vertex_id] = false;
  }
  (*shortest_distances)[source_id] =  (weight_t)0.0;
  to_update[source_id] = true;

  // Check whether the graph has vertices, but an empty edge set.
  if ((graph->vertex_count > 0) && (graph->edge_count == 0)) {
    mem_free(to_update);
    return SUCCESS;
  }

  // get direct access to graph members
  id_t  vertex_count = graph->vertex_count;
  id_t* vertices     = graph->vertices;
  id_t* edges        = graph->edges;
  weight_t* weights  = graph->weights;

  // Initialize the mutex.
  // TODO(elizeu): This line generates a "unreferenced variable" warning,
  //               even though the variable is referenced implicitely by
  //               omp below. We need to find a way to disable/enable it.
  int mutex = 0;
  mutex = mutex + 0;

  bool changed = true;
  while (changed) {
    changed = false;

    #ifdef _OPENMP
    #pragma omp parallel for
    #endif // _OPENMP
    for (id_t vertex_id = 0; vertex_id < vertex_count; vertex_id++) {
      if (!to_update[vertex_id]) {
        continue;
      }
      to_update[vertex_id] = false;

      id_t* neighbors = &edges[vertices[vertex_id]];
      weight_t* local_weights = &weights[vertices[vertex_id]];
      uint64_t neighbor_count = vertices[vertex_id + 1] - vertices[vertex_id];

      for (id_t i = 0; i < neighbor_count; i++) {
        id_t neighbor_id = neighbors[i];
        // TODO(elizeu): This global lock may be inefficient. One approach to
        //               solve this is to have one lock per vertex.
        #ifdef _OPENMP
        #pragma omp critical (mutex)
        {
        #endif // _OPENMP
        weight_t current_distance =
            (*shortest_distances)[vertex_id] + local_weights[i];
        if ((*shortest_distances)[neighbor_id] > current_distance) {
          (*shortest_distances)[neighbor_id] = current_distance;
          to_update[neighbor_id] = true;
          changed = true;
        }
        #ifdef _OPENMP
        } // critical
        #endif // _OPENMP
      } // for
    } // for
  } // while
  return SUCCESS;
}

