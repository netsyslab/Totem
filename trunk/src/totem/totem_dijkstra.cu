#include "hip/hip_runtime.h"
/**
 * // TODO(elizeu): Add license.
 *
 * Implements Dijkstra's single source shortest path algorithm. This
 * implementation is based on the algorithms presented by [Harish07]
 * P. Harish and P. Narayanan, "Accelerating large graph algorithms on the GPU
 * using CUDA," in High Performance Computing - HiPC 2007, LNCS v. 4873, ch. 21,
 * doi: http://dx.doi.org/10.1007/978-3-540-77220-0_21
 *
 *  Created on: 2011-03-04
 *      Author: Elizeu Santos-Neto (elizeus@ece.ubc.ca)
 */

// totem includes
#include "totem_bitmap.cuh"
#include "totem_comdef.h"
#include "totem_comkernel.cuh"
#include "totem_graph.h"
#include "totem_mem.h"

/**
   This structure is used by the virtual warp-based implementation. It stores a
   batch of work. It is typically allocated on shared memory and is processed by
   a single virtual warp.
 */
 // TODO(elizeu): Consider moving the edges weights of the neighbors to this
 // structure. It might be tricky to keep alignment or even to fit the data
 // into shared memory.
typedef struct {
  id_t vertices[VWARP_BATCH_SIZE + 1];
  weight_t distances[VWARP_BATCH_SIZE];
  bool to_update[VWARP_BATCH_SIZE];
  // the following ensures 64-bit alignment, it assumes that the cost and
  // vertices arrays are of 32-bit elements.
  // TODO(abdullah) a portable way to do this (what if id_t is 64-bit?)
  int pad;
} vwarp_mem_t;


/**
 * Checks for input parameters and special cases. This is invoked at the
 * beginning of public interfaces (GPU and CPU)
*/
PRIVATE
error_t check_special_cases(const graph_t* graph, id_t source_id,
                            weight_t **shortest_distances, bool* finished) {
  *finished = true;
  if ((graph == NULL) || !graph->weighted
      || (source_id >= graph->vertex_count)) {
    *shortest_distances = NULL;
    return FAILURE;
  } else if (graph->vertex_count == 1) {
    *shortest_distances = (weight_t*)mem_alloc(sizeof(weight_t));
    (*shortest_distances)[0] = 0;
    return SUCCESS;
  }

  // Check whether the graph has vertices, but an empty edge set.
  if ((graph->vertex_count > 0) && (graph->edge_count == 0)) {
    *shortest_distances =
      (weight_t*)mem_alloc(graph->vertex_count * sizeof(weight_t));
    for (id_t node_id = 0; node_id < graph->vertex_count; node_id++) {
      (*shortest_distances)[node_id] = WEIGHT_MAX;
    }
    (*shortest_distances)[source_id] =  (weight_t)0.0;
    return SUCCESS;
  }

  *finished = false;
  return SUCCESS;
}


/**
 * A common initialization function for GPU implementations of Dijkstra's
 * algorithm. It allocates memory in the device and initalizes state on the GPU.
*/
PRIVATE
error_t initialize_gpu(const graph_t* graph, id_t source_id,
                       uint64_t distance_length, graph_t** graph_d,
                       bool** changed_d, bool** has_true_d,
                       weight_t** distances_d, weight_t** new_distances_d) {

  // Kernel configuration parameters.
  dim3 block_count;
  dim3 threads_per_block;

  // Allocate and transfer the vertex array to the device.
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);

  // The distance array from the source vertex to every node in the graph.
  CHK_CU_SUCCESS(hipMalloc((void**)distances_d,
                            distance_length * sizeof(weight_t)),
                 err_free_graph);

  // An array that contains the newly computed array of distances.
  CHK_CU_SUCCESS(hipMalloc((void**)new_distances_d,
                            distance_length * sizeof(weight_t)),
                 err_free_distances);

  // An entry in this array indicate whether the corresponding vertex should
  // try to compute new distances.
  CHK_CU_SUCCESS(hipMalloc((void **)changed_d, distance_length * sizeof(bool)),
                 err_free_new_distances);

  // Compute the number of blocks.
  KERNEL_CONFIGURE(distance_length, block_count, threads_per_block);

  // Set all distances to infinite.
  memset_device<<<block_count, threads_per_block>>>
      (*distances_d, WEIGHT_MAX, distance_length);
  memset_device<<<block_count, threads_per_block>>>
      (*new_distances_d, WEIGHT_MAX, distance_length);

  // Set the distance to the source to zero.
  CHK_CU_SUCCESS(hipMemset(&((*distances_d)[source_id]), (weight_t)0,
                 sizeof(weight_t)), err_free_new_distances);

  // Activate the source vertex to compute distances.
  CHK_CU_SUCCESS(hipMemset(*changed_d, false, distance_length * sizeof(bool)),
                 err_free_new_distances);
  CHK_CU_SUCCESS(hipMemset(&((*changed_d)[source_id]), true, sizeof(bool)),
                 err_free_new_distances);

  // Initialize the flags that indicate whether the distances were updated
  CHK_CU_SUCCESS(hipMalloc((void **)has_true_d, sizeof(bool)),
            err_free_new_distances);
  CHK_CU_SUCCESS(hipMemset(*has_true_d, false, sizeof(bool)),
            err_free_all);

  return SUCCESS;

  // error handlers
  err_free_all:
    hipFree(*has_true_d);
  err_free_new_distances:
    hipFree(*new_distances_d);
  err_free_distances:
    hipFree(*distances_d);
  err_free_graph:
    graph_finalize_device(*graph_d);
  err:
    return FAILURE;
}


/**
 * A common finalize function for GPU implementations. It allocates the host
 * output buffer, moves the final results from GPU to the host buffers and
 * frees up some resources.
*/
PRIVATE
error_t finalize_gpu(graph_t* graph_d, weight_t* distances_d, bool* changed_d,
                     weight_t* new_distances_d, weight_t** shortest_distances) {

  // Copy the pointer to the output parameter
  *shortest_distances =
      (weight_t*)mem_alloc(graph_d->vertex_count * sizeof(weight_t));
  CHK_CU_SUCCESS(hipMemcpy(*shortest_distances, distances_d,
                            graph_d->vertex_count * sizeof(weight_t),
                            hipMemcpyDeviceToHost), err);

  // Release the allocated memory
  hipFree(distances_d);
  hipFree(changed_d);
  hipFree(new_distances_d);
  graph_finalize_device(graph_d);
  return SUCCESS;

 err:
  mem_free(shortest_distances);
  *shortest_distances = NULL;
  return FAILURE;
}

/**
 * Computes the new distances for each neighbor in the graph.
 * @param[in] graph the input graph used to compute the distances
 * @param[in] to_update an array to indicate which nodes will update distances
 * @param[in] distances an array that contains the current state of distances
 * @param[out] new_distances an array with distances updated in this round
 */
__global__
void dijkstra_kernel(graph_t graph, bool* to_update, weight_t* distances,
                     weight_t* new_distances) {

  // get direct access to graph members
  id_t  vertex_count = graph.vertex_count;
  id_t* vertices     = graph.vertices;
  id_t* edges        = graph.edges;
  weight_t* weights  = graph.weights;

  const id_t vertex_id = THREAD_GLOBAL_INDEX;
  if ((vertex_id >= vertex_count) || !to_update[vertex_id]) {
    return;
  }

  id_t* neighbors = &(edges[vertices[vertex_id]]);
  weight_t* local_weights = &(weights[vertices[vertex_id]]);
  id_t neighbor_count = vertices[vertex_id + 1] - vertices[vertex_id];
  weight_t distance_to_vertex = distances[vertex_id];

  for (id_t i = 0; i < neighbor_count; i++) {
    id_t neighbor_id = neighbors[i];
    weight_t current_distance = distance_to_vertex + local_weights[i];
    atomicMin(&(new_distances[neighbor_id]), current_distance);
  } // for
}


/**
 * The neighbors processing function. This function computes the distance from
 * the source node to the each of the neighbors to the current vertex. The
 * assumption is that the threads of a warp invoke this function to process the
 * warp's batch of work. In each iteration of the for loop, each thread
 * processes a neighbor. For example, thread 0 in the warp processes neighbors
 * at indices 0, VWARP_WARP_SIZE, (2 * VWARP_WARP_SIZE) etc. in the edges array,
 * while thread 1 in the warp processes neighbors 1, (1 + VWARP_WARP_SIZE),
 * (1 + 2 * VWARP_WARP_SIZE) and so on.
*/
inline __device__
void vwarp_process_neighbors(int warp_offset, id_t neighbor_count,
                             id_t* neighbors, weight_t* weights,
                             weight_t distance_to_vertex,
                             weight_t* new_distances) {
  for(int i = warp_offset; i < neighbor_count; i += VWARP_WARP_SIZE) {
    id_t neighbor_id = neighbors[i];
    weight_t current_distance = distance_to_vertex + weights[i];
    atomicMin(&(new_distances[neighbor_id]), current_distance);
  } // for
}


/**
 * An implementation of the Dijkstra kernel that implements the virtual warp
 * technique.
 */
__global__
void vwarp_dijkstra_kernel(graph_t graph, bool* to_update, weight_t* distances,
                           weight_t* new_distances, uint32_t thread_count) {

  if (THREAD_GLOBAL_INDEX >= thread_count) return;

  int warp_offset = THREAD_GLOBAL_INDEX % VWARP_WARP_SIZE;
  int warp_id     = THREAD_GLOBAL_INDEX / VWARP_WARP_SIZE;

  __shared__ vwarp_mem_t shared_memory[(MAX_THREADS_PER_BLOCK
                                        / VWARP_WARP_SIZE)];
  vwarp_mem_t* my_space = shared_memory + (THREAD_GRID_INDEX / VWARP_WARP_SIZE);

  // copy my work to local space
  int v_ = warp_id * VWARP_BATCH_SIZE;
  vwarp_memcpy(my_space->distances, &distances[v_], VWARP_BATCH_SIZE,
               warp_offset);
  vwarp_memcpy(my_space->vertices, &(graph.vertices[v_]), VWARP_BATCH_SIZE + 1,
               warp_offset);
  vwarp_memcpy(my_space->to_update, &(to_update[v_]), VWARP_BATCH_SIZE,
               warp_offset);

  // iterate over my work
  for(uint32_t v = 0; v < VWARP_BATCH_SIZE; v++) {
    weight_t distance_to_vertex = my_space->distances[v];
    if (my_space->to_update[v]) {
      id_t* neighbors = &(graph.edges[my_space->vertices[v]]);
      weight_t* local_weights = &(graph.weights[my_space->vertices[v]]);
      id_t neighbor_count = my_space->vertices[v + 1] - my_space->vertices[v];
      vwarp_process_neighbors(warp_offset, neighbor_count, neighbors,
                              local_weights, distance_to_vertex, new_distances);
    }
  }
}


/**
 * Make the new distances permanent if the new distances are smaller than
 * current distances.
 * @param[in] graph the input graph used to compute the distances
 * @param[in] to_update an array to indicate which nodes will update distances
 * @param[in] distances an array that contains the current state of distances
 * @param[in] mutex a mutex variable used to implement an atomicMin.
 * @param[out] new_distances an array with distances updated in this round
 */
__global__
void dijkstra_final_kernel(graph_t graph, bool* to_update, weight_t* distances,
                           weight_t* new_distances, bool* has_true) {
  const uint32_t vertex_id = THREAD_GLOBAL_INDEX;
  if (vertex_id >= graph.vertex_count) {
    return;
  }
  if (new_distances[vertex_id] < distances[vertex_id]) {
    distances[vertex_id] = new_distances[vertex_id];
    to_update[vertex_id] = true;
    *has_true = true;
  }
  new_distances[vertex_id] = distances[vertex_id];
}

error_t dijkstra_gpu(const graph_t* graph, id_t source_id,
                     weight_t** shortest_distances) {
  // Check for special cases
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, shortest_distances,
                                   &finished);
  if (finished) return rc;

  // Allocate and initialize GPU state
  bool *changed_d;
  bool *has_true_d;
  graph_t* graph_d;
  weight_t* distances_d;
  weight_t* new_distances_d;
  CHK_SUCCESS(initialize_gpu(graph, source_id, graph->vertex_count, &graph_d,
                             &changed_d, &has_true_d, &distances_d,
                             &new_distances_d), err);

  {
  dim3 block_count, threads_per_block;
  KERNEL_CONFIGURE(graph->vertex_count, block_count, threads_per_block);
  bool has_true = true;
  while (has_true) {
    dijkstra_kernel<<<block_count, threads_per_block>>>
      (*graph_d, changed_d, distances_d, new_distances_d);
    CHK_CU_SUCCESS(hipMemset(changed_d, false, graph->vertex_count *
                              sizeof(bool)), err_free_all);
    CHK_CU_SUCCESS(hipMemset(has_true_d, false, sizeof(bool)), err_free_all);
    dijkstra_final_kernel<<<block_count, threads_per_block>>>
      (*graph_d, changed_d, distances_d, new_distances_d, has_true_d);
    CHK_CU_SUCCESS(hipMemcpy(&has_true, has_true_d, sizeof(bool),
                              hipMemcpyDeviceToHost), err_free_all);
  }
  }

  // Copy the output shortest distances from the device mem to the host
  // Finalize GPU
  CHK_SUCCESS(finalize_gpu(graph_d, distances_d, changed_d, new_distances_d,
                      shortest_distances), err_free_all);
  return SUCCESS;

  // error handlers
  err_free_all:
   hipFree(changed_d);
   hipFree(has_true_d);
   hipFree(distances_d);
   hipFree(new_distances_d);
   graph_finalize_device(graph_d);
  err:
    return FAILURE;
}

error_t dijkstra_vwarp_gpu(const graph_t* graph, id_t source_id,
                           weight_t** shortest_distances) {

  // Check for special cases
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, shortest_distances,
                                   &finished);
  if (finished) return rc;

  // Allocate and initialize GPU state
  bool *changed_d;
  bool *has_true_d;
  graph_t* graph_d;
  weight_t* distances_d;
  weight_t* new_distances_d;
  uint64_t distance_length;
  distance_length = VWARP_BATCH_SIZE * VWARP_BATCH_COUNT(graph->vertex_count);
  CHK_SUCCESS(initialize_gpu(graph, source_id, distance_length, &graph_d,
                             &changed_d, &has_true_d, &distances_d,
                             &new_distances_d), err);

  {
  bool has_true = true;
  dim3 block_count, threads_per_block;
  int thread_count = VWARP_WARP_SIZE * VWARP_BATCH_COUNT(graph->vertex_count);
  KERNEL_CONFIGURE(thread_count, block_count, threads_per_block);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(vwarp_dijkstra_kernel), hipFuncCachePreferShared);
  dim3 block_count_final, threads_per_block_final;
  KERNEL_CONFIGURE(graph->vertex_count, block_count_final,
                   threads_per_block_final);
  while (has_true) {
    vwarp_dijkstra_kernel<<<block_count, threads_per_block>>>
      (*graph_d, changed_d, distances_d, new_distances_d, thread_count);
    CHK_CU_SUCCESS(hipMemset(changed_d, false, distance_length * sizeof(bool)),
                   err_free_all);
    CHK_CU_SUCCESS(hipMemset(has_true_d, false, sizeof(bool)), err_free_all);
    dijkstra_final_kernel<<<block_count_final, threads_per_block_final>>>
      (*graph_d, changed_d, distances_d, new_distances_d, has_true_d);
    CHK_CU_SUCCESS(hipMemcpy(&has_true, has_true_d, sizeof(bool),
                              hipMemcpyDeviceToHost), err_free_all);
  }
  }

  // Finalize GPU
  CHK_SUCCESS(finalize_gpu(graph_d, distances_d, changed_d, new_distances_d,
                      shortest_distances), err_free_all);
  return SUCCESS;

  // error handlers
  err_free_all:
   hipFree(changed_d);
   hipFree(has_true_d);
   hipFree(distances_d);
   hipFree(new_distances_d);
   graph_finalize_device(graph_d);
  err:
    return FAILURE;
}

__host__ error_t dijkstra_cpu(const graph_t* graph, id_t source_id,
                              weight_t** shortest_distances) {
  // Check for special cases
  bool finished = false;
  error_t rc = check_special_cases(graph, source_id, shortest_distances,
                                   &finished);
  if (finished) return rc;

  // Initialize the shortest_distances to infinite
  *shortest_distances = 
    (weight_t*)mem_alloc(graph->vertex_count * sizeof(weight_t));
  OMP(omp parallel for)
  for (id_t vertex_id = 0; vertex_id < graph->vertex_count; vertex_id++) {
    (*shortest_distances)[vertex_id] = WEIGHT_MAX;
  }

  // An entry in this bitmap indicates whether the corresponding vertex is
  // active and that it should try to update the distances of its neighbors
  bitmap_t active = bitmap_init_cpu(graph->vertex_count);

  // Initialize the distance of the source vertex
  (*shortest_distances)[source_id] =  (weight_t)0.0;
  bitmap_set_cpu(active, source_id);

  finished = false;
  while (!finished) {
    finished = true;
    OMP(omp parallel for reduction(& : finished))
    for (id_t vertex_id = 0; vertex_id < graph->vertex_count; vertex_id++) {
      if (!bitmap_is_set(active, vertex_id)) {
        continue;
      }
      bitmap_unset_cpu(active, vertex_id);
      
      for (id_t i = graph->vertices[vertex_id]; 
           i < graph->vertices[vertex_id + 1]; i++) {
        const id_t neighbor_id = graph->edges[i];
        weight_t new_distance = (*shortest_distances)[vertex_id] + 
          graph->weights[i];
        weight_t old_distance =
          __sync_fetch_and_min_float(&((*shortest_distances)[neighbor_id]),
                                       new_distance);
        if (new_distance < old_distance) {
          bitmap_set_cpu(active, neighbor_id);
          finished = false;
        }
      } // for
    } // for
  } // while
  bitmap_finalize_cpu(active);
  return SUCCESS;
}

