/**
 * Implements the graph partitioning interface defined in totem_partition.h
 *
 *  Created on: 2011-12-29
 *  Author: Abdullah Gharaibeh
 */

// totem includes
#include "totem_comkernel.cuh"
#include "totem_mem.h"
#include "totem_partition.h"
#include "totem_util.h"

// TODO (scott): Non global variables
vid_t* map_g[MAX_PARTITION_COUNT] = {NULL};
vid_t* id_in_partition_g = NULL;

/**
 *  Will randomize the placement of vertices across multiple GPUs.
 *
 *  @param graph       Graph to work with.
 *  @param partitions  The location of the partitions with ids attached.
 *  @param gpu_count   The amount of GPUs to randomize across
 *                     (Should be larger than 1)
 */
PRIVATE void randomize_across_gpus(graph_t* graph, vid_t* partitions,
                                   int gpu_count){
  if (gpu_count < 2) return; // Nothing to do

  uint32_t seed = GLOBAL_SEED;

  // Randomize the placement of vertices across GPUs, can be done in parallel.
  OMP(omp parallel for schedule(static))
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    if (partitions[v] < gpu_count) {
      partitions[v] = rand_r(&seed) % gpu_count;
    }
  }
}

/**
 *  Will map vertices in order according to degree.
 *
 *  @param graph            Graph to work with.
 *  @param partition_count  The number of partitions in use.
 *  @param partitions       The vertex to partition map.
 *  @param partition_random True if called by a random partitioning strategy.
 *  @param asc              True if ascending order, false if descending.
 *                          (Determined by partitioning algorithm -i)
 *  @param vd               Map of vertices to degrees. May be NULL if
 *                          partition_random is true.
 *
 *  @return Error status.
 */
PRIVATE error_t map_vertices_by_degree(graph_t* graph, int partition_count,
                                       vid_t* partitions,
                                       bool partition_random,
                                       bool asc, vdegree_t* vd){
  vid_t partition_vertex_count[MAX_PARTITION_COUNT];
  totem_memset(partition_vertex_count, (vid_t)0, MAX_PARTITION_COUNT,
               TOTEM_MEM_HOST);
  // Calculate the amount of vertices in each partition.
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    partition_vertex_count[partitions[v]]++;
  }
  // Allocate memory for the main vertex map.
  CALL_SAFE(totem_calloc(graph->vertex_count * sizeof(vid_t), TOTEM_MEM_HOST,
                         (void**)&id_in_partition_g));
  // Allocate memory for each partition's map.
  for (int pid = 0; pid < partition_count; pid++) {
    CALL_SAFE(totem_calloc(partition_vertex_count[pid] * sizeof(vid_t),
                           TOTEM_MEM_HOST, (void**)&map_g[pid]));
    partition_vertex_count[pid] = 0;
  }

  for (vid_t i = 0; i < graph->vertex_count; i++) {
    // Invert the index if we are using descending order.
    vid_t index = (asc) ? i : graph->vertex_count - i - 1;
    // Assign the vertex id based off of the index and strategy.
    vid_t v = (partition_random) ? index : vd[index].id;
    int pid = partitions[v];
    vid_t* local_map = map_g[pid];
    vid_t local_id = partition_vertex_count[pid]++;
    local_map[local_id] = v;
    id_in_partition_g[v] = SET_PARTITION_ID(local_id, pid);
  }

  return SUCCESS;
}

// Overloaded for random partitioning. (Ascending, no vertex-degree array.)
PRIVATE error_t map_vertices_by_degree(graph_t* graph, int partition_count,
                                       vid_t* partitions){
  map_vertices_by_degree(graph, partition_count, partitions,
                         true, /* Partitioning by random flag set.     */
                         true, /* No effect, ascending is fine.        */
                         NULL  /* No vertex degree array - is ignored. */);
}

// Overloaded for ordered calls.
PRIVATE error_t map_vertices_by_degree(graph_t* graph, int partition_count,
                                       vid_t* partitions, bool asc,
                                       vdegree_t* vd){
  map_vertices_by_degree(graph, partition_count, partitions,
                         false, /* Not partitioning randomly. */
                         asc, vd);
}

error_t partition_modularity(graph_t* graph, partition_set_t* partition_set,
                             double* modularity) {
  assert(graph && partition_set);
  if ((graph->edge_count == 0) || (partition_set->partition_count <= 1)) {
    *modularity = 0;
    return SUCCESS;
  }
  // The final modularity value
  double Q = 0.0;
  for (int p = 0; p < partition_set->partition_count; p++) {
    eid_t local_edges = 0;
    eid_t remote_edges = 0;
    partition_t* partition = &partition_set->partitions[p];
    graph_t* subgraph = &partition->subgraph;
    for (vid_t v = 0; v < subgraph->vertex_count; v++) {
      for (eid_t e = subgraph->vertices[v];
           e < subgraph->vertices[v + 1]; e++) {
        if (p == GET_PARTITION_ID(subgraph->edges[e])) {
          local_edges++;
        } else {
          remote_edges++;
        }
      }
    }
    double local = local_edges / (double)graph->edge_count;
    double remote = (remote_edges * remote_edges)
                    / (double)(graph->edge_count * graph->edge_count);
    Q += local - remote;
  }
  *modularity = Q;
  return SUCCESS;
}

PRIVATE error_t partition_check(graph_t* graph, int partition_count,
                                double* partition_fraction,
                                vid_t** partition_labels) {
  *partition_labels = NULL;
  if (graph == NULL || (partition_count <= 0) || (graph->vertex_count == 0)) {
    return FAILURE;
  }
  if (graph == NULL) {
    // TODO(elizeu): Use Lauro's beautiful logging library.
    printf("ERROR: Graph object is NULL, cannot proceed with partitioning.\n");
    return FAILURE;
  }
  // The requested number of partitions should be positive
  if ((partition_count <= 0) || (graph->vertex_count == 0)) {
    printf("ERROR: Invalid number of partitions or empty graph: %d (|V|),"
           " %d (partitions).\n", graph->vertex_count, partition_count);
    return FAILURE;
  }

  if (partition_fraction != NULL) {
    // Ensure the partition fractions are >= 0.0 and add up to 1.0
    double sum = 0.0;
    for (int par_id = 0; par_id < partition_count; par_id++) {
      sum += partition_fraction[par_id];
      if (partition_fraction[par_id] < 0.0) {
        return FAILURE;
      }
    }
    // The following trick is to avoid getting stuck in precision errors
    sum = (int)(sum * 100.0);
    if (sum > 101 || sum < 99) {
      return FAILURE;
    }
  }
  return SUCCESS;
}

PRIVATE error_t partition_random(graph_t* graph, int partition_count,
                                 vid_t** partition_labels,
                                 totem_attr_t* attr) {
  // Allocate the partition vector
  vid_t* partitions = (vid_t*)malloc((graph->vertex_count) * sizeof(vid_t));

  // Initialize the random number generator
  // TODO(abdullah): pass the seed as an argument to control the randomness
  //                 of the algorithm if the experiments show variability in
  //                 performance or the characteristics of the partitions.
  srand(GLOBAL_SEED);

  for (vid_t vertex_id = 0; vertex_id < graph->vertex_count; vertex_id++) {
    // Assign each vertex to a random partition within the range
    // (0, PARTITION_COUNT - 1)
    partitions[vertex_id] = rand() % partition_count;
  }
  *partition_labels = partitions;

  if (attr->sorted){
    map_vertices_by_degree(graph, partition_count, partitions);
  }

  return SUCCESS;
}

error_t partition_random(graph_t* graph, int partition_count,
                         double* partition_fraction, vid_t** partition_labels,
                         totem_attr_t* attr) {
  // Check pre-conditions
  if (partition_check(graph, partition_count, partition_fraction,
                      partition_labels) == FAILURE) {
    return FAILURE;
  }

  // Check if the client is asking for equal divide among partitions
  if (partition_fraction == NULL) {
    return partition_random(graph, partition_count, partition_labels, attr);
  }

  // Allocate the partition vector
  vid_t* partitions = (vid_t*)malloc(graph->vertex_count * sizeof(vid_t));
  assert(partitions != NULL);

  // Initialize the random number generator
  srand(GLOBAL_SEED);

  // Allocate all the partition ids to the id vector
  vid_t v = 0;
  for (int pid = 0; pid < partition_count; pid++) {
    vid_t end = (pid == partition_count - 1) ? graph->vertex_count :
      v + ((double)graph->vertex_count * partition_fraction[pid]);
    for (; v < end; v++) {
      partitions[v] = pid;
    }
  }

  /* Randomize the vector to achieve a random distribution. This is using the
   * Fisher-Yates "Random permutation" algorithm */
  for (vid_t i = graph->vertex_count - 1; i > 0; i--) {
    vid_t j = rand() % (i + 1);
    vid_t temp = partitions[i];
    partitions[i] = partitions[j];
    partitions[j] = temp;
  }

  *partition_labels = partitions;

  if (attr->sorted) {
    map_vertices_by_degree(graph, partition_count, partitions);
  }

  return SUCCESS;
}

PRIVATE bool compare_degrees_asc(const vdegree_t &a, const vdegree_t &b) {
  return (a.degree < b.degree);
}

PRIVATE bool compare_degrees_dsc(const vdegree_t &a, const vdegree_t &b) {
  return (a.degree > b.degree);
}

PRIVATE
error_t partition_by_sorted_degree(graph_t* graph, int partition_count,
                                   bool asc, double* partition_fraction,
                                   vid_t** partition_labels,
                                   totem_attr_t* attr) {
  // Check pre-conditions
  if (partition_check(graph, partition_count, partition_fraction,
                      partition_labels) == FAILURE) {
    return FAILURE;
  }

  bool even_fractions = false;
  if (partition_fraction == NULL) {
    even_fractions = true;
    partition_fraction = (double*)calloc(partition_count, sizeof(double));
    for (int pid = 0; pid < partition_count; pid++) {
      partition_fraction[pid] = 1.0/(double)partition_count;
    }
  }

  // Prepare the degree-sorted list of vertices
  vdegree_t* vd = (vdegree_t*)calloc(graph->vertex_count, sizeof(vdegree_t));
  assert(vd);

  // Calculate the degree for each vertex (# in destination, less source)
  OMP(omp parallel for)
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    vd[v].id = v;
    vd[v].degree = graph->vertices[v + 1] - graph->vertices[v];
  }

  // Sort the vertices by degree, in ascending or descending order,
  // (based off of the -i partitioning flag; high or low)
  if (asc) {
    tbb::parallel_sort(vd, vd + graph->vertex_count, compare_degrees_asc);
  } else {
    tbb::parallel_sort(vd, vd + graph->vertex_count, compare_degrees_dsc);
  }

  // Allocate the labels array
  *partition_labels = (vid_t*)calloc(graph->vertex_count, sizeof(vid_t));
  assert(*partition_labels);

  // At the beginning, assume that all vertices belong to the last partition
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    (*partition_labels)[v] = partition_count - 1;
  }

  // Assign vertices to partitions.
  double total_elements = (double)graph->edge_count;
  vid_t index = 0;
  for (int pid = 0; pid < partition_count - 1; pid++) {
    double assigned = 0;
    while ((assigned / total_elements < partition_fraction[pid]) &&
           (index < graph->vertex_count)) {
      assigned += vd[index].degree;
      (*partition_labels)[vd[index].id] = pid;
      index++;
    }
  }

  if (attr->gpu_par_randomized){
    randomize_across_gpus(graph, (*partition_labels), attr->gpu_count);
  }
  if (attr->sorted){
    map_vertices_by_degree(graph, partition_count, (*partition_labels),
                            asc, vd);
  }

  // Clean up
  if (even_fractions) {
    free(partition_fraction);
  }
  free(vd);
  return SUCCESS;
}

error_t partition_by_asc_sorted_degree(graph_t* graph, int partition_count,
                                       double* partition_fraction,
                                       vid_t** partition_labels,
                                       totem_attr_t* attr) {
  return partition_by_sorted_degree(graph, partition_count, true,
                                    partition_fraction, partition_labels,
                                    attr);
}

error_t partition_by_dsc_sorted_degree(graph_t* graph, int partition_count,
                                       double* partition_fraction,
                                       vid_t** partition_labels,
                                       totem_attr_t* attr) {
  return partition_by_sorted_degree(graph, partition_count, false,
                                    partition_fraction, partition_labels,
                                    attr);
}

PRIVATE error_t init_allocate_struct_space(graph_t* graph, int pcount,
                                           size_t push_msg_size,
                                           size_t pull_msg_size,
                                           partition_set_t** pset,
                                           totem_attr_t* attr) {
  *pset = (partition_set_t*)calloc(1, sizeof(partition_set_t));
  assert(*pset);
  (*pset)->partitions = (partition_t*)calloc(pcount, sizeof(partition_t));
  assert((*pset)->partitions);

  // TODO (scott): can we simplify this to not need the attribute?
  // Assign the location for mapping id's to partitions.
  if (attr->sorted) {
    (*pset)->id_in_partition = id_in_partition_g;
  } else {
    (*pset)->id_in_partition =
      (vid_t*)calloc(graph->vertex_count, sizeof(vid_t));
  }
  assert((*pset)->id_in_partition);

  (*pset)->graph = graph;
  (*pset)->partition_count = pcount;
  (*pset)->push_msg_size = push_msg_size;
  (*pset)->pull_msg_size = pull_msg_size;
  (*pset)->weighted = graph->weighted;
  return SUCCESS;
}

PRIVATE
void init_compute_partitions_sizes(partition_set_t* pset, vid_t* plabels) {
  graph_t* graph = pset->graph;
  OMP(omp parallel for)
  for (vid_t vid = 0; vid < graph->vertex_count; vid++) {
    vid_t nbr_count = graph->vertices[vid + 1] - graph->vertices[vid];
    int pid = plabels[vid];
    partition_t* partition = &(pset->partitions[pid]);
    __sync_fetch_and_add(&(partition->subgraph.vertex_count), 1);
    __sync_fetch_and_add(&(partition->subgraph.edge_count), nbr_count);
  }
}

PRIVATE void init_allocate_partitions_space(partition_set_t* pset,
                                            processor_t* pproc,
                                            totem_attr_t* attr) {
  for (int pid = 0; pid < pset->partition_count; pid++) {
    partition_t* partition = &pset->partitions[pid];
    partition->id = pid;
    partition->processor = pproc[pid];
    graph_t* subgraph = &partition->subgraph;
    if (subgraph->vertex_count > 0) {
      subgraph->vertices =
          (eid_t*)malloc(sizeof(eid_t) * (subgraph->vertex_count + 1));
      assert(subgraph->vertices);

      //TODO (scott): can we simplify this to not need the attribute?
      // Assign the partition map.
      if (attr->sorted){
        partition->map = map_g[pid];
      } else {
        partition->map = (vid_t*)calloc(subgraph->vertex_count, sizeof(vid_t));
      }

      if (subgraph->edge_count > 0) {
        subgraph->edges = (vid_t*)malloc(sizeof(vid_t) * subgraph->edge_count);
        assert(subgraph->edges);
        if (pset->graph->weighted) {
          // Update edge-weight state of the subgraph
          subgraph->weighted = pset->graph->weighted;
          subgraph->weights = (weight_t*)malloc(sizeof(weight_t) *
                                                subgraph->edge_count);
          assert(subgraph->weights);
        }
      }
    }
  }
}

PRIVATE void init_build_map(partition_set_t* pset, vid_t* plabels) {
  // Reset the vertex and edge count, will be set again while building the map
  for (int pid = 0; pid < pset->partition_count; pid++) {
    pset->partitions[pid].subgraph.vertex_count = 0;
  }
  for (vid_t vid = 0; vid < pset->graph->vertex_count; vid++) {
    vid_t pid = plabels[vid];
    graph_t* subgraph = &pset->partitions[pid].subgraph;
    // Forward map.
    pset->id_in_partition[vid] = SET_PARTITION_ID(subgraph->vertex_count, pid);
    // Reverse map.
    pset->partitions[pid].map[subgraph->vertex_count] = vid;
    subgraph->vertex_count++;
  }
}

PRIVATE void init_build_partitions_vertices_array(partition_set_t* pset,
                                                  vid_t* plabels) {
  // Reset the vertex count, will be set again next.
  for (int pid = 0; pid < pset->partition_count; pid++) {
    pset->partitions[pid].subgraph.vertex_count = 0;
  }

  // Identify the local id of each vertex and set the number of neighbours.
  // This is done in preparation to computing the vertex array next.
  graph_t* graph = pset->graph;
  for (vid_t vid = 0; vid < graph->vertex_count; vid++) {
    partition_t* partition = &pset->partitions[plabels[vid]];
    graph_t* subgraph = &partition->subgraph;
    vid_t local_id = subgraph->vertex_count++;
    vid_t global_id = partition->map[local_id];
    subgraph->vertices[local_id + 1] = graph->vertices[global_id + 1] -
        graph->vertices[global_id];
  }

  // Compute the vertex array of each partition (prefix sum).
  OMP(omp parallel for schedule(static))
  for (int pid = 0; pid < pset->partition_count; pid++) {
    partition_t* partition = &pset->partitions[pid];
    graph_t* subgraph = &partition->subgraph;
    if (subgraph->vertex_count == 0) continue;
    subgraph->vertices[0] = 0;
    for (vid_t vid = 1; vid <= subgraph->vertex_count; vid++) {
      subgraph->vertices[vid] += subgraph->vertices[vid - 1];
    }
  }
}

PRIVATE void init_build_partitions_edges_array(partition_set_t* pset) {
  graph_t* graph = pset->graph;
  OMP(omp parallel for schedule(guided))
  for (vid_t vid = 0; vid < graph->vertex_count; vid++) {
    vid_t local_id = GET_VERTEX_ID(pset->id_in_partition[vid]);
    partition_t* partition =
        &pset->partitions[GET_PARTITION_ID(pset->id_in_partition[vid])];
    graph_t* subgraph = &partition->subgraph;
    eid_t edge_index = subgraph->vertices[local_id];
    for (eid_t i = graph->vertices[vid]; i < graph->vertices[vid + 1]; i++) {
      subgraph->edges[edge_index] = pset->id_in_partition[graph->edges[i]];
      if (graph->weighted) {
        subgraph->weights[edge_index] = graph->weights[i];
      }
      edge_index++;
    }
  }
}

PRIVATE void init_build_partitions(partition_set_t* pset, vid_t* plabels,
                                   totem_attr_t* attr) {
  // Build the map. The map maps the old vertex id to its new id in the
  // partition. This is necessary because the vertices assigned to a
  // partition will be renamed so that the ids are contiguous from 0 to
  // partition->subgraph.vertex_count - 1.

  // TODO (scott): can we simplify this to not need the attribute?
  // The init function is unnecessary if the vertex degree is mapped sorted,
  // it is taken care of before the partitions are built.
  if(!attr->sorted) {
    init_build_map(pset, plabels);
  }

  // Build the vertices array of each partition.
  init_build_partitions_vertices_array(pset, plabels);

  // Build the edge and weight arrays of each partition.
  init_build_partitions_edges_array(pset);
}

PRIVATE void init_sort_nbrs(partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  for (uint32_t pid = 0; pid < pcount; pid++) {
    graph_t* subgraph = &pset->partitions[pid].subgraph;
    graph_sort_nbrs(subgraph);
  }
}

PRIVATE void init_build_partitions_gpu(partition_set_t* pset,
                                       gpu_graph_mem_t gpu_graph_mem) {
  uint32_t pcount = pset->partition_count;
  for (uint32_t pid = 0; pid < pcount; pid++) {
    partition_t* partition = &pset->partitions[pid];
    if (partition->processor.type != PROCESSOR_GPU) continue;
    CALL_CU_SAFE(hipSetDevice(partition->processor.id));
    CALL_CU_SAFE(hipStreamCreate(&partition->streams[0]));
    CALL_CU_SAFE(hipStreamCreate(&partition->streams[1]));
    CALL_CU_SAFE(hipEventCreate(&partition->event_start));
    CALL_CU_SAFE(hipEventCreate(&partition->event_end));
    graph_t* subgraph_h = (graph_t*)malloc(sizeof(graph_t));
    assert(subgraph_h);
    memcpy(subgraph_h, &partition->subgraph, sizeof(graph_t));
    graph_t* subgraph_d = NULL;
    CALL_SAFE(graph_initialize_device(subgraph_h, &subgraph_d,
                                      gpu_graph_mem));
    graph_finalize(subgraph_h);
    memcpy(&partition->subgraph, subgraph_d, sizeof(graph_t));
    free(subgraph_d);
  }
}

error_t partition_set_initialize(graph_t* graph, vid_t* plabels,
                                 processor_t* pproc, int pcount,
                                 gpu_graph_mem_t gpu_graph_mem,
                                 size_t push_msg_size, size_t pull_msg_size,
                                 partition_set_t** pset, totem_attr_t* attr) {
  assert(graph && plabels && pproc);
  if (pcount > MAX_PARTITION_COUNT) return FAILURE;

  // Setup space and initialize the partition set data structure
  CHK_SUCCESS(init_allocate_struct_space(graph, pcount, push_msg_size,
                                         pull_msg_size, pset, attr), err);

  // Get the partition sizes
  init_compute_partitions_sizes(*pset, plabels);

  // Allocate partitions space
  init_allocate_partitions_space(*pset, pproc, attr);

  // Build the state of each partition
  init_build_partitions(*pset, plabels, attr);

  // Sort nbrs of each each vertex to improve access locality
  init_sort_nbrs(*pset);

  // Initialize grooves' inbox and outbox state
  grooves_initialize(*pset);

  // Build the state on the GPU(s) for GPU residing partitions
  init_build_partitions_gpu(*pset, gpu_graph_mem);

  return SUCCESS;
 err:
  return FAILURE;
}

error_t partition_set_finalize(partition_set_t* pset) {
  assert(pset);
  assert(pset->partitions);
  for (int pid = 0; pid < pset->partition_count; pid++) {
    partition_t* partition = &pset->partitions[pid];
    graph_t* subgraph = &partition->subgraph;
    if (partition->processor.type == PROCESSOR_GPU) {
      CALL_CU_SAFE(hipSetDevice(partition->processor.id));
      CALL_CU_SAFE(hipStreamDestroy(partition->streams[0]));
      CALL_CU_SAFE(hipStreamDestroy(partition->streams[1]));
      CALL_CU_SAFE(hipEventDestroy(partition->event_start));
      CALL_CU_SAFE(hipEventDestroy(partition->event_end));
      // TODO(abdullah): use graph_finalize instead of manually
      // freeing the buffers
      if (subgraph->gpu_graph_mem == GPU_GRAPH_MEM_MAPPED ||
          subgraph->gpu_graph_mem == GPU_GRAPH_MEM_MAPPED_VERTICES) {
        totem_free(subgraph->mapped_vertices, TOTEM_MEM_HOST_MAPPED);
      } else {
        totem_free(subgraph->vertices, TOTEM_MEM_DEVICE);
      }

      if (subgraph->edge_count) {
        if (subgraph->gpu_graph_mem == GPU_GRAPH_MEM_MAPPED ||
            subgraph->gpu_graph_mem == GPU_GRAPH_MEM_MAPPED_EDGES) {
          totem_free(subgraph->mapped_edges, TOTEM_MEM_HOST_MAPPED);
        } else if ((subgraph->gpu_graph_mem == GPU_GRAPH_MEM_DEVICE) ||
                   ((subgraph->gpu_graph_mem ==
                     GPU_GRAPH_MEM_PARTITIONED_EDGES) &&
                    (subgraph->vertex_ext < subgraph->vertex_count))) {
          totem_free(subgraph->edges, TOTEM_MEM_DEVICE);
        } else if (subgraph->gpu_graph_mem == GPU_GRAPH_MEM_PARTITIONED_EDGES) {
          totem_free(subgraph->edges, TOTEM_MEM_DEVICE);
          totem_free(subgraph->mapped_edges, TOTEM_MEM_HOST_MAPPED);
        }
      }

      if (subgraph->weighted && subgraph->weights)
        CALL_CU_SAFE(hipFree(subgraph->weights));
    } else {
      assert(partition->processor.type == PROCESSOR_CPU);
      if (subgraph->vertices) free(subgraph->vertices);
      if (subgraph->edges) free(subgraph->edges);
      if (pset->weighted && subgraph->weights) {
        free(subgraph->weights);
      }
    }
    if (subgraph->vertices) free(partition->map);
  }
  grooves_finalize(pset);
  free(pset->partitions);
  free(pset->id_in_partition);
  free(pset);
  return SUCCESS;
}

void partition_set_update_msg_size(partition_set_t* pset,
                                   grooves_direction_t dir, size_t msg_size) {
  assert(pset);
  if (dir == GROOVES_PUSH) pset->push_msg_size = msg_size;
  if (dir == GROOVES_PULL) pset->pull_msg_size = msg_size;
}
