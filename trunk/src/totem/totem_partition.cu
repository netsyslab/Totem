/**
 * Implements the graph partitionining interface defined in totem_partition.h
 *
 *  Created on: 2011-12-29
 *  Author: Abdullah Gharaibeh
 */

// totem includes
#include "totem_comkernel.cuh"
#include "totem_mem.h"
#include "totem_partition.h"
#include "totem_util.h"

/*
 * A vertex-degree data type used in the partitioning algorithms that depend 
 * on sorting the vertices by edge degree.
 */
typedef struct vdegree_s {
  vid_t id;     // vertex id
  vid_t degree; // vertex degree
} vdegree_t;


error_t partition_modularity(graph_t* graph, partition_set_t* partition_set,
                             double* modularity) {
  assert(graph && partition_set);
  if ((graph->edge_count == 0) || (partition_set->partition_count <= 1)) {
    *modularity = 0;
    return SUCCESS;
  }
  // The final modularity value
  double Q = 0.0;
  for (int p = 0; p < partition_set->partition_count; p++) {
    eid_t local_edges = 0;
    eid_t remote_edges = 0;
    partition_t* partition = &partition_set->partitions[p];
    graph_t* subgraph = &partition->subgraph;
    for (vid_t v = 0; v < subgraph->vertex_count; v++) {
      for (eid_t e = subgraph->vertices[v];
           e < subgraph->vertices[v + 1]; e++) {
        if (p == GET_PARTITION_ID(subgraph->edges[e])) {
          local_edges++;
        } else {
          remote_edges++;
        }
      }
    }
    double local = local_edges / (double)graph->edge_count;
    double remote = (remote_edges * remote_edges)
                    / (double)(graph->edge_count * graph->edge_count);
    Q += local - remote;
  }
  *modularity = Q;
  return SUCCESS;
}

PRIVATE error_t partition_check(graph_t* graph, int partition_count, 
                                double* partition_fraction, 
                                vid_t** partition_labels) {
  *partition_labels = NULL;
  if (graph == NULL || (partition_count <= 0) || (graph->vertex_count == 0)) {
    return FAILURE;
  }
  if (graph == NULL) {
    // TODO(elizeu): Use Lauro's beautiful logging library.
    printf("ERROR: Graph object is NULL, cannot proceed with partitioning.\n");
    return FAILURE;
  }
  // The requested number of partitions should be positive
  if ((partition_count <= 0) || (graph->vertex_count == 0)) {
    printf("ERROR: Invalid number of partitions or empty graph: %d (|V|),"
           " %d (partitions).\n", graph->vertex_count, partition_count);
    return FAILURE;
  }

  if (partition_fraction != NULL) {
    // Ensure the partition fractions are >= 0.0 and add up to 1.0
    double sum = 0.0;
    for (int par_id = 0; par_id < partition_count; par_id++) {
      sum += partition_fraction[par_id];
      if (partition_fraction[par_id] < 0.0) {
        return FAILURE;
      }
    }
    // The following trick is to avoid getting stuck in precision errors
    sum = (int)(sum * 100.0);
    if (sum > 101 || sum < 99) {
      return FAILURE;
    }
  }
  return SUCCESS;
}

PRIVATE error_t partition_random(graph_t* graph, int partition_count,
                                 vid_t** partition_labels) {
  // Allocate the partition vector
  vid_t* partitions = (vid_t*)malloc((graph->vertex_count) * sizeof(vid_t));

  // Initialize the random number generator
  // TODO(abdullah): pass the seed as an argument to control the randomness
  //                 of the algorithm if the experiments show variability in 
  //                 performance or the characteristics of the partitions.
  srand(time(NULL));

  for (vid_t vertex_id = 0; vertex_id < graph->vertex_count; vertex_id++) {
    // Assign each vertex to a random partition within the range
    // (0, PARTITION_COUNT - 1)
    partitions[vertex_id] = rand() % partition_count;
  }
  *partition_labels = partitions;
  return SUCCESS;
}

error_t partition_random(graph_t* graph, int partition_count,
                         double* partition_fraction, vid_t** partition_labels) {
  // Check pre-conditions
  if (partition_check(graph, partition_count, partition_fraction,
                      partition_labels) == FAILURE) {
    return FAILURE;
  }

  // Check if the client is asking for equal divide among partitions
  if (partition_fraction == NULL) {
    return partition_random(graph, partition_count, partition_labels);
  }

  // Allocate the partition vector
  vid_t* partitions = (vid_t*)malloc(graph->vertex_count * sizeof(vid_t));
  assert(partitions != NULL);

  // Initialize the random number generator
  srand(time(NULL));

  // Allocate all the partition ids to the id vector
  vid_t v = 0;
  for (int pid = 0; pid < partition_count; pid++) {
    vid_t end = (pid == partition_count - 1) ? graph->vertex_count :
      v + ((double)graph->vertex_count * partition_fraction[pid]);
    for (; v < end; v++) {
      partitions[v] = pid;
    }
  }

  /* Randomize the vector to achieve a random distribution. This is using the
   * Fisher-Yates "Random permutation" algorithm */
  for (vid_t i = graph->vertex_count - 1; i > 0; i--) {
    vid_t j = rand() % (i + 1);
    vid_t temp = partitions[i];
    partitions[i] = partitions[j];
    partitions[j] = temp;
  }

  *partition_labels = partitions;
  return SUCCESS;
}

PRIVATE int compare_degrees_asc(const void *a, const void *b) {
  vdegree_t* d1 = (vdegree_t*)a;
  vdegree_t* d2 = (vdegree_t*)b;
  if (d1->degree < d2->degree) return -1;
  if (d1->degree == d2->degree) return 0;
  return 1;
}

PRIVATE int compare_degrees_dsc(const void *a, const void *b) {
  vdegree_t* d1 = (vdegree_t*)a;
  vdegree_t* d2 = (vdegree_t*)b;
  if (d1->degree > d2->degree) return -1;
  if (d1->degree == d2->degree) return 0;
  return 1;
}

PRIVATE 
error_t partition_by_sorted_degree(graph_t* graph, int partition_count, 
                                   bool asc, double* partition_fraction, 
                                   vid_t** partition_labels) {
  // Check pre-conditions
  if (partition_check(graph, partition_count, partition_fraction,
                      partition_labels) == FAILURE) {
    return FAILURE;
  }

  bool even_fractions = false;
  if (partition_fraction == NULL) {
    even_fractions = true;
    partition_fraction = (double*)calloc(partition_count, sizeof(double));
    for (int pid = 0; pid < partition_count; pid++) {
      partition_fraction[pid] = 1.0/(double)partition_count;
    }
  }

  // Prepare the degree-sorted list of vertices 
  vdegree_t* vd = (vdegree_t*)calloc(graph->vertex_count, sizeof(vdegree_t));
  assert(vd);
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    vd[v].id = v;
    vd[v].degree = graph->vertices[v + 1] - graph->vertices[v];
  }
  if (asc) {
    qsort(vd, graph->vertex_count, sizeof(vdegree_t), compare_degrees_asc);
  } else {
    qsort(vd, graph->vertex_count, sizeof(vdegree_t), compare_degrees_dsc);
  }

  // Allocate the labels array
  *partition_labels = (vid_t*)calloc(graph->vertex_count, sizeof(vid_t));
  assert(*partition_labels);

  // Assign vertices to partitions. Considering the sum of edges and vertices as
  // the normalizing factor allows to create a partition with space complexity 
  // proportional to the requested fraction. This is important when creating 
  // partitions with few edges but many vertices (e.g., a partition dominated by
  // low-degree vertices). 
  double total_elements = (double)graph->vertex_count + 
    (double)graph->edge_count;
  vid_t index = 0;
  for (int pid = 0; pid < partition_count - 1; pid++) {
    double assigned = 0;
    while ((assigned / total_elements < partition_fraction[pid]) &&
           (index < graph->vertex_count)) {
      assigned += vd[index].degree + 1;
      (*partition_labels)[vd[index].id] = pid;
      index++;
    }
  }
  // Assign the rest to the last partition
  for (; index < graph->vertex_count; index++) {
    (*partition_labels)[vd[index].id] = partition_count - 1;
  }

  // Clean up
  if (even_fractions) {
    free(partition_fraction);
  }
  free(vd);
  return SUCCESS;
}

error_t partition_by_asc_sorted_degree(graph_t* graph, int partition_count,
                                       double* partition_fraction, 
                                       vid_t** partition_labels) {
  return partition_by_sorted_degree(graph, partition_count, true, 
                                    partition_fraction, partition_labels);
}

error_t partition_by_dsc_sorted_degree(graph_t* graph, int partition_count,
                                       double* partition_fraction, 
                                       vid_t** partition_labels) {
  return partition_by_sorted_degree(graph, partition_count, false, 
                                    partition_fraction, partition_labels);
}

PRIVATE error_t init_allocate_struct_space(graph_t* graph, int pcount,
                                           size_t push_msg_size,
                                           size_t pull_msg_size,
                                           partition_set_t** pset) {
  *pset = (partition_set_t*)calloc(1, sizeof(partition_set_t));
  assert(*pset);
  (*pset)->partitions = (partition_t*)calloc(pcount, sizeof(partition_t));
  assert((*pset)->partitions);
  (*pset)->id_in_partition = (vid_t*)calloc(graph->vertex_count, sizeof(vid_t));
  assert((*pset)->id_in_partition);
  (*pset)->graph = graph;
  (*pset)->partition_count = pcount;
  (*pset)->push_msg_size = push_msg_size;
  (*pset)->pull_msg_size = pull_msg_size;
  (*pset)->weighted = graph->weighted;
  return SUCCESS;
}

PRIVATE
void init_compute_partitions_sizes(partition_set_t* pset, vid_t* plabels) {
  graph_t* graph = pset->graph;
  OMP(omp parallel for)
  for (vid_t vid = 0; vid < graph->vertex_count; vid++) {
    vid_t nbr_count = graph->vertices[vid + 1] - graph->vertices[vid];
    int pid = plabels[vid];
    partition_t* partition = &(pset->partitions[pid]);
    __sync_fetch_and_add(&(partition->subgraph.vertex_count), 1);
    __sync_fetch_and_add(&(partition->subgraph.edge_count), nbr_count);
  }
}

PRIVATE void init_allocate_partitions_space(partition_set_t* pset) {
  for (int pid = 0; pid < pset->partition_count; pid++) {
    partition_t* partition = &pset->partitions[pid];
    graph_t* subgraph = &partition->subgraph;
    if (subgraph->vertex_count > 0) {
      subgraph->vertices =
        (eid_t*)malloc(sizeof(eid_t) * (subgraph->vertex_count + 1));
      assert(subgraph->vertices);
      partition->map = (vid_t*)calloc(subgraph->vertex_count, sizeof(vid_t));
      if (subgraph->edge_count > 0) {
        subgraph->edges = (vid_t*)malloc(sizeof(vid_t) * subgraph->edge_count);
        assert(subgraph->edges);
        if (pset->graph->weighted) {
          subgraph->weights = (weight_t*)malloc(sizeof(weight_t) *
                                                subgraph->edge_count);
          assert(subgraph->weights);
        }
      }
    }
  }
}

PRIVATE void init_build_map(partition_set_t* pset, vid_t* plabels) {
  // Reset the vertex and edge count, will be set again while building the map
  for (int pid = 0; pid < pset->partition_count; pid++) {
    pset->partitions[pid].subgraph.vertex_count = 0;
  }
  for (vid_t vid = 0; vid < pset->graph->vertex_count; vid++) {
    vid_t pid = plabels[vid];
    graph_t* subgraph = &pset->partitions[pid].subgraph;
     // forward map
    pset->id_in_partition[vid] = SET_PARTITION_ID(subgraph->vertex_count, pid);
    pset->partitions[pid].map[subgraph->vertex_count] = vid; // reverse map
    subgraph->vertex_count++;
  }
}

PRIVATE void init_build_partitions(partition_set_t* pset, vid_t* plabels,
                                   processor_t* pproc) {
  // build the map. The map maps the old vertex id to its new id in the
  // partition. This is necessary because the vertices assigned to a
  // partition will be renamed so that the ids are contiguous from 0 to
  // partition->subgraph.vertex_count - 1.
  init_build_map(pset, plabels);

  // Set the processor type and reset the vertex count, will be set again next
  for (int pid = 0; pid < pset->partition_count; pid++) {
    pset->partitions[pid].id = pid;
    pset->partitions[pid].processor = pproc[pid];
    pset->partitions[pid].subgraph.edge_count = 0;
    pset->partitions[pid].subgraph.vertex_count = 0;
  }

  // Construct the partitions vertex, edge and weight lists
  {
  graph_t* graph = pset->graph;
  for (vid_t vid = 0; vid < graph->vertex_count; vid++) {
    partition_t* partition = &pset->partitions[plabels[vid]];
    graph_t* subgraph = &partition->subgraph;
    subgraph->vertices[subgraph->vertex_count] =
      subgraph->edge_count;
    for (eid_t i = graph->vertices[vid]; i < graph->vertices[vid + 1]; i++) {
      subgraph->edges[subgraph->edge_count] =
        pset->id_in_partition[graph->edges[i]];
      if (graph->weighted) {
        subgraph->weights[subgraph->edge_count] =
          graph->weights[i];
      }
      subgraph->edge_count++;
    }
    subgraph->vertices[subgraph->vertex_count + 1] =
      subgraph->edge_count;
    subgraph->vertex_count++;
  }
  }
}

PRIVATE void init_sort_nbrs(partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  for (uint32_t pid = 0; pid < pcount; pid++) {
    graph_t* subgraph = &pset->partitions[pid].subgraph;
    OMP(omp parallel for)
    for (vid_t v = 0; v < subgraph->vertex_count; v++) {
      vid_t* nbrs = &subgraph->edges[subgraph->vertices[v]];
      qsort(nbrs, subgraph->vertices[v+1] - subgraph->vertices[v],
            sizeof(vid_t), compare_ids);
    }
  }
}

PRIVATE void init_build_partitions_gpu(partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  for (uint32_t pid = 0; pid < pcount; pid++) {
    partition_t* partition = &pset->partitions[pid];
    if (partition->processor.type != PROCESSOR_GPU) continue;
    CALL_CU_SAFE(hipSetDevice(partition->processor.id));
    CALL_CU_SAFE(hipStreamCreate(&partition->streams[0]));
    CALL_CU_SAFE(hipStreamCreate(&partition->streams[1]));
    CALL_CU_SAFE(hipEventCreate(&partition->event_start));
    CALL_CU_SAFE(hipEventCreate(&partition->event_end));
    graph_t* subgraph_h = (graph_t*)malloc(sizeof(graph_t));
    assert(subgraph_h);
    memcpy(subgraph_h, &partition->subgraph, sizeof(graph_t));
    graph_t* subgraph_d = NULL;
    CALL_SAFE(graph_initialize_device(subgraph_h, &subgraph_d));
    graph_finalize(subgraph_h);
    memcpy(&partition->subgraph, subgraph_d, sizeof(graph_t));
    free(subgraph_d);
  }
}

error_t partition_set_initialize(graph_t* graph, vid_t* plabels,
                                 processor_t* pproc, int pcount,
                                 size_t push_msg_size, size_t pull_msg_size,
                                 partition_set_t** pset) {
  assert(graph && plabels && pproc);
  if (pcount > MAX_PARTITION_COUNT) return FAILURE;

  // Setup space and initialize the partition set data structure
  CHK_SUCCESS(init_allocate_struct_space(graph, pcount, push_msg_size, 
                                         pull_msg_size, pset), err);

  // Get the partition sizes
  init_compute_partitions_sizes(*pset, plabels);

  // Allocate partitions space
  init_allocate_partitions_space(*pset);

  // Build the state of each partition
  init_build_partitions(*pset, plabels, pproc);

  // Sort nbrs of each each vertex to improve access locality
  init_sort_nbrs(*pset);

  // Initialize grooves' inbox and outbox state
  grooves_initialize(*pset);

  // Build the state on the GPU(s) for GPU residing partitions
  init_build_partitions_gpu(*pset);

  return SUCCESS;
 err:
  return FAILURE;
}

error_t partition_set_finalize(partition_set_t* pset) {
  assert(pset);
  assert(pset->partitions);
  for (int pid = 0; pid < pset->partition_count; pid++) {
    partition_t* partition = &pset->partitions[pid];
    graph_t* subgraph = &partition->subgraph;
    if (partition->processor.type == PROCESSOR_GPU) {
      CALL_CU_SAFE(hipSetDevice(partition->processor.id));
      CALL_CU_SAFE(hipStreamDestroy(partition->streams[0]));
      CALL_CU_SAFE(hipStreamDestroy(partition->streams[1]));
      CALL_CU_SAFE(hipEventDestroy(partition->event_start));
      CALL_CU_SAFE(hipEventDestroy(partition->event_end));
      CALL_CU_SAFE(hipFree(subgraph->edges));
      CALL_CU_SAFE(hipFree(subgraph->vertices));
      if (subgraph->weighted && subgraph->weights)
        CALL_CU_SAFE(hipFree(subgraph->weights));
    } else {
      assert(partition->processor.type == PROCESSOR_CPU);
      if (subgraph->vertices) free(subgraph->vertices);
      if (subgraph->edges) free(subgraph->edges);
      if (pset->weighted && subgraph->weights) {
        free(subgraph->weights);
      }
    }
    if (subgraph->vertices) free(partition->map);
  }
  grooves_finalize(pset);
  free(pset->partitions);
  free(pset->id_in_partition);
  free(pset);
  return SUCCESS;
}
