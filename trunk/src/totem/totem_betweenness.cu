#include "hip/hip_runtime.h"
/**
 *  Defines Betweenness Centrality functions for both CPU and GPU.
 *
 *  Created on: 2012-05-24
 *  Author: Greg Redekop
 */

// totem includes
#include "totem_centrality.h"
#include "totem_comdef.h"
#include "totem_comkernel.cuh"
#include "totem_graph.h"
#include "totem_mem.h"

/**
 * Checks for input parameters and special cases. This is invoked at the
 * beginning of public interfaces (GPU and CPU).
*/
PRIVATE
error_t check_special_cases(const graph_t* graph, bool* finished,
                            weight_t** centrality_score) {
  if (graph == NULL || graph->vertex_count == 0 || centrality_score == NULL) {
    return FAILURE;
  }

  if (graph->edge_count == 0) {
    *centrality_score = (weight_t*)mem_alloc(graph->vertex_count
                                             * sizeof(weight_t));
    memset(*centrality_score, (weight_t)0.0, graph->vertex_count
           * sizeof(weight_t));
    return SUCCESS;
  }

  *finished = false;
  return SUCCESS;
}

/**
 * Allocates and initializes memory on the GPU for the successors implementation
 * of betweenness centrality.
 */
PRIVATE
error_t initialize_succs_gpu(const graph_t* graph, uint64_t vertex_count,
                             graph_t** graph_d, vid_t** sigma_d,
                             int32_t** dists_d, vid_t** succ_d,
                             uint32_t** succ_count_d, vid_t** stack_d,
                             uint32_t** stack_count_d, weight_t** delta_d,
                             bool** finished_d,
                             weight_t** betweenness_centrality_d) {
  // Allocate space on GPU
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);
  CHK_CU_SUCCESS(hipMalloc((void**)sigma_d, vertex_count * sizeof(vid_t)),
                 err_free_graph_d);
  CHK_CU_SUCCESS(hipMalloc((void**)dists_d, vertex_count * sizeof(int32_t)),
                 err_free_sigma_d);
  CHK_CU_SUCCESS(hipMalloc((void**)succ_d, graph->edge_count * sizeof(vid_t)),
                 err_free_dists_d);
  CHK_CU_SUCCESS(hipMalloc((void**)succ_count_d, vertex_count
                            * sizeof(uint32_t)), err_free_succ_d);
  CHK_CU_SUCCESS(hipMalloc((void**)stack_d, vertex_count * vertex_count
                            * sizeof(vid_t)), err_free_succ_count_d);
  CHK_CU_SUCCESS(hipMalloc((void**)stack_count_d, vertex_count
                            * sizeof(uint32_t)), err_free_stack_d);
  CHK_CU_SUCCESS(hipMalloc((void**)delta_d, vertex_count * sizeof(weight_t)),
                 err_free_stack_count_d);
  CHK_CU_SUCCESS(hipMalloc((void**)finished_d, sizeof(bool)),
                 err_free_delta_d);
  CHK_CU_SUCCESS(hipMalloc((void**)betweenness_centrality_d, vertex_count
                            * sizeof(weight_t)), err_free_finished_d);

  // Setup initial parameters
  CHK_CU_SUCCESS(hipMemset(*betweenness_centrality_d, (weight_t)0.0,
                            vertex_count * sizeof(weight_t)), err_free_all);
  return SUCCESS;

 err_free_all:
  hipFree(betweenness_centrality_d);
 err_free_finished_d:
  hipFree(finished_d);
 err_free_delta_d:
  hipFree(delta_d);
 err_free_stack_count_d:
  hipFree(stack_count_d);
 err_free_stack_d:
  hipFree(stack_d);
 err_free_succ_count_d:
  hipFree(succ_count_d);
 err_free_succ_d:
  hipFree(succ_d);
 err_free_dists_d:
  hipFree(dists_d);
 err_free_sigma_d:
  hipFree(sigma_d);
 err_free_graph_d:
  graph_finalize_device(*graph_d);
 err:
  return FAILURE;
}

/**
 * Allocates and initializes memory on the GPU for the predecessors
 * implementation of betweenness centrality.
 */
PRIVATE
error_t initialize_preds_gpu(const graph_t* graph, uint64_t vertex_count,
                             vid_t* r_edges, graph_t** graph_d, 
                             vid_t** r_edges_d, bool** preds_d, 
                             vid_t** sigma_d, int32_t** dist_d,
                             weight_t** delta_d, bool** finished_d,
                             weight_t** betweenness_centrality_d) {
  // Allocate space on GPU
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);
  CHK_CU_SUCCESS(hipMalloc((void**)r_edges_d, graph->edge_count
                            * sizeof(vid_t)), err_free_graph_d);
  CHK_CU_SUCCESS(hipMalloc((void**)preds_d, graph->edge_count * sizeof(vid_t)),
                 err_free_r_edges_d);
  CHK_CU_SUCCESS(hipMalloc((void**)sigma_d, vertex_count * sizeof(vid_t)),
                 err_free_preds_d);
  CHK_CU_SUCCESS(hipMalloc((void**)dist_d, vertex_count * sizeof(int32_t)),
                 err_free_sigma_d);
  CHK_CU_SUCCESS(hipMalloc((void**)delta_d, vertex_count * sizeof(weight_t)),
                 err_free_dist_d);
  CHK_CU_SUCCESS(hipMalloc((void**)finished_d, sizeof(bool)),
                 err_free_delta_d);
  CHK_CU_SUCCESS(hipMalloc((void**)betweenness_centrality_d,
                             graph->vertex_count * sizeof(weight_t)),
                 err_free_finished_d);

  // Setup initial parameters
  CHK_CU_SUCCESS(hipMemcpy(*r_edges_d, r_edges, graph->edge_count
                            * sizeof(vid_t), hipMemcpyHostToDevice),
                 err_free_all);
  CHK_CU_SUCCESS(hipMemset(*betweenness_centrality_d, 0, graph->vertex_count
                            * sizeof(weight_t)), err_free_all);
  return SUCCESS;

 err_free_all:
  hipFree(betweenness_centrality_d);
 err_free_finished_d:
  hipFree(finished_d);
 err_free_delta_d:
  hipFree(delta_d);
 err_free_dist_d:
  hipFree(dist_d);
 err_free_sigma_d:
  hipFree(sigma_d);
 err_free_preds_d:
  hipFree(preds_d);
 err_free_r_edges_d:
  hipFree(r_edges_d);
 err_free_graph_d:
  graph_finalize_device(*graph_d);
 err:
  return FAILURE;
}

/**
 * Finalize function for the successor stack GPU implementation. It allocates
 * the host output buffer, moves the final results from GPU to the host buffers
 * and frees up GPU resources.
 */
PRIVATE
error_t finalize_succs_gpu(graph_t* graph_d, vid_t* sigma_d, int32_t* dist_d,
                           vid_t* succ_d, uint32_t* succ_count_d, 
                           vid_t* stack_d, uint32_t* stack_count_d, 
                           weight_t* delta_d, bool* finished_d, 
                           weight_t* betweenness_centrality_d,
                           weight_t* betweenness_centrality) {
  // Copy back the centrality scores
  CHK_CU_SUCCESS(hipMemcpy(betweenness_centrality, betweenness_centrality_d,
                            graph_d->vertex_count * sizeof(weight_t),
                            hipMemcpyDeviceToHost), err);
  graph_finalize_device(graph_d);
  hipFree(sigma_d);
  hipFree(dist_d);
  hipFree(succ_d);
  hipFree(succ_count_d);
  hipFree(stack_d);
  hipFree(stack_count_d);
  hipFree(delta_d);
  hipFree(finished_d);
  hipFree(betweenness_centrality_d);
  return SUCCESS;

 err:
  return FAILURE;
}

/**
 * Finalize function for the predecessor map GPU implementation. It allocates
 * the host output buffer, moves the final results from GPU to the host buffers
 * and frees up GPU resources.
 */
PRIVATE
error_t finalize_preds_gpu(graph_t* graph_d, vid_t* r_edges_d, bool* preds_d,
                           vid_t* sigma_d, int32_t* dist_d, weight_t* delta_d,
                           bool* finished_d, weight_t* betweenness_centrality_d,
                           weight_t* betweenness_centrality) {
  // Copy back the centrality scores
  CHK_CU_SUCCESS(hipMemcpy(betweenness_centrality, betweenness_centrality_d,
                            graph_d->vertex_count * sizeof(weight_t),
                            hipMemcpyDeviceToHost), err);
  graph_finalize_device(graph_d);
  hipFree(r_edges_d);
  hipFree(preds_d);
  hipFree(sigma_d);
  hipFree(dist_d);
  hipFree(delta_d);
  hipFree(finished_d);
  hipFree(betweenness_centrality_d);
  return SUCCESS;

 err:
  return FAILURE;
}

/**
 * This kernel is invoked for each iteration of the successors GPU betweenness
 * algorithm. It re-initializes variables for the SSSP problem using a different
 * source vertex.
 */
__global__
void unweighted_bc_succs_init_kernel(vid_t source, vid_t* sigma, int32_t* dist,
                                     uint32_t* stack_count, vid_t* stack) {
  sigma[source] = 1;
  dist[source] = 0;
  stack_count[0] = 1;
  stack[0] = source;
}

/**
 * This kernel is invoked for each iteration of the predecessors GPU betweenness
 * algorithm. It re-initializes variables for the SSSP problem using a different
 * source vertex.
 */
__global__
void unweighted_bc_preds_init_kernel(vid_t source, int32_t* dist, 
                                     vid_t* sigma) {
  dist[source] = 0;
  sigma[source] = 1;
}

/**
 * For each iteration of the successors GPU betweenness algorithm, we have to
 * reset all the variables and setup the initial parameters for the SSSP problem
 * using the new source vertex.
 */
PRIVATE
error_t unweighted_succs_init(const graph_t* graph, vid_t source, vid_t* sigma,
                              int32_t* dist, vid_t* succ, uint32_t* succ_count,
                              vid_t* stack, uint32_t* stack_count,
                              weight_t* delta) {
  // Perform the memsets directly on the GPU
  dim3 blocks;
  dim3 threads_per_block;
  CHK_CU_SUCCESS(hipMemset(succ, 0, graph->edge_count * sizeof(vid_t)), err);
  CHK_CU_SUCCESS(hipMemset(stack, 0, graph->vertex_count * graph->vertex_count
                            * sizeof(vid_t)), err);
  CHK_CU_SUCCESS(hipMemset(succ_count, 0, graph->vertex_count
                            * sizeof(uint32_t)), err);
  CHK_CU_SUCCESS(hipMemset(stack_count, 0, graph->vertex_count
                            * sizeof(uint32_t)), err);
  CHK_CU_SUCCESS(hipMemset(sigma, 0, graph->vertex_count * sizeof(vid_t)), 
                 err);
  CHK_CU_SUCCESS(hipMemset(dist, -1, graph->vertex_count * sizeof(int32_t)),
                 err);
  CHK_CU_SUCCESS(hipDeviceSynchronize(), err);

  // Initialize the appropriate counts for the source vertex
  KERNEL_CONFIGURE(1, blocks, threads_per_block);
  unweighted_bc_succs_init_kernel<<<blocks, threads_per_block>>>
    (source, sigma, dist, stack_count, stack);
  CALL_CU_SAFE(hipGetLastError());
  return SUCCESS;

 err:
  return FAILURE;
}

/**
 * Unweighted betweenness centrality dependence accumulation kernel for the
 * successors stack GPU implementation. After finding the APSP solution and
 * counts of shortest paths through each node, this function calculates the
 * dependence for each node.
 */
__global__ void
unweighted_dep_acc_kernel(graph_t graph, int64_t phase, uint32_t* stack_count, 
                          vid_t* sigma, vid_t* stack, vid_t* succ, 
                          uint32_t* succ_count, weight_t* delta, 
                          weight_t* betweenness_centrality) {
  const vid_t thread_id = THREAD_GLOBAL_INDEX;

  if (thread_id < stack_count[phase]) {
    vid_t w = stack[graph.vertex_count * phase + thread_id];
    weight_t dsw = 0.0;
    weight_t sw = sigma[w];
    for (vid_t i = 0; i < succ_count[w]; i++) {
      vid_t v = succ[graph.vertices[w] + i];
      dsw = dsw + (sw / sigma[v]) * (1.0 + delta[v]);
    }
    delta[w] = dsw;
    atomicAdd(&betweenness_centrality[w], dsw);
  }
}

/**
 * Unweighted betweenness centrality back sum kernel for predecessor map
 * implementation. This function calculates the actual betweenness centrality
 * score by summing dependences for each vertex.
 */
__global__
void unweighted_back_sum_kernel(graph_t graph, vid_t source, int32_t dist,
                                int32_t* dists, weight_t* delta,
                                weight_t* betweenness_centrality) {
  const vid_t thread_id = THREAD_GLOBAL_INDEX;
  if (thread_id < graph.vertex_count) {
    if (thread_id != source && dists[thread_id] == (dist - 1)) {
      betweenness_centrality[thread_id] += delta[thread_id];
    }
  }
}

/**
 * Implements the parallel Brandes betweenness centrality algorithm using a
 * successor stack, as described in "A Faster Parallel Algorithm and Efficient
 * Multithreaded Implementations for Evaluating Betweenness Centrality on
 * Massive Datasets" [Madduri09]
 */
error_t betweenness_unweighted_gpu(const graph_t* graph,
                                   weight_t** centrality_score) {
  // Sanity check on input
  bool finished = true;
  error_t rc = check_special_cases(graph, &finished, centrality_score);
  if (finished) return rc;

  // Allocate space for the results
  weight_t* betweenness_centrality =
    (weight_t*)mem_alloc(graph->vertex_count * sizeof(weight_t));
  // Allocate memory and initialize state on the GPU
  graph_t* graph_d;
  vid_t* sigma_d;
  int32_t* dist_d;
  vid_t* succ_d;
  uint32_t* succ_count_d;
  vid_t* stack_d;
  uint32_t* stack_count_d;
  weight_t* delta_d;
  bool* finished_d;
  weight_t* betweenness_centrality_d;

  // Initialization stage
  CHK_SUCCESS(initialize_succs_gpu(graph, graph->vertex_count, &graph_d,
                                   &sigma_d, &dist_d, &succ_d, &succ_count_d,
                                   &stack_d, &stack_count_d, &delta_d,
                                   &finished_d, &betweenness_centrality_d),
              err_free_betweenness);

  // {} used to limit scope and avoid problems with error handles.
  {
  dim3 blocks;
  dim3 threads_per_block;

  // Find and count all shortest paths from every source vertex to every other
  // vertex in the graph. These paths and counts are used to determine the
  // betweenness centrality for each vertex
  for (vid_t source = 0; source < graph->vertex_count; source++) {
    // Initializations for this iteration
    CHK_SUCCESS(unweighted_succs_init(graph, source, sigma_d, dist_d, succ_d,
                                      succ_count_d, stack_d, stack_count_d,
                                      delta_d), err_free_all);
    CHK_CU_SUCCESS(hipGetLastError(), err_free_all);

    // SSSP and path counting stage
    KERNEL_CONFIGURE(graph->edge_count, blocks, threads_per_block);
    bool finished = false;
    int64_t phase = 0;
    // Keep counting distances until the BFS kernel completes
    while (!finished) {
      CHK_CU_SUCCESS(hipMemset(finished_d, true, sizeof(bool)), err_free_all);
      CHK_CU_SUCCESS(hipDeviceSynchronize(), err_free_all);
      unweighted_sssp_succs_kernel<<<blocks, threads_per_block>>>
        (*graph_d, phase, sigma_d, dist_d, succ_d, succ_count_d, stack_d,
         stack_count_d, finished_d);
      CHK_CU_SUCCESS(hipMemcpy(&finished, finished_d, sizeof(bool),
                                hipMemcpyDeviceToHost), err_free_all);
      phase++;
    }

    // Dependency accumulation stage
    phase -= 2;
    CHK_CU_SUCCESS(hipMemset(delta_d, (weight_t)0.0,
                              graph->vertex_count * sizeof(vid_t)),
                   err_free_all);
    KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);
    while (phase > 0) {
      unweighted_dep_acc_kernel<<<blocks, threads_per_block>>>
        (*graph_d, phase, stack_count_d, sigma_d, stack_d, succ_d, succ_count_d,
         delta_d, betweenness_centrality_d);
      CHK_CU_SUCCESS(hipDeviceSynchronize(), err_free_all);
      CHK_CU_SUCCESS(hipGetLastError(), err_free_all);
      phase--;
    }
  }}

  // Cleanup phase
  CHK_SUCCESS(finalize_succs_gpu(graph_d, sigma_d, dist_d, succ_d, succ_count_d,
                                 stack_d, stack_count_d, delta_d, finished_d,
                                 betweenness_centrality_d,
                                 betweenness_centrality), err_free_all);

  // If the graph is undirected, divide centrality scores by 2
  if (graph->directed == false) {
    for (vid_t v = 0; v < graph->vertex_count; v++) {
      betweenness_centrality[v] /= 2.0;
    }
  }

  // Return the centrality
  *centrality_score = betweenness_centrality;
  return SUCCESS;

 err_free_all:
  graph_finalize_device(graph_d);
  hipFree(sigma_d);
  hipFree(dist_d);
  hipFree(succ_d);
  hipFree(succ_count_d);
  hipFree(stack_d);
  hipFree(stack_count_d);
  hipFree(delta_d);
  hipFree(finished_d);
  hipFree(betweenness_centrality_d);
 err_free_betweenness:
  mem_free(betweenness_centrality);
  return FAILURE;
}

/**
 * Implements the parallel Brandes betweenness centrality algorithm using
 * predecessor maps as described in "Fast Network Centrality Analysis Using
 * GPUs" [Shi11]
 */
error_t betweenness_unweighted_shi_gpu(const graph_t* graph,
                                       weight_t** centrality_score) {
  // Sanity check on input
  bool finished = true;
  error_t rc = check_special_cases(graph, &finished, centrality_score);
  if (finished) return rc;

  // Allocate space for the results
  weight_t* betweenness_centrality =
    (weight_t*)mem_alloc(graph->vertex_count * sizeof(weight_t));
  // Construct the reverse edges list (graph->edges is a list of destination
  // vertices, r_edges is a list of source vertices, indexed by edge id)
  vid_t* r_edges = (vid_t*)mem_alloc(graph->edge_count * sizeof(vid_t));
  vid_t v = 0;
  for (eid_t e = 0; e < graph->edge_count; e++) {
    while (v <= graph->vertex_count &&
           !(e >= graph->vertices[v] && e < graph->vertices[v+1])) {
      v++;
    }
    r_edges[e] = v;
  }

  // Allocate memory and initialize state on the GPU
  graph_t* graph_d;
  vid_t* r_edges_d;
  bool* preds_d;
  vid_t* sigma_d;
  int32_t* dist_d;
  weight_t* delta_d;
  bool* finished_d;
  weight_t* betweenness_centrality_d;


  CHK_SUCCESS(initialize_preds_gpu(graph, graph->vertex_count, r_edges,
                                   &graph_d, &r_edges_d, &preds_d, &sigma_d,
                                   &dist_d, & delta_d, &finished_d,
                                   &betweenness_centrality_d),
              err_free_betweenness);

  // {} used to limit scope and avoid problems with error handles.
  {
  dim3 blocks;
  dim3 threads_per_block;

  // Find and count all shortest paths from every source vertex to every other
  // vertex in the graph. These paths and counts are used to determine the
  // betweenness centrality for each vertex
  for (vid_t source = 0; source < graph->vertex_count; source++) {
    // APSP
    int32_t dist = 0;
    CHK_CU_SUCCESS(hipMemset(dist_d, -1, graph->vertex_count
                                          * sizeof(int32_t)), err_free_all);
    CHK_CU_SUCCESS(hipMemset(preds_d, false, graph->edge_count * sizeof(bool)),
                   err_free_all);
    CHK_CU_SUCCESS(hipMemset(sigma_d, 0, graph->vertex_count * sizeof(vid_t)),
                   err_free_all);
    CHK_CU_SUCCESS(hipMemset(delta_d, 0, graph->vertex_count
                              * sizeof(weight_t)), err_free_all);
    KERNEL_CONFIGURE(1, blocks, threads_per_block);
    unweighted_bc_preds_init_kernel<<<blocks, threads_per_block>>>
      (source, dist_d, sigma_d);
    CHK_CU_SUCCESS(hipDeviceSynchronize(), err_free_all);

    KERNEL_CONFIGURE(graph->edge_count, blocks, threads_per_block);
    bool finished = false;
    while (!finished) {
      CHK_CU_SUCCESS(hipMemset(finished_d, true, sizeof(bool)), err_free_all);
      unweighted_sssp_preds_kernel<<<blocks, threads_per_block>>>
        (*graph_d, r_edges_d, dist, dist_d, sigma_d, preds_d, finished_d);
      CHK_CU_SUCCESS(hipDeviceSynchronize(), err_free_all);
      CHK_CU_SUCCESS(hipMemcpy(&finished, finished_d, sizeof(bool),
                                hipMemcpyDeviceToHost), err_free_all);
      dist++;
    }
    // Back Propogation
    while (dist > 1) {
      KERNEL_CONFIGURE(graph->edge_count, blocks, threads_per_block);
      unweighted_back_prop_kernel<<<blocks, threads_per_block>>>
        (*graph_d, r_edges_d, dist_d, sigma_d, preds_d, dist, delta_d);
      KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);
      unweighted_back_sum_kernel<<<blocks, threads_per_block>>>
        (*graph_d, source, dist, dist_d, delta_d, betweenness_centrality_d);
      dist--;
    }
  }}

  CHK_SUCCESS(finalize_preds_gpu(graph_d, r_edges_d, preds_d, sigma_d, dist_d,
                                 delta_d, finished_d, betweenness_centrality_d,
                                 betweenness_centrality), err_free_all);
  mem_free(r_edges);

  // If the graph is undirected, divide all the centrality scores by two
  if (graph->directed == false) {
    for (vid_t v = 0; v < graph->vertex_count; v++) {
      betweenness_centrality[v] /= 2.0;
    }
  }

  // Return the centrality
  *centrality_score = betweenness_centrality;
  return SUCCESS;

 err_free_all:
  graph_finalize_device(graph_d);
  hipFree(r_edges_d);
  hipFree(preds_d);
  hipFree(sigma_d);
  hipFree(dist_d);
  hipFree(delta_d);
  hipFree(finished_d);
  hipFree(betweenness_centrality_d);
 err_free_betweenness:
  mem_free(betweenness_centrality);
  mem_free(r_edges);
  return FAILURE;
}

/**
 * Implements the parallel Brandes betweenness centrality algorithm using a
 * successor stack, as described in "A Faster Parallel Algorithm and Efficient
 * Multithreaded Implementations for Evaluating Betweenness Centrality on
 * Massive Datasets" [Madduri09]
 */
error_t betweenness_unweighted_cpu(const graph_t* graph,
                                   weight_t** centrality_score) {
  // Sanity check on input
  bool finished = true;
  error_t rc = check_special_cases(graph, &finished, centrality_score);
  if (finished) return rc;

  // Allocate memory for the shortest paths problem
  weight_t* betweenness_centrality =
    (weight_t*)mem_alloc(graph->vertex_count * sizeof(weight_t));
  uint32_t* sigma = 
    (uint32_t*)mem_alloc(graph->vertex_count * sizeof(uint32_t));
  int32_t* dist = (int32_t*)mem_alloc(graph->vertex_count * sizeof(int32_t));
  vid_t* succ = (vid_t*)mem_alloc(graph->edge_count * sizeof(vid_t));
  vid_t* succ_count = (vid_t*)mem_alloc(graph->vertex_count * sizeof(vid_t));
  vid_t* stack = (vid_t*)mem_alloc(graph->vertex_count * graph->vertex_count
                                 * sizeof(vid_t));
  vid_t* stack_count = (vid_t*)mem_alloc(graph->vertex_count * sizeof(vid_t));
  weight_t* delta =
    (weight_t*)mem_alloc(graph->vertex_count * sizeof(weight_t));
  int64_t phase = 0;

  // Initialization stage
  OMP(omp parallel for)
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    betweenness_centrality[v] = (weight_t)0.0;
  }

  // Find and count all shortest paths from every source vertex to every other
  // vertex in the graph. These paths and counts are used to determine the
  // betweenness centrality for each vertex
  for (vid_t source = 0; source < graph->vertex_count; source++) {
    // Initializations for this iteration
    memset(succ, 0, graph->edge_count * sizeof(vid_t));
    memset(succ_count, 0, graph->vertex_count * sizeof(vid_t));
    memset(stack, 0, graph->vertex_count * graph->vertex_count * sizeof(vid_t));
    memset(stack_count, 0,  graph->vertex_count * sizeof(vid_t));
    OMP(omp parallel for)
    for (vid_t t = 0; t < graph->vertex_count; t++) {
      sigma[t] = 0;
      dist[t] = -1;
    }
    sigma[source] = 1;
    dist[source] = 0;
    phase = 0;
    stack_count[phase] = 1;
    stack[graph->vertex_count * phase] = source;

    // SSSP and path counting
    bool finished = false;
    while (!finished) {
      finished = true;
      for (vid_t v_index = 0; v_index < stack_count[phase]; v_index++) {
        vid_t v = stack[graph->vertex_count * phase + v_index];
        // For all neighbors of v in parallel, iterate over paths
        OMP(omp parallel for)
        for (eid_t e = graph->vertices[v]; e < graph->vertices[v + 1]; e++) {
          vid_t w = graph->edges[e];
          int32_t dw = __sync_val_compare_and_swap(&dist[w], (uint32_t)-1,
                                                   phase + 1);
          if (dw == -1) {
            finished = false;
            vid_t p = __sync_fetch_and_add(&stack_count[phase + 1], 1);
            stack[graph->vertex_count * (phase + 1) + p] = w;
            dw = phase + 1;
          }
          if (dw == phase + 1) {
            vid_t p = (vid_t)__sync_fetch_and_add(&succ_count[v], 1);
            succ[graph->vertices[v] + p] = w;
            __sync_fetch_and_add(&sigma[w], sigma[v]);
          }
        }
      }
      phase++;
    }
    phase--;

    // Dependency accumulation stage
    memset(delta, (weight_t)0.0, graph->vertex_count * sizeof(vid_t));
    phase--;
    while (phase > 0) {
      OMP(omp parallel for)
      for (vid_t p = 0; p < stack_count[phase]; p++) {
        vid_t w = stack[graph->vertex_count * phase + p];
        weight_t dsw = 0.0;
        weight_t sw = sigma[w];
        for (vid_t i = 0; i < succ_count[w]; i++) {
          vid_t v = succ[graph->vertices[w] + i];
          dsw = dsw + (sw / sigma[v]) * (1.0 + delta[v]);
        }
        delta[w] = dsw;
        betweenness_centrality[w] = betweenness_centrality[w] + dsw;
      }
      phase--;
    }
  }

  // If the graph is undirected, divide centrality scores by 2
  if (graph->directed == false) {
    OMP(omp parallel for)
    for (vid_t v = 0; v < graph->vertex_count; v++) {
      betweenness_centrality[v] /= 2.0;
    }
  }

  // Cleanup phase
  mem_free(sigma);
  mem_free(dist);
  mem_free(delta);
  mem_free(stack_count);
  mem_free(succ);
  mem_free(succ_count);
  mem_free(stack);

  // Return the centrality
  *centrality_score = betweenness_centrality;
  return SUCCESS;
}

/**
 * Implements the forward propagation phase of the Betweenness Centrality
 * Algorithm described in Chapter 2 of GPU Computing Gems. Utilized by:
 * error_t betweenness_cpu(const graph_t* graph, weight_t** centrality_score)
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[in] source the source node for the shortest paths
 * @param[in] level the shared level variable between backward and forward
 *            propagations
 * @param[in] numSPs an array which counts the number of shortest paths in
 *            which each node is involved 
 * @param[in] distance an array which stores the distance of the shortest
 *            path for each node
 * @return void
 */
PRIVATE void betweenness_cpu_forward_propagation(const graph_t* graph, 
                                        vid_t source, int64_t& level,
                                        uint32_t* numSPs, int32_t* distance) {
  // Initialize the shortest path count to 0 and distance to infinity given
  // this source node
  OMP(omp parallel for)
  for (vid_t v = 0; v < graph->vertex_count; v++) {
    numSPs[v] = 0;
    distance[v] = -1;
  }
  // Set the distance from source to itself to 0
  distance[source] = 0;
  // Set the shortest path count to 1 (from source to itself)
  numSPs[source] = 1;

  bool done = false;
  while (!done) {
    done = true;
    // In parallel, iterate over vertices which are at the current level
    OMP(omp parallel for)
    for (vid_t v = 0; v < graph->vertex_count; v++) {
      if (distance[v] == level) {
        // For all neighbors of v, iterate over paths
        for (eid_t e = graph->vertices[v]; e < graph->vertices[v + 1]; e++) {
          vid_t w = graph->edges[e];
          if (distance[w] == -1) {
            distance[w] = level + 1;
            done = false;
          }
          if (distance[w] == level + 1) {
            __sync_fetch_and_add(&numSPs[w], numSPs[v]);
          }
        }
      }
    }
    level++;
  }
}

/**
 * Implements the backward propagation phase of the Betweenness Centrality
 * Algorithm described in Chapter 2 of GPU Computing Gems. Utilized by:
 * error_t betweenness_cpu(const graph_t* graph, weight_t** centrality_score)
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[in] source the source node for the shortest paths
 * @param[in] level the shared level variable between backward and forward
 *            propagations
 * @param[in] numSPs an array which counts the number of shortest paths in
 *            which each node is involved 
 * @param[in] distance an array which stores the distance of the shortest
 *            path for each node
 * @param[in] delta an array of the dependencies for each node, which are used
 *            to compute the betweenness centrality measure
 * @param[out] betweenness_centrality the output list which contains the
 *             betweenness centrality values computed for each node
 * @return void
 */
PRIVATE void betweenness_cpu_backward_propagation(const graph_t* graph,
                                          vid_t source, int64_t& level, 
                                          uint32_t* numSPs, int32_t* distance,
                                          weight_t* delta, 
                                          weight_t* betweenness_centrality) {
  // Set deltas to 0 for every input node
  memset(delta, 0, graph->vertex_count * sizeof(vid_t));
  while (level > 1) {
    level--;
    // In parallel, iterate over vertices which are at the current level
    OMP(omp parallel for)
    for (vid_t v = 0; v < graph->vertex_count; v++) {
      if (distance[v] == level) {
        // For all neighbors of v, iterate over paths
        for (eid_t e = graph->vertices[v]; e < graph->vertices[v + 1]; e++) {
          vid_t w = graph->edges[e];
          if (distance[w] == level + 1) {
            delta[v] = (delta[v] + ((((weight_t)numSPs[v]) /
                       ((weight_t)numSPs[w]))*(delta[w] + 1)));
          }
        }
        // Add the dependency to the BC sum
        betweenness_centrality[v] = betweenness_centrality[v] + delta[v];
      }
    }
  }
}

/**
 * Parallel CPU implementation of  Bewteenness Centrality algorithm described
 * in Chapter 2 of GPU Computing Gems (Algorithm 1 - Sequential BC Computation)
 * @param[in] graph the graph for which the centrality measure is calculated
 * @param[out] centrality_score the output list of betweenness centrality scores
 *             per vertex
 * @return generic success or failure
 */
error_t betweenness_cpu(const graph_t* graph, weight_t** centrality_score) {
  // Sanity check on input
  bool finished = true;
  error_t rc = check_special_cases(graph, &finished, centrality_score);
  if (finished) return rc;

  // Allocate memory for the shortest paths problem
  int32_t* distance = (int32_t*)malloc(graph->vertex_count * sizeof(int32_t));
  uint32_t* numSPs = (uint32_t*)malloc(graph->vertex_count * sizeof(uint32_t));
  weight_t* delta = (weight_t*)malloc(graph->vertex_count * sizeof(weight_t));
  weight_t* betweenness_centrality = (weight_t*)mem_alloc(graph->vertex_count
                                                          * sizeof(weight_t));

  // Initialization stage
  // Set BC(v) to 0 for every input node
  memset(betweenness_centrality, 0, graph->vertex_count * sizeof(vid_t));

  // Main loop - iterate over every node and perform both forward propagation
  // and backward propagation for that node, which in turn computes the
  // Betweenness Centrality, as described by the reference algorithm
  for (vid_t source = 0; source < graph->vertex_count; source++) {
    // Initialize variable to keep track of level
    int64_t level = 0;

    // Perform the forward propagation phase for this source node
    betweenness_cpu_forward_propagation(graph, source, level, numSPs, distance);

    // Perform the backward propagation phase for this source node
    betweenness_cpu_backward_propagation(graph, source, level, numSPs, distance,
                                         delta, betweenness_centrality);
  }

  // Cleanup the allocated memory
  free(numSPs);
  free(distance);
  free(delta);

  // Return the centrality
  *centrality_score = betweenness_centrality;
  return SUCCESS;
}

