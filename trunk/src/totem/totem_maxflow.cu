#include "hip/hip_runtime.h"
/* TODO(lauro,abdullah,elizeu): Add license.
 *
 * Implements the push-relabel algorithm for determining the maximum flow
 * through a directed graph. This implementation is based on the algorithms
 * presented by [Hong08] Z. He, B. Hong, "Dynamically Tuned Push-Relabel
 * Algorithm for the Maximum Flow Problem on CPU-GPU-Hybrid Platforms".
 * http://users.ece.gatech.edu/~bhong/papers/ipdps10.pdf
 *
 *  Created on: 2011-10-21
 *      Author: Greg Redekop
 */

// system includes
#include <hip/hip_runtime.h>

// totem includes
#include "totem_comdef.h"
#include "totem_comkernel.cuh"
#include "totem_graph.h"
#include "totem_mem.h"

// For each stage of the algorithm, a kernel loops over each vertex this many
// times attempting pushes and relabels. This will also control the frequency
// of global relabeling.
#define KERNEL_CYCLES 35

// Static function declarations
__global__
void init_preflow(graph_t graph, id_t edge_base, id_t edge_end, weight_t* flow,
                  weight_t* excess, id_t* reverse_indices);

/**
   This structure is used by the virtual warp-based implementation. It stores a
   batch of work. It is typically allocated on shared memory and is processed by
   a single virtual warp.
 */
typedef struct {
  uint32_t height[VWARP_BATCH_SIZE];
  id_t vertices[VWARP_BATCH_SIZE + 1];
  // the following ensures 64-bit alignment, it assumes that the cost and
  // vertices arrays are of 32-bit elements.
  // TODO(abdullah) a portable way to do this (what if id_t is 64-bit?)
  int pad;
} vwarp_mem_t;


/**
 * Checks for input parameters and special cases. This is invoked at the
 * beginning of public interfaces (GPU and CPU)
 */
PRIVATE
error_t check_special_cases(graph_t* graph, id_t source_id, id_t sink_id) {
  if((graph == NULL) || (graph->vertex_count == 0) || (!graph->weighted) ||
     (!graph->directed) || (source_id >= graph->vertex_count) ||
     (sink_id >= graph->vertex_count) || (source_id == sink_id)) {
    return FAILURE;
  }
  return SUCCESS;
}


/**
 * A common initialization function for GPU implementations. It allocates and
 * initalizes state on the GPU
 */
PRIVATE
error_t initialize_gpu(graph_t* graph, id_t source_id, uint64_t vwarp_length,
                       id_t* reverse_indices, graph_t** graph_d,
                       weight_t** flow_d, weight_t** excess_d,
                       uint32_t** height_d, id_t** reverse_indices_d,
                       bool** finished_d) {

  dim3 blocks;
  dim3 threads_per_block;

  // Calculate the source excess directly prior to allocation. This prevents
  // compilation errors about variable declaration after a jump
  weight_t source_excess = (weight_t)0;
  for (id_t edge_id = graph->vertices[source_id];
       edge_id < graph->vertices[source_id + 1]; edge_id++) {
    source_excess -= graph->weights[edge_id];
  }
  // Allocate space on GPU
  CHK_SUCCESS(graph_initialize_device(graph, graph_d), err);
  CHK_CU_SUCCESS(hipMalloc((void**)flow_d, graph->edge_count *
                            sizeof(weight_t)), err_free_graph_d);
  CHK_CU_SUCCESS(hipMalloc((void**)reverse_indices_d, graph->edge_count *
                            sizeof(id_t)), err_free_flow_d);
  CHK_CU_SUCCESS(hipMalloc((void**)excess_d, graph->vertex_count *
                            sizeof(weight_t)), err_free_reverse_indices_d);
  CHK_CU_SUCCESS(hipMalloc((void**)height_d, vwarp_length * sizeof(uint32_t)),
                 err_free_excess_d);
  // Initialize flow, height, and excess to 0.
  KERNEL_CONFIGURE(graph->edge_count, blocks, threads_per_block);
  memset_device<<<blocks, threads_per_block>>>((*flow_d), (weight_t)0,
                                               graph->edge_count);
  CHK_CU_SUCCESS(hipGetLastError(), err_free_all_d);
  KERNEL_CONFIGURE(vwarp_length, blocks, threads_per_block);
  memset_device<<<blocks, threads_per_block>>>((*height_d), (uint32_t)0,
                                               vwarp_length);
  KERNEL_CONFIGURE(graph->vertex_count, blocks, threads_per_block);
  memset_device<<<blocks, threads_per_block>>>((*excess_d), (weight_t)0,
                                               graph->vertex_count);
  CHK_CU_SUCCESS(hipGetLastError(), err_free_all_d);
  CHK_CU_SUCCESS(hipMemcpy((*reverse_indices_d), reverse_indices,
                            graph->edge_count * sizeof(id_t),
                            hipMemcpyHostToDevice), err_free_all_d);

  // From the source vertex, initialize preflow
  CHK_CU_SUCCESS(hipMemset(&((*height_d)[source_id]), graph->vertex_count,
                            sizeof(uint32_t)), err_free_all_d);
  KERNEL_CONFIGURE((graph->vertices[source_id + 1] -
                    graph->vertices[source_id]), blocks, threads_per_block);
  init_preflow<<<blocks, threads_per_block>>>
    (**graph_d, graph->vertices[source_id], graph->vertices[source_id + 1],
     *flow_d, *excess_d, *reverse_indices_d);
  CHK_CU_SUCCESS(hipGetLastError(), err_free_all_d);

  CHK_CU_SUCCESS(hipMemset(&((*excess_d)[source_id]), source_excess,
                            sizeof(weight_t)), err_free_all_d);
  // Allocate the termination flag
  CHK_CU_SUCCESS(hipMalloc((void**)finished_d, sizeof(bool)),
                 err_free_all_d);

  return SUCCESS;

  err_free_all_d:
    hipFree(height_d);
  err_free_excess_d:
    hipFree(excess_d);
  err_free_reverse_indices_d:
    hipFree(reverse_indices_d);
  err_free_flow_d:
    hipFree(flow_d);
  err_free_graph_d:
    graph_finalize_device(*graph_d);
  err:
    return FAILURE;
}


/**
 * Initializes preflow from a given source index. Each edge connected to
 * the source vertex is initialized with its capacity.
 */
__global__
void init_preflow(graph_t graph, id_t edge_base, id_t edge_end, weight_t* flow,
                  weight_t* excess, id_t* reverse_indices) {
  const int offset = THREAD_GLOBAL_INDEX;
  if (offset >= edge_end) return;
  if (graph.weights[edge_base + offset] == 0) return;

  flow[edge_base + offset] = graph.weights[edge_base + offset];
  flow[reverse_indices[edge_base + offset]] = -flow[edge_base + offset];
  excess[graph.edges[edge_base + offset]] = graph.weights[edge_base + offset];
}


/**
 * Implements the push relabel kernel, as per [Hong08]
 */
__global__
void push_relabel_kernel(graph_t graph, weight_t* flow, weight_t* excess,
                         uint32_t* height, id_t* reverse_indices,
                         id_t source_id, id_t sink_id, bool* finished) {
  const id_t u = THREAD_GLOBAL_INDEX;
  if (u >= graph.vertex_count) return;
  if (u == source_id || u == sink_id) return;

  uint32_t count = KERNEL_CYCLES;
  while (count--) {
    if (excess[u] <= 0 || height[u] >= graph.vertex_count) continue;

    weight_t e_prime = excess[u];
    uint32_t h_prime = INFINITE;
    id_t best_edge_id = INFINITE;

    // Find the lowest neighbor connected by a residual edge
    for (id_t edge_id = graph.vertices[u]; edge_id < graph.vertices[u + 1];
         edge_id++) {
      if (graph.weights[edge_id] == flow[edge_id]) continue;
      uint32_t h_pprime = height[graph.edges[edge_id]];
      if (h_pprime < h_prime) {
        best_edge_id = edge_id;
        h_prime = h_pprime;
      }
    }

    // If a push applies
    if (height[u] > h_prime) {
      weight_t push_amt = min(e_prime, graph.weights[best_edge_id] -
                              flow[best_edge_id]);
      atomicAdd(&flow[best_edge_id], push_amt);
      atomicAdd(&flow[reverse_indices[best_edge_id]], -push_amt);
      atomicAdd(&excess[u], -push_amt);
      atomicAdd(&excess[graph.edges[best_edge_id]], push_amt);
      *finished = false;
    }
    // Otherwise perform a relabel
    else if (h_prime != INFINITE) {
      height[u] = h_prime + 1;
      *finished = false;
    }
  }
}


/**
 * The neighbors processing function. This function finds the smallest neighbor
 * height and sets the corresponding best edge index for the vertex. The
 * assumption is that the threads of a warp invoke this function to process the
 * warp's batch of work. In each iteration of the for loop, each thread
 * processes a neighbor. For example, thread 0 in the warp processes neighbors
 * at indices 0, VWARP_WARP_SIZE, (2 * VWARP_WARP_SIZE) etc. in the edges array,
 * while thread 1 in the warp processes neighbors 1, (1 + VWARP_WARP_SIZE),
 * (1 + 2 * VWARP_WARP_SIZE) and so on.
*/
__device__
void vwarp_process_neighbors(int warp_offset, int warp_id, int neighbor_count,
                             id_t* neighbors, weight_t* flow, weight_t* weight,
                             uint32_t* height, uint32_t* lowest_height,
                             id_t* best_edge_id) {
  for (int i = warp_offset; i < neighbor_count; i += VWARP_WARP_SIZE) {
    id_t neighbor_id = neighbors[i];
    if (weight[i] > flow[i]) {
      uint32_t h_pprime = height[neighbor_id];
      while (*lowest_height > h_pprime) {
        *lowest_height = h_pprime;
        // TODO: remove synchronization when VWARP_WARP_SIZE <= 32
        __threadfence();
        if (height[neighbor_id] == *lowest_height) {
          *best_edge_id = i;
        }
      }
    }
  } // for
}


/**
 * Implements the push relabel kernel, as per [Hong08]. Modified to employ the
 * virtual warp technique.
 */
__global__
void vwarp_push_relabel_kernel(graph_t graph, weight_t* flow, weight_t* excess,
                               uint32_t* height, id_t* reverse_indices,
                               id_t source_id, id_t sink_id, bool* finished,
                               uint32_t thread_count) {
  const id_t thread_id = THREAD_GLOBAL_INDEX;
  if (thread_id >= thread_count) return;

  int warp_offset = thread_id % VWARP_WARP_SIZE;
  int warp_id     = thread_id / VWARP_WARP_SIZE;

  __shared__ vwarp_mem_t shared_memory[(MAX_THREADS_PER_BLOCK /
                                        VWARP_WARP_SIZE)];
  __shared__ id_t best_edge_ids[(MAX_THREADS_PER_BLOCK / VWARP_WARP_SIZE)];
  __shared__ uint32_t lowest_heights[(MAX_THREADS_PER_BLOCK / VWARP_WARP_SIZE)];
  vwarp_mem_t* my_space = shared_memory + (THREAD_GRID_INDEX / VWARP_WARP_SIZE);

  // copy my work to local space
  int v_ = warp_id * VWARP_BATCH_SIZE;
  vwarp_memcpy(my_space->height, &(height[v_]), VWARP_BATCH_SIZE, warp_offset);
  vwarp_memcpy(my_space->vertices, &(graph.vertices[v_]), VWARP_BATCH_SIZE + 1,
               warp_offset);

  int count = KERNEL_CYCLES;
  while(count--) {
    // iterate over my work
    for(uint32_t v = 0; v < VWARP_BATCH_SIZE; v++) {
      id_t vertex_id = v_ + v;
      if (excess[vertex_id] > 0 && (vertex_id != sink_id) &&
          my_space->height[v] < graph.vertex_count) {
        id_t* best_edge_id = &(best_edge_ids[(THREAD_GRID_INDEX /
                                             VWARP_WARP_SIZE)]);
        uint32_t* lowest_height = &(lowest_heights[(THREAD_GRID_INDEX /
                                                    VWARP_WARP_SIZE)]);
        *best_edge_id = INFINITE;
        *lowest_height = INFINITE;
        // TODO: remove synchronization when VWARP_WARP_SIZE <= 32
        __threadfence();

        id_t* edges = &(graph.edges[my_space->vertices[v]]);
        weight_t* weights = &(graph.weights[my_space->vertices[v]]);
        weight_t* flows = &(flow[my_space->vertices[v]]);

        // Find the lowest neighbor connected by a residual edge
        int neighbor_count = my_space->vertices[v + 1] - my_space->vertices[v];
        vwarp_process_neighbors(warp_offset, warp_id, neighbor_count, edges,
                                flows, weights, height, lowest_height,
                                best_edge_id);
        // TODO: remove synchronization when VWARP_WARP_SIZE <= 32
        __threadfence();

        // Only one thread does this per vertex
        if (warp_offset == 0) {
          id_t edge = my_space->vertices[v] + *best_edge_id;
          // If a push applies
          if (height[vertex_id] > *lowest_height && *best_edge_id != INFINITE &&
              (graph.weights[edge] != flow[edge])) {
            weight_t push_amt = min(excess[vertex_id],
                                    graph.weights[edge] - flow[edge]);
            atomicAdd(&flow[edge], push_amt);
            atomicAdd(&flow[reverse_indices[edge]], -push_amt);
            atomicAdd(&excess[vertex_id], -push_amt);
            atomicAdd(&excess[graph.edges[edge]], push_amt);
            *finished = false;
          }
          // Otherwise perform a relabel
          else if (*lowest_height != INFINITE) {
            height[vertex_id] = *lowest_height + 1;
            *finished = false;
          }
        }
      }
    } // for
  } // while
}


/**
 * A common finalize function for GPU implementations. It allocates the host
 * output buffer, moves the final results from GPU to the host buffers and
 * frees up some resources.
 */
PRIVATE
error_t finalize_gpu(graph_t* graph_d, weight_t* flow_d, weight_t* excess_d,
                     uint32_t* height_d, id_t* reverse_indices_d,
                     bool* finished_d, weight_t* flow_ret, id_t sink_id) {
  CHK_CU_SUCCESS(hipMemcpy(flow_ret, (weight_t*)&(excess_d[sink_id]),
                            sizeof(weight_t), hipMemcpyDeviceToHost), err);
  graph_finalize_device(graph_d);
  hipFree(flow_d);
  hipFree(excess_d);
  hipFree(height_d);
  hipFree(reverse_indices_d);
  hipFree(finished_d);
  return SUCCESS;
 err:
  return FAILURE;
}


/**
 * GPU implementation of the Push-Relabel algorithm, as described in [Hong08],
 * implementing the virtual warping technique.
 */
__host__
error_t maxflow_vwarp_gpu(graph_t* graph, id_t source_id, id_t sink_id,
                          weight_t* flow_ret) {
  error_t rc = check_special_cases(graph, source_id, sink_id);
  if (rc != SUCCESS) return rc;

  // Setup reverse edges. This creates a new graph and updates the graph
  // pointer to point to this new graph. Thus, we have to do this step before
  // any other allocations/initialization.
  id_t* reverse_indices = NULL;
  graph_t* local_graph = graph_create_bidirectional(graph, &reverse_indices);

  uint32_t* height = (uint32_t*)mem_alloc(local_graph->vertex_count *
                                          sizeof(uint32_t));
  weight_t* excess = (weight_t*)mem_alloc(local_graph->vertex_count *
                                         sizeof(weight_t));
  weight_t* flow = (weight_t*)mem_alloc(local_graph->edge_count *
                                        sizeof(weight_t));

  // Create and initialize state on GPU
  uint64_t vwarp_length = VWARP_BATCH_SIZE *
                          VWARP_BATCH_COUNT(graph->vertex_count);
  graph_t* graph_d;
  weight_t* flow_d;
  weight_t* excess_d;
  uint32_t* height_d;
  id_t* reverse_indices_d;
  bool* finished_d;
  CHK_SUCCESS(initialize_gpu(local_graph, source_id, vwarp_length,
                             reverse_indices, &graph_d, &flow_d, &excess_d,
                             &height_d, &reverse_indices_d, &finished_d),
              err_free_all);

  // {} used to limit scope and avoid problems with error handles.
  {
  dim3 blocks;
  dim3 threads_per_block;
  KERNEL_CONFIGURE(local_graph->edge_count, blocks, threads_per_block);
  memset_device<<<blocks, threads_per_block>>>(flow_d, (weight_t)0,
                                               local_graph->edge_count);

  uint32_t thread_count = VWARP_WARP_SIZE *
                          VWARP_BATCH_COUNT(graph->vertex_count);
  KERNEL_CONFIGURE(thread_count, blocks, threads_per_block);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(vwarp_push_relabel_kernel), hipFuncCachePreferShared);
  bool finished = false;
  // While there exists an applicable push or relabel operation, perform it
  while (!finished) {
    CHK_CU_SUCCESS(hipMemset(finished_d, true, sizeof(bool)), err_free_all);
    // Perform push-relabel on each vertex, according to [Hong08]
    vwarp_push_relabel_kernel<<<blocks, threads_per_block>>>
      (*graph_d, flow_d, excess_d, height_d, reverse_indices_d, source_id,
       sink_id, finished_d, thread_count);
    CHK_CU_SUCCESS(hipGetLastError(), err_free_all);
    CHK_CU_SUCCESS(hipMemcpy(&finished, finished_d, sizeof(bool),
                              hipMemcpyDeviceToHost), err_free_all);
    // TODO (Greg): Copy heights back to main memory and perform the global
    //              relabel operation on the CPU side
  }}

  // We are done, get the results back and clean up state
  CHK_SUCCESS(finalize_gpu(graph_d, flow_d, excess_d, height_d,
                           reverse_indices_d, finished_d, flow_ret, sink_id),
              err_free_all);
  mem_free(height);
  mem_free(excess);
  mem_free(flow);
  mem_free(reverse_indices);
  graph_finalize(local_graph);

  return SUCCESS;

  // error handlers
  err_free_all:
    hipFree(finished_d);
    hipFree(height_d);
    hipFree(excess_d);
    hipFree(flow_d);
    hipFree(reverse_indices_d);
    graph_finalize_device(graph_d);
    mem_free(height);
    mem_free(excess);
    mem_free(flow);
    mem_free(reverse_indices);
    graph_finalize(local_graph);
    return FAILURE;
}


/**
 * GPU implementation of the Push-Relabel algorithm, as described in [Hong08]
 */
__host__
error_t maxflow_gpu(graph_t* graph, id_t source_id, id_t sink_id,
                    weight_t* flow_ret) {
  error_t rc = check_special_cases(graph, source_id, sink_id);
  if (rc != SUCCESS) return rc;

  // Setup reverse edges. This creates a new graph and updates the graph
  // pointer to point to this new graph. Thus, we have to do this step before
  // any other allocations/initialization.
  id_t* reverse_indices = NULL;
  graph_t* local_graph = graph_create_bidirectional(graph, &reverse_indices);

  uint32_t* height = (uint32_t*)mem_alloc(local_graph->vertex_count *
                                          sizeof(uint32_t));
  weight_t* excess = (weight_t*)mem_alloc(local_graph->vertex_count *
                                         sizeof(weight_t));
  weight_t* flow = (weight_t*)mem_alloc(local_graph->edge_count *
                                        sizeof(weight_t));

  // Create and initialize state on GPU
  graph_t* graph_d;
  weight_t* flow_d;
  weight_t* excess_d;
  uint32_t* height_d;
  id_t* reverse_indices_d;
  bool* finished_d;
  CHK_SUCCESS(initialize_gpu(local_graph, source_id, graph->vertex_count,
                             reverse_indices, &graph_d, &flow_d, &excess_d,
                             &height_d, &reverse_indices_d, &finished_d),
              err_free_all);

  // {} used to limit scope and avoid problems with error handles.
  {
  dim3 blocks;
  dim3 threads_per_block;
  KERNEL_CONFIGURE(local_graph->edge_count, blocks, threads_per_block);
  memset_device<<<blocks, threads_per_block>>>(flow_d, (weight_t)0,
                                               local_graph->edge_count);

  // While there exists an applicable push or relabel operation, perform it
  bool finished = false;
  KERNEL_CONFIGURE(local_graph->vertex_count, blocks, threads_per_block);
  while (!finished) {
    CHK_CU_SUCCESS(hipMemset(finished_d, true, sizeof(bool)), err_free_all);
    // Perform push-relabel on each vertex, according to [Hong08]
    push_relabel_kernel<<<blocks, threads_per_block>>>
      (*graph_d, flow_d, excess_d, height_d, reverse_indices_d, source_id,
       sink_id, finished_d);
    CHK_CU_SUCCESS(hipGetLastError(), err_free_all);
    CHK_CU_SUCCESS(hipMemcpy(&finished, finished_d, sizeof(bool),
                              hipMemcpyDeviceToHost), err_free_all);
    // TODO (Greg): Copy heights back to main memory and perform the global
    //              relabel operation on the CPU side
  }}

  // We are done, get the results back and clean up state
  CHK_SUCCESS(finalize_gpu(graph_d, flow_d, excess_d, height_d,
                           reverse_indices_d, finished_d, flow_ret, sink_id),
              err_free_all);
  mem_free(height);
  mem_free(excess);
  mem_free(flow);
  mem_free(reverse_indices);
  graph_finalize(local_graph);

  return SUCCESS;

  // error handlers
  err_free_all:
    hipFree(finished_d);
    hipFree(height_d);
    hipFree(excess_d);
    hipFree(flow_d);
    hipFree(reverse_indices_d);
    graph_finalize_device(graph_d);
    mem_free(height);
    mem_free(excess);
    mem_free(flow);
    mem_free(reverse_indices);
    graph_finalize(local_graph);
    return FAILURE;
}


/**
 * CPU Push-relabel operation
 * On a particular vertex u, attempt a push operation along any of its edges.
 * If the push operation fails, perform a relabel.
 */
PRIVATE
void push_relabel_cpu(graph_t* graph, id_t u, id_t source_id, id_t sink_id,
                      weight_t* flow, weight_t* excess, uint32_t* height,
                      id_t* reverse_indices, bool* finished) {
  if (excess[u] <= 0 || height[u] >= graph->vertex_count) return;

  weight_t e_prime = excess[u];
  uint32_t h_prime = INFINITE;
  id_t best_edge_id = INFINITE;

  // Find the lowest neighbor connected by a residual edge
  for (id_t edge_id = graph->vertices[u]; edge_id < graph->vertices[u + 1];
       edge_id++) {
    if (graph->weights[edge_id] <= flow[edge_id]) continue;
    uint32_t h_pprime = height[graph->edges[edge_id]];
    if (h_pprime < h_prime) {
      best_edge_id = edge_id;
      h_prime = h_pprime;
    }
  }

  // If a push applies
  if (height[u] > h_prime) {
    weight_t push_amt = min(e_prime, graph->weights[best_edge_id] -
                            flow[best_edge_id]);
    __sync_fetch_and_add_float(&flow[best_edge_id], push_amt);
    __sync_fetch_and_add_float(&flow[reverse_indices[best_edge_id]], -push_amt);
    __sync_fetch_and_add_float(&excess[u], -push_amt);
    __sync_fetch_and_add_float(&excess[graph->edges[best_edge_id]], push_amt);
    *finished = false;
  }
  // Otherwise perform a relabel
  else if (h_prime != INFINITE) {
    height[u] = h_prime + 1;
    *finished = false;
  }
}


error_t maxflow_cpu(graph_t* graph, id_t source_id, id_t sink_id,
                    weight_t* flow_ret) {
  error_t rc = check_special_cases(graph, source_id, sink_id);
  if (rc != SUCCESS) return rc;

  // Setup residual edges. This creates a new graph and updates the graph
  // pointer to point to this new graph. Thus, we have to do this step before
  // any other allocations/initialization.
  id_t* reverse_indices = NULL;
  graph_t* local_graph = graph_create_bidirectional(graph, &reverse_indices);

  weight_t* excess = (weight_t*)mem_alloc(local_graph->vertex_count *
                                          sizeof(weight_t));
  uint32_t* height = (uint32_t*)mem_alloc(local_graph->vertex_count *
                                          sizeof(uint32_t));
  weight_t* flow = (weight_t*)mem_alloc(local_graph->edge_count *
                                        sizeof(weight_t));

  // Initialize flows, height, and excess to 0
  memset(excess, 0, local_graph->vertex_count * sizeof(weight_t));
  memset(height, 0, local_graph->vertex_count * sizeof(uint32_t));
  memset(flow, 0, local_graph->edge_count * sizeof(weight_t));
  // Initialize source's height to the vertex count
  height[source_id] = (uint32_t) local_graph->vertex_count;

  // Initialize preflow
  for (id_t edge_id = local_graph->vertices[source_id];
       edge_id < local_graph->vertices[source_id + 1]; edge_id++) {
    // Don't setup preflow on residual edges
    if (local_graph->weights[edge_id] == 0) continue;
    flow[edge_id] = local_graph->weights[edge_id];
    flow[reverse_indices[edge_id]] = -local_graph->weights[edge_id];
    excess[local_graph->edges[edge_id]] = local_graph->weights[edge_id];
    excess[source_id] -= local_graph->weights[edge_id];
  }

  // While there exists an applicable push or relabel operation, perform it
  bool finished = false;
  while (!finished) {
    finished = true;
    int count = KERNEL_CYCLES;
    while(count--) {
      #ifdef _OPENMP
      #pragma omp parallel for
      #endif // _OPENMP
      for (id_t u = 0; u < local_graph->vertex_count; u++) {
        if (u == sink_id || u == source_id) continue;
        // Perform a push/relabel operation
        push_relabel_cpu(local_graph, u, source_id, sink_id, flow, excess,
                         height, reverse_indices, &finished);
      }
    }
  }

  // The final flow is the sum of all flows into the sink (ie, the excess
  // value at the sink node)
  *flow_ret = excess[sink_id];

  mem_free(reverse_indices);
  mem_free(height);
  mem_free(excess);
  mem_free(flow);
  // Free our modified new graph
  graph_finalize(local_graph);

  return SUCCESS;
}
