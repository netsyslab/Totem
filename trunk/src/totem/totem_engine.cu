#include "hip/hip_runtime.h"
/**
 * Implements the core execution engine of Totem
 *
 *  Created on: 2012-02-02
 *  Author: Abdullah Gharaibeh
 */

#include "totem_engine.cuh"

engine_context_t context = {false, NULL, NULL, 0, ENGINE_DEFAULT_CONFIG, 
                            0, 0, 0, 0};

/**
 * Clears allocated state
 */
PRIVATE void engine_finalize() {
  assert(context.pset && context.par_labels);
  if (context.config.finalize_func) {
    context.config.finalize_func(&context.pset->partitions[0]);
    for (int pid = 1; pid < context.pset->partition_count; pid++) {
      CALL_CU_SAFE(hipSetDevice(context.pset->partitions[pid].processor.id));
      context.config.finalize_func(&context.pset->partitions[pid]);
    }
  }
  CALL_SAFE(partition_set_finalize(context.pset));
  free(context.par_labels);
  free(context.finished);
  context.initialized = false;
}

/**
 * Returns true if all partitions reported a finished state
 */
inline PRIVATE bool superstep_check_finished() {
  bool finished = true;
  for (int pid = 0; pid < context.pset->partition_count; pid++) {
    finished &= context.finished[pid];
  }
  return finished;
}

/**
 * Blocks until all kernels initiated by the client have finished.
 */
inline PRIVATE void superstep_compute_synchronize() {
  for (int pid = 0; pid < context.pset->partition_count; pid++) {
    partition_t* par = &context.pset->partitions[pid];
    if (par->processor.type == PROCESSOR_CPU) continue;
    CALL_CU_SAFE(hipStreamSynchronize(par->streams[1]));
  }
}

/**
 * Launches the compute kernel on each partition
 */
inline PRIVATE void superstep_compute() {
  stopwatch_t stopwatch;
  stopwatch_start(&stopwatch);
  // The assumption is that the first partition is the CPU one, and the
  // rest are GPU ones. This is guaranteed by engine_init.
  for (int pid = 1; pid < context.pset->partition_count; pid++) {
    // The kernel for GPU partitions is supposed not to block. The client is 
    // supposedly invoking the GPU kernel asynchronously, and using the compute 
    // "stream" available for each partition
    partition_t* partition = &context.pset->partitions[pid];
    CALL_CU_SAFE(hipSetDevice(partition->processor.id));
    context.config.kernel_func(partition);
  }
  partition_t* partition = &context.pset->partitions[0];
  context.config.kernel_func(partition);
  superstep_compute_synchronize();
  context.time_comp += stopwatch_elapsed(&stopwatch);
}

/**
 * Triggers grooves to synchronize state across partitions
 */
inline PRIVATE void superstep_communicate() {
  stopwatch_t stopwatch;
  stopwatch_start(&stopwatch);
  grooves_launch_communications(context.pset);
  grooves_synchronize(context.pset);
  if (!context.config.scatter_func) return;
  // The assumption is that the first partition is the CPU one, and the
  // rest are GPU ones. This is guaranteed by engine_init.
  for (int pid = 1; pid < context.pset->partition_count; pid++) {
    partition_t* partition = &context.pset->partitions[pid];
    CALL_CU_SAFE(hipSetDevice(partition->processor.id));
    context.config.scatter_func(partition);
  }
  partition_t* partition = &context.pset->partitions[0];
  context.config.scatter_func(partition);  
  context.time_comm += stopwatch_elapsed(&stopwatch);
}

/**
 * Prepares state for the next superstep
 */
inline PRIVATE void superstep_next() {
  context.superstep++;
  memset(context.finished, 0, context.pset->partition_count * sizeof(bool));
}

PRIVATE void engine_aggregate() {
  if (context.config.aggr_func) {
    context.config.aggr_func(&context.pset->partitions[0]);
    for (int pid = 1; pid < context.pset->partition_count; pid++) {
      CALL_CU_SAFE(hipSetDevice(context.pset->partitions[pid].processor.id));
      context.config.aggr_func(&context.pset->partitions[pid]);
    }
  }
}

error_t engine_execute() {
  stopwatch_t stopwatch;
  stopwatch_start(&stopwatch);
  while (true) {
    superstep_next();                      // prepare state for the next round
    superstep_compute();                   // compute phase
    if (superstep_check_finished()) break; // check for termination
    superstep_communicate();               // communication/synchronize phase
  }
  engine_aggregate();
  context.time_exec = stopwatch_elapsed(&stopwatch);

  engine_finalize();
  return SUCCESS;
}

error_t engine_init(engine_config_t* config) {
  stopwatch_t stopwatch;
  stopwatch_start(&stopwatch);
  if (context.initialized) return FAILURE;
  memset(&context, 0, sizeof(engine_context_t));
  context.config = *config;

  int pcount;
  CALL_CU_SAFE(hipGetDeviceCount(&pcount));
  pcount += 1;
  processor_t* processors = (processor_t*)calloc(pcount, sizeof(processor_t));
  assert(processors);
  processors[0].type = PROCESSOR_CPU;
  for (int gpu_id = 0; gpu_id < pcount - 1; gpu_id++) {
    processors[gpu_id + 1].type = PROCESSOR_GPU;
    processors[gpu_id + 1].id = gpu_id;
  }

  // partition the graph
  stopwatch_t stopwatch_par;
  stopwatch_start(&stopwatch_par);
  switch (config->par_algo) {
    case PAR_RANDOM:
      CALL_SAFE(partition_random(config->graph, (uint32_t)pcount, 13, 
                                 &(context.par_labels)));
      break;
    default:
      // TODO(abdullah): Use Lauro's logging library.
      printf("ERROR: Undefined partition algorithm.\n"); fflush(stdout);
      assert(false);
  }
  context.time_par = stopwatch_elapsed(&stopwatch_par);
  CALL_SAFE(partition_set_initialize(config->graph, context.par_labels,
                                     processors, pcount, config->msg_size, 
                                     &context.pset));
  free(processors);

  // callback the per-partition initialization function
  if (context.config.init_func) {
    context.config.init_func(&context.pset->partitions[0]);
    for (int pid = 1; pid < context.pset->partition_count; pid++) {
      partition_t* par = &context.pset->partitions[pid];
      CALL_CU_SAFE(hipSetDevice(par->processor.id));
      context.config.init_func(par);
    }
  }

  // get largest gpu graph
  uint64_t largest = 0;
  for (int pid = 1; pid < context.pset->partition_count; pid++) {
    uint64_t vcount = context.pset->partitions[pid].subgraph.vertex_count;
    largest = vcount > largest ? vcount : largest;
  }
  context.largest_gpu_par = largest;

  context.finished = (bool*)calloc(pcount, sizeof(bool));
  context.initialized = true;
  context.time_init = stopwatch_elapsed(&stopwatch);
  return SUCCESS;
}
