#include "hip/hip_runtime.h"
/**
 * Implements the core execution engine of Totem
 *
 *  Created on: 2012-02-02
 *  Author: Abdullah Gharaibeh
 */

#include "totem_engine.h"

/**
 * defines the execution context of the engine
 */
typedef struct engine_context_s {
  bool             initialized;
  partition_set_t* pset;
  id_t*            par_labels;
  uint32_t         superstep;
  engine_config_t  config;
  bool*            finished;
} engine_context_t;
PRIVATE engine_context_t context = {false, NULL, NULL, 0, 
                                    ENGINE_DEFAULT_CONFIG};

/**
 * Clears allocated state
 */
PRIVATE void engine_finalize() {
  assert(context.pset && context.par_labels);
  if (context.config.finalize_func) {
    context.config.finalize_func(&context.pset->partitions[0]);
    for (int pid = 1; pid < context.pset->partition_count; pid++) {
      CALL_CU_SAFE(hipSetDevice(context.pset->partitions[pid].processor.id));
      context.config.finalize_func(&context.pset->partitions[pid]);
    }
  }
  CALL_SAFE(partition_set_finalize(context.pset));
  free(context.par_labels);
  free(context.finished);
  // reset the state
  memset(&context, 0, sizeof(engine_context_t));
  // g++ does not allow reinitializing a struct with a predefined value, it 
  // only allows copying from another instance
  engine_config_t config = ENGINE_DEFAULT_CONFIG;
  context.config = config;
}

/**
 * Returns true if all partitions reported a finished state
 */
inline PRIVATE bool superstep_check_finished() {
  bool finished = true;
  for (int pid = 0; pid < context.pset->partition_count; pid++) {
    finished &= context.finished[pid];
  }
  return finished;
}

/**
 * Launches the compute kernel on each partition
 */
inline PRIVATE void superstep_compute() {
  // The assumption is that the first partition is the CPU one, and the
  // rest are GPU ones. This is guaranteed by engine_init.
  for (int pid = 1; pid < context.pset->partition_count; pid++) {
    // The kernel for GPU partitions is supposed not to block. The client is 
    // supposedly invoking the GPU kernel asynchronously, and using the compute 
    // "stream" available for each partition
    partition_t* partition = &context.pset->partitions[pid];
    CALL_CU_SAFE(hipSetDevice(partition->processor.id));
    context.config.kernel_func(partition);
  }
  partition_t* partition = &context.pset->partitions[0];
  context.config.kernel_func(partition);
}

/**
 * Triggers grooves to synchronize state across partitions
 */
inline PRIVATE void superstep_communicate() {
  grooves_launch_communications(context.pset);
  grooves_synchronize(context.pset);
  if (!context.config.scatter_func) return;
  // The assumption is that the first partition is the CPU one, and the
  // rest are GPU ones. This is guaranteed by engine_init.
  for (int pid = 1; pid < context.pset->partition_count; pid++) {
    partition_t* partition = &context.pset->partitions[pid];
    CALL_CU_SAFE(hipSetDevice(partition->processor.id));
    context.config.scatter_func(partition);
  }
  partition_t* partition = &context.pset->partitions[0];
  context.config.scatter_func(partition);  
}

/**
 * Prepares state for the next superstep
 */
inline PRIVATE void superstep_next() {
  context.superstep++;
  memset(context.finished, 0, context.pset->partition_count * sizeof(bool));
}

PRIVATE void engine_aggregate() {
  if (context.config.aggr_func) {
    context.config.aggr_func(&context.pset->partitions[0]);
    for (int pid = 1; pid < context.pset->partition_count; pid++) {
      CALL_CU_SAFE(hipSetDevice(context.pset->partitions[pid].processor.id));
      context.config.aggr_func(&context.pset->partitions[pid]);
    }
  }
}

error_t engine_start() {
  while (true) {
    superstep_compute();                   // compute phase
    if (superstep_check_finished()) break; // check for termination   
    superstep_communicate();               // communication/synchronize phase
    superstep_next();                      // prepare state for the next round
  }

  engine_aggregate();  
  engine_finalize();
  return SUCCESS;
}

error_t engine_init(engine_config_t* config) {
  if (context.initialized) return FAILURE;
  assert(!context.pset && !context.par_labels && !context.finished);
  context.config = *config;

  int pcount;
  CALL_CU_SAFE(hipGetDeviceCount(&pcount));
  pcount += 1;
  processor_t* processors = (processor_t*)calloc(pcount, sizeof(processor_t));
  assert(processors);
  processors[0].type = PROCESSOR_CPU;
  for (int gpu_id = 0; gpu_id < pcount - 1; gpu_id++) {
    processors[gpu_id + 1].type = PROCESSOR_GPU;
    processors[gpu_id + 1].id = gpu_id;
  }

  // partition the graph
  switch (config->par_algo) {
    case PAR_RANDOM:
      CALL_SAFE(partition_random(config->graph, (uint32_t)pcount, 13, 
                                 &(context.par_labels)));
      break;
    default:
      // TODO(abdullah): Use Lauro's logging library.
      printf("ERROR: Undefined partition algorithm.\n"); fflush(stdout);
      assert(false);
  }
  CALL_SAFE(partition_set_initialize(config->graph, context.par_labels,
                                     processors, pcount, config->msg_size, 
                                     &context.pset));
  free(processors);

  // callback the per-partition initialization function
  if (context.config.init_func) {
    context.config.init_func(&context.pset->partitions[0]);
    for (int pid = 1; pid < context.pset->partition_count; pid++) {
      CALL_CU_SAFE(hipSetDevice(context.pset->partitions[pid].processor.id));
      context.config.init_func(&context.pset->partitions[pid]);
    }
  }
  context.finished = (bool*)calloc(pcount, sizeof(bool));
  context.initialized = true;
  return SUCCESS;
}

uint32_t engine_partition_count() {
  assert(context.pset);
  return context.pset->partition_count;
}

uint32_t engine_superstep() {
  assert(context.pset);
  return context.superstep;
}

uint32_t engine_vertex_count() {
  assert(context.pset);
  return context.pset->graph->vertex_count;
}

uint32_t engine_edge_count() {
  assert(context.pset);
  return context.pset->graph->edge_count;
}

void engine_report_finished(uint32_t pid) {
  assert(pid < context.pset->partition_count);
  context.finished[pid] = true;
}
