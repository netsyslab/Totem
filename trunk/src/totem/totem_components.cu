
/* TODO(lauro,abdullah,elizeu): Add license.
 *
 * Implements an algorithm to identify the weakly connected components of a 
 * graph. The algorithm is based on BFS.
 *
 *  Created on: 2011-11-23
 *      Author: Abdullah Gharaibeh
 */

#include "totem_comdef.h"
#include "totem_graph.h"
#include "totem_mem.h"

/**
 * Checks for input parameters and special cases. This is invoked at the 
 * beginning of public interface
*/
PRIVATE
error_t check_special_cases(graph_t* graph, component_set_t** comp_set_ret, 
                            bool* finished) {
  
#define COMP_ALLOC(graph, comp_count, comp_set)                         \
  do {                                                                  \
    (comp_set) = (component_set_t*)calloc(1, sizeof(component_set_t));  \
    (comp_set)->graph = graph;                                          \
    (comp_set)->marker = (id_t*)malloc((graph)->vertex_count * sizeof(id_t)); \
    memset((comp_set)->marker, -1, (graph)->vertex_count * sizeof(id_t)); \
    (comp_set)->vertex_count = (id_t*)calloc((comp_count), sizeof(id_t)); \
    (comp_set)->edge_count = (id_t*)calloc((comp_count), sizeof(id_t)); \
    (comp_set)->count = comp_count;                                     \
    (comp_set)->biggest = 0;                                            \
  } while(0)
  
  *finished = true;
  if (graph == NULL) {
    return FAILURE;
  } else if (graph->vertex_count == 0) {
    return FAILURE;
  } else if (graph->vertex_count > 0 && graph->edge_count == 0) {
    COMP_ALLOC(graph, graph->vertex_count, *comp_set_ret);
    for (id_t vid = 0; vid < graph->vertex_count; vid++) {
      (*comp_set_ret)->marker[vid] = vid;
      (*comp_set_ret)->vertex_count[vid] = 1;
      (*comp_set_ret)->edge_count[vid] = 0;
    }
    return SUCCESS;
  }
  *finished = false;
  return SUCCESS;
}


/**
 * performs BFS traversal starting from src and marks all visited nodes 
 * with component id comp.
 * @param[in] graph
 * @param[in] src the vertex at which BFS starts the traversal
 * @param[in] marker vertices visited will be marked with comp
 * @param[in] comp the id of the current component
 */
PRIVATE void mark_component(const graph_t* graph, id_t src, id_t* marker, 
                            int comp) {

  // TODO(abdullah): use bfs_* functions implemented in totem_bfs.cu to minimize
  // code maintenance overhead. The difference between this bfs-like  
  // iplementation and the ones in totem_bfs.cu is that this one marks the 
  // vertices with their component id on the fly which has the potential to 
  // improve performance in the case of graphs with large number of components. 
  // Also, it assumes that all the vertices less than src has already been 
  // visited, hence it skips iterating over them. An advantage of using the 
  // bfs_* functions is modularity. One way to enable such a thing in the 
  // original bfs implementation is to have callbacks in them.

  assert(graph && (src < graph->vertex_count) && (marker[src] == (id_t)-1));

  marker[src] = comp;
  // single vertex component
  if ((graph->vertices[src + 1] - graph->vertices[src]) == 0) {
    return;
  }

  // while the current level has vertices to be processed
  bool finished = false;
  for (int level = comp; !finished; level++) {
    finished = true;
    #ifdef _OPENMP
    #pragma omp parallel for
    #endif // _OPENMP
    for (id_t vid = src; vid < graph->vertex_count; vid++) {
      // the assumption is that all the vertices less than src has alredy been 
      // marked, therefore we can safely skip them and start the loop from src.
      if (marker[vid] != level) continue;
      marker[vid] = comp;
      for (id_t i = graph->vertices[vid]; i < graph->vertices[vid + 1]; i++) {
        const id_t nbr = graph->edges[i];
        if (marker[nbr] == (id_t)-1) {
          finished = false;
          marker[nbr] = level + 1;
        }
      }
    }
  }
}

error_t get_components_cpu(graph_t* graph, component_set_t** comp_set_ret) {

  assert(graph);
  // Check for special cases
  bool finished = false;
  error_t rc = check_special_cases(graph, comp_set_ret, &finished);
  if (finished) return rc;

  component_set_t* comp_set = 
    (component_set_t*)calloc(1, sizeof(component_set_t));
  comp_set->graph = graph;
  comp_set->marker = (id_t*)malloc(graph->vertex_count * sizeof(id_t));
  memset(comp_set->marker, -1, graph->vertex_count * sizeof(id_t));

  int comp_count = 0;
  for (id_t vid = 0; vid < graph->vertex_count; vid++) {
    if (comp_set->marker[vid] == (id_t)-1) {
      mark_component(graph, vid, comp_set->marker, comp_count);
      comp_count++;
    }
  }
  comp_set->count = comp_count;

  // compute the vertex and edge count of each component
  comp_set->vertex_count = (id_t*)calloc(comp_count, sizeof(id_t));
  comp_set->edge_count   = (id_t*)calloc(comp_count, sizeof(id_t));

#ifdef _OPENMP
#pragma omp parallel for
#endif // _OPENMP
  for (id_t vid = 0; vid < graph->vertex_count; vid++) {
    id_t comp = comp_set->marker[vid];
    __sync_fetch_and_add(&(comp_set->vertex_count[comp]), 1);
    id_t nbr_count = graph->vertices[vid + 1] - graph->vertices[vid];
    __sync_fetch_and_add(&(comp_set->edge_count[comp]), nbr_count);    
  }

  // identify the biggest component
  comp_set->biggest = 0;
  for (id_t comp = 1; comp < comp_set->count; comp++) {
    if (comp_set->vertex_count[comp] > 
        comp_set->vertex_count[comp_set->biggest]) {
      comp_set->biggest = comp;
    }    
  }

  *comp_set_ret = comp_set;
  return SUCCESS;
}


error_t finalize_component_set(component_set_t* comp_set) {
  if (!comp_set) return FAILURE;
  if (comp_set->marker) free(comp_set->marker);
  if (comp_set->vertex_count) free(comp_set->vertex_count);
  if (comp_set->edge_count) free(comp_set->edge_count);
  free(comp_set);
  return SUCCESS;
}
