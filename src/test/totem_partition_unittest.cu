#include "hip/hip_runtime.h"
/* TODO(lauro,abdullah,elizeu): Add license.
 *
 * Contains unit tests for partition.
 *
 *  Created on: 2011-12-30
 *      Author: Elizeu Santos-Neto
 *              Abdullah Gharaibeh
 */

// totem includes
#include "totem_common_unittest.h"
#include "totem_comkernel.cuh"
#include "totem_grooves.h"
#include "totem_partition.h"

#if GTEST_HAS_PARAM_TEST

using ::testing::TestWithParam;
using ::testing::Values;

__global__ void VerifyPartitionGPUKernel(partition_t partition, uint32_t pid,
                                         uint32_t pcount) {
  const graph_t* subgraph = &partition.subgraph;
  const vid_t vid = THREAD_GLOBAL_INDEX;
  if (vid >= subgraph->vertex_count) return;
  for (eid_t i = subgraph->vertices[vid];
       i < subgraph->vertices[vid + 1]; i++) {
    vid_t nbr     = subgraph->edges[i];
    int  nbr_pid = GET_PARTITION_ID(nbr);
    KERNEL_EXPECT_TRUE(nbr_pid < pcount);
    if (nbr_pid != pid) {
      KERNEL_EXPECT_TRUE(partition.outbox[nbr_pid].count > 0);
    }
  }
}

__global__ void VerifyPartitionInboxGPUKernel(partition_t partition,
                                              uint32_t pid, uint32_t pcount) {
  const int index = THREAD_GLOBAL_INDEX;
  for (int r = 0; r < pcount; r++) {
    grooves_box_table_t* inbox = &partition.inbox[r];
    if (index >= inbox->count) continue;
    KERNEL_EXPECT_TRUE(inbox->rmt_nbrs[index] <
                       partition.subgraph.vertex_count);
  }
}

__global__ void CheckInboxValuesGPUKernel(uint32_t pid, int* values,
                                          uint32_t count) {
  const int index = THREAD_GLOBAL_INDEX;
  if (index >= count) return;
  KERNEL_EXPECT_TRUE(values[index] == pid + 1);
}

class GraphPartitionTest : public TestWithParam<partition_func_t> {
 public:
  virtual void SetUp() {
    // Ensure the minimum CUDA architecture is supported
    CUDA_CHECK_VERSION();
    partition_func_ = GetParam();
    int gpu_count = get_gpu_count();
    graph_ = NULL;
    partitions_ = NULL;
    partition_set_ = NULL;
    partition_count_ = gpu_count + 1;
    partition_processor_ =
      (processor_t*)calloc(partition_count_, sizeof(processor_t));
    partition_processor_[0].type = PROCESSOR_CPU;
    for (int gpu = 0; gpu < gpu_count; gpu++) {
      partition_processor_[gpu + 1].type = PROCESSOR_GPU;
      partition_processor_[gpu + 1].id = gpu;
    }
  }

  virtual void TearDown() {
    free(partition_processor_);
    if (graph_ != NULL) {
      graph_finalize(graph_);
    }
    if (partitions_ != NULL) {
      free(partitions_);
    }
    if (partition_set_ != NULL) {
      EXPECT_EQ(SUCCESS, partition_set_finalize(partition_set_));
    }
  }

  void VerifyPartitionGPU(uint32_t pid) {
    ASSERT_EQ(hipSuccess,
              hipSetDevice(partition_set_->partitions[pid].processor.id));
    dim3 blocks, threads_per_block;
    KERNEL_CONFIGURE(partition_set_->partitions[pid].subgraph.vertex_count,
                     blocks, threads_per_block);
    VerifyPartitionGPUKernel<<<blocks,
      threads_per_block>>>(partition_set_->partitions[pid], pid,
                           partition_set_->partition_count);
    ASSERT_EQ(hipSuccess, hipGetLastError());
    ASSERT_EQ(hipSuccess, hipDeviceSynchronize());

    VerifyPartitionInboxGPUKernel<<<blocks,
      threads_per_block>>>(partition_set_->partitions[pid], pid,
                           partition_set_->partition_count);
    ASSERT_EQ(hipSuccess, hipGetLastError());
    ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
  }

  void VerifyPartitionCPU(uint32_t pid) {
    partition_t* partition = &partition_set_->partitions[pid];
    graph_t* subgraph = &partition->subgraph;
    uint32_t pcount = partition_set_->partition_count;
    for (vid_t vid = 0; vid < subgraph->vertex_count; vid++) {
      for (eid_t i = subgraph->vertices[vid];
           i < subgraph->vertices[vid + 1]; i++) {
        uint32_t nbr_pid = GET_PARTITION_ID(subgraph->edges[i]);
        EXPECT_TRUE((nbr_pid < pcount));
        partition_t* nbr_partition = &partition_set_->partitions[nbr_pid];
        vid_t nbr_id = GET_VERTEX_ID(subgraph->edges[i]);
        EXPECT_TRUE((nbr_id < nbr_partition->subgraph.vertex_count));
        if (nbr_pid != pid) {
          grooves_box_table_t* outbox =
            &partition->outbox[nbr_pid];
          EXPECT_GT(outbox->count, (uint32_t)0);
        }
      }
    }
    // verify inbox tables, all the vertices in the table must belong to this
    // partition
    for (int r = 0; r < pcount; r++) {
      grooves_box_table_t* inbox = &partition->inbox[r];
      for (int index = 0; index < inbox->count; index++) {
        KERNEL_EXPECT_TRUE(inbox->rmt_nbrs[index] <
                           partition->subgraph.vertex_count);
      }
    }
  }

  void TestState() {
    uint32_t pcount = partition_set_->partition_count;
    for (uint32_t pid = 0; pid < pcount; pid++) {
      partition_t* partition = &partition_set_->partitions[pid];
      if (!partition->subgraph.vertex_count) return;
      EXPECT_EQ(partition_processor_[pid].type, partition->processor.type);
      EXPECT_EQ(partition_processor_[pid].id, partition->processor.id);
      if (partition->processor.type == PROCESSOR_CPU) VerifyPartitionCPU(pid);
      if (partition->processor.type == PROCESSOR_GPU) VerifyPartitionGPU(pid);
    }
  }

  void InitOutboxValues() {
    uint32_t pcount = partition_set_->partition_count;
    for (uint32_t pid = 0; pid < pcount; pid++) {
      partition_t* partition = &partition_set_->partitions[pid];
      if (!partition->subgraph.vertex_count) continue;
      EXPECT_EQ(pid, partition->id);
      for (uint32_t remote_pid = 0; remote_pid < pcount; remote_pid++) {
        if (remote_pid == pid) continue;
        grooves_box_table_t* remote_outbox = &partition->outbox[remote_pid];
        if (remote_outbox->count == 0) continue;
        if (partition->processor.type == PROCESSOR_GPU) {
          ASSERT_EQ(hipSuccess, hipSetDevice(partition->processor.id));
          ASSERT_EQ(SUCCESS, totem_memset((int*)remote_outbox->push_values,
                                          (int)(remote_pid + 1),
                                          remote_outbox->count,
                                          TOTEM_MEM_DEVICE));
          ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
        } else {
          ASSERT_EQ(PROCESSOR_CPU, partition->processor.type);
          int* values = (int*)remote_outbox->push_values;
          for (int i = 0; i < remote_outbox->count; i++) {
            values[i] = remote_pid + 1;
          }
        }
      }
    }
  }

  void CheckInboxValues() {
    for (uint32_t pid = 0; pid < partition_set_->partition_count; pid++) {
      partition_t* partition = &partition_set_->partitions[pid];
      if (!partition->subgraph.vertex_count) return;
      grooves_box_table_t* inbox = partition->inbox;
      uint32_t bcount = partition_set_->partition_count;
      for (uint32_t bindex = 0; bindex < bcount; bindex++) {
        if (inbox[bindex].count == 0 || bindex == pid) continue;
        int* values = (int*)inbox[bindex].push_values;
        if (partition->processor.type == PROCESSOR_GPU) {
          ASSERT_EQ(hipSuccess, hipSetDevice(partition->processor.id));
          dim3 blocks, threads_per_block;
          KERNEL_CONFIGURE(inbox[bindex].count, blocks, threads_per_block);
          CheckInboxValuesGPUKernel<<<blocks, threads_per_block>>>
            (pid, values, inbox[bindex].count);
          ASSERT_EQ(hipSuccess, hipGetLastError());
          ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
        } else {
          ASSERT_EQ(PROCESSOR_CPU, partition->processor.type);
          for (vid_t v = 0; v < inbox[bindex].count; v++) {
              EXPECT_EQ(pid + 1, values[v]);
          }
        }
      }
    }
  }

  void TestCommunication() {
    InitOutboxValues();
    for (int pid = 0; pid < partition_set_->partition_count; pid++) {
      EXPECT_EQ(SUCCESS, grooves_launch_communications(partition_set_, pid,
                                                       GROOVES_PUSH));
    }
    EXPECT_EQ(SUCCESS, grooves_synchronize(partition_set_, GROOVES_PUSH));
    CheckInboxValues();
  }

  void TestPartitionedGraphDataStructure() {
    totem_attr_t attr = TOTEM_DEFAULT_ATTR;
    EXPECT_EQ(SUCCESS, partition_func_(graph_, partition_count_, NULL,
                                       &partitions_, &attr));
    EXPECT_TRUE(partition_count_ <= MAX_PARTITION_COUNT);
    EXPECT_EQ(SUCCESS, partition_set_initialize(graph_, partitions_,
                                                partition_processor_,
                                                partition_count_,
                                                &attr, &partition_set_));
    TestState();
    TestCommunication();
  }

 protected:
  partition_func_t  partition_func_;
  graph_t*           graph_;
  vid_t*             partitions_;
  uint32_t           partition_count_;
  processor_t*       partition_processor_;
  partition_set_t*   partition_set_;
};

TEST_P(GraphPartitionTest , PartitionInvalidPartitionNumber) {
  EXPECT_EQ(SUCCESS, graph_initialize(DATA_FOLDER("single_node.totem"),
                                      false, &graph_));
  totem_attr_t attr = TOTEM_DEFAULT_ATTR;
  EXPECT_EQ(FAILURE, partition_func_(graph_, -1, NULL, &partitions_, &attr));
}

TEST_P(GraphPartitionTest , PartitionFractionInvalidFraction) {
  EXPECT_EQ(SUCCESS, graph_initialize(DATA_FOLDER("single_node.totem"),
                                      false, &graph_));
  double* partition_fraction = (double *) calloc(2, sizeof(double));
  partition_fraction[0] = 2.0;
  partition_fraction[1] = -1.0; // Invalid fraction
  totem_attr_t attr = TOTEM_DEFAULT_ATTR;
  EXPECT_EQ(FAILURE, partition_func_(graph_, 2, partition_fraction,
                                     &partitions_, &attr));
  partition_fraction[0] = 0.8;
  partition_fraction[1] = 0.1; // Invalid fraction sum
  EXPECT_EQ(FAILURE, partition_func_(graph_, 2, partition_fraction,
                                     &partitions_, &attr));
  free(partition_fraction);
}

TEST_P(GraphPartitionTest , PartitionSingleNodeGraph) {
  EXPECT_EQ(SUCCESS, graph_initialize(DATA_FOLDER("single_node.totem"),
                                      false, &graph_));
  totem_attr_t attr = TOTEM_DEFAULT_ATTR;
  EXPECT_EQ(SUCCESS, partition_func_(graph_, 10, NULL, &partitions_, &attr));
  EXPECT_TRUE(partitions_[0] < 10);
}

TEST_P(GraphPartitionTest , PartitionFractionSingleNodeGraph) {
  EXPECT_EQ(SUCCESS, graph_initialize(DATA_FOLDER("single_node.totem"),
                                      false, &graph_));
  double* partition_fraction = (double *) calloc(10, sizeof(double));
  for (int i = 0; i < 10; i++) {
    partition_fraction[i] = (1.0 / 10);
  }
  totem_attr_t attr = TOTEM_DEFAULT_ATTR;
  EXPECT_EQ(SUCCESS, partition_func_(graph_, 10, partition_fraction,
                                     &partitions_, &attr));
  EXPECT_TRUE(partitions_[0] < 10);
  free(partition_fraction);
}

TEST_P(GraphPartitionTest , PartitionChainGraph) {
  EXPECT_EQ(SUCCESS, graph_initialize(DATA_FOLDER("chain_1000_nodes.totem"),
                                      false, &graph_));
  totem_attr_t attr = TOTEM_DEFAULT_ATTR;
  EXPECT_EQ(SUCCESS, partition_func_(graph_, 10, NULL, &partitions_, &attr));
  for (vid_t i = 0; i < graph_->vertex_count; i++) {
    EXPECT_TRUE(partitions_[i] < 10);
  }
}

TEST_P(GraphPartitionTest , PartitionFractionChainGraph) {
  EXPECT_EQ(SUCCESS, graph_initialize(DATA_FOLDER("chain_1000_nodes.totem"),
                                      false, &graph_));
  double* partition_fraction = (double *) calloc(10, sizeof(double));
  for (int i = 0; i < 10; i++) {
    partition_fraction[i] = (1.0 / 10);
  }
  totem_attr_t attr = TOTEM_DEFAULT_ATTR;
  EXPECT_EQ(SUCCESS, partition_func_(graph_, 10, partition_fraction,
                                     &partitions_, &attr));
  for (vid_t i = 0; i < graph_->vertex_count; i++) {
    EXPECT_TRUE(partitions_[i] < 10);
  }
  free(partition_fraction);
}

TEST_P(GraphPartitionTest , GetPartitionsSingleNodeGraph) {
  EXPECT_EQ(SUCCESS, graph_initialize(DATA_FOLDER("single_node.totem"),
                                      false, &graph_));
  partition_count_ = 1;
  totem_attr_t attr = TOTEM_DEFAULT_ATTR;
  EXPECT_EQ(SUCCESS, partition_func_(graph_, partition_count_, NULL,
                                     &partitions_, &attr));
  EXPECT_EQ(SUCCESS, partition_set_initialize(graph_, partitions_,
                                              partition_processor_,
                                              partition_count_, &attr,
                                              &partition_set_));
  EXPECT_EQ(partition_set_->partition_count, 1);
  partition_t* partition = &partition_set_->partitions[0];
  EXPECT_EQ(partition->subgraph.vertex_count, (vid_t)1);
  EXPECT_EQ(partition->subgraph.edge_count, (eid_t)0);
}

TEST_P(GraphPartitionTest, GetPartitionsChainGraph) {
  graph_initialize(DATA_FOLDER("chain_1000_nodes.totem"), false, &graph_);
  TestPartitionedGraphDataStructure();
}

TEST_P(GraphPartitionTest, GetPartitionsStarGraph) {
  graph_initialize(DATA_FOLDER("star_1000_nodes.totem"), false, &graph_);
  TestPartitionedGraphDataStructure();
}

TEST_P(GraphPartitionTest, GetPartitionsCompleteGraph) {
  graph_initialize(DATA_FOLDER("complete_graph_300_nodes.totem"),
                   false, &graph_);
  TestPartitionedGraphDataStructure();
}

TEST_P(GraphPartitionTest, GetPartitionsImbalancedChainGraph) {
  EXPECT_EQ(SUCCESS, graph_initialize(DATA_FOLDER("chain_1000_nodes.totem"),
                                      false, &graph_));
  // set the processor of all partitions to CPU
  for (uint32_t pid = 0; pid < partition_count_; pid++) {
    partition_processor_[pid].type = PROCESSOR_CPU;
  }
  // Divide the graph in two partitions, one node in one partition and the
  // other 999 in the second partition.
  partitions_ = (vid_t*)calloc(1000, sizeof(vid_t));
  partitions_[0] = 1;
  totem_attr_t attr = TOTEM_DEFAULT_ATTR;
  EXPECT_EQ(SUCCESS, partition_set_initialize(graph_, partitions_,
                                              partition_processor_,
                                              partition_count_,
                                              &attr, &partition_set_));
  for (int pid = 0; pid < partition_set_->partition_count; pid++) {
    partition_t* partition = &partition_set_->partitions[pid];
    EXPECT_EQ(pid, partition->id);
    for (vid_t vid = 0; vid < partition->subgraph.vertex_count; vid++) {
      // Only the vertex-0 and vertex-999 in the original graph have a single
      // neighbor. Vertex-0 is in partition-1, and vertex-999 is renamed to 998
      // in partition-0.
      vid_t expected = (pid == 1 || vid == 998 ? 1 : 2);
      EXPECT_EQ(expected,
                partition->subgraph.vertices[vid + 1] -
                partition->subgraph.vertices[vid]);
    }
  }
}

// From Google documentation:
// In order to run value-parameterized tests, we need to instantiate them,
// or bind them to a list of values which will be used as test parameters.
//
// Values() receives a list of parameters and the framework will execute the
// whole set of tests PCoreTest for each element of Values()
INSTANTIATE_TEST_CASE_P(GRAPHPARTITIONTEST, GraphPartitionTest,
                        Values(&partition_random,
                               &partition_by_asc_sorted_degree,
                               &partition_by_dsc_sorted_degree));

#else

// From Google documentation:
// Google Test may not support value-parameterized tests with some
// compilers. This dummy test keeps gtest_main linked in.
TEST_P(DummyTest, ValueParameterizedTestsAreNotSupportedOnThisPlatform) {}

#endif  // GTEST_HAS_PARAM_TEST
