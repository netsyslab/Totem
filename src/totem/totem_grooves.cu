/**
 * Implements the Grooves interface.
 *
 *  Created on: 2012-01-25
 *  Author: Abdullah Gharaibeh
 */

// totem includes
#include "totem_comkernel.cuh"
#include "totem_grooves.h"
#include "totem_mem.h"
#include "totem_partition.h"
#include "totem_util.h"
#include "totem_engine.cuh"

void init_get_rmt_nbrs(partition_set_t* pset, int32_t pid,
                       vid_t** rmt_nbrs, vid_t* count_per_par);

PRIVATE void init_outbox_table(partition_t* partition, uint32_t pcount,
                               vid_t** rmt_nbrs, vid_t* count_per_par,
                               size_t push_msg_size, size_t pull_msg_size) {
  grooves_box_table_t* outbox = partition->outbox;
  uint32_t pid = partition->id;
  for (int rmt_pid = (pid + 1) % pcount; rmt_pid != pid;
       rmt_pid = (rmt_pid + 1) % pcount) {
    outbox[rmt_pid].count = count_per_par[rmt_pid];
    if (outbox[rmt_pid].count) {
      assert(rmt_nbrs[rmt_pid]);
      outbox[rmt_pid].rmt_nbrs = rmt_nbrs[rmt_pid];
      if (partition->processor.type == PROCESSOR_CPU) {
        // Allocate the values array for the cpu-based partitions. The gpu-based
        // partitions will have their values array allocated later when their
        // state is initialized on the gpu
        if (push_msg_size > 0) {
          CALL_SAFE(totem_malloc(bits_to_bytes(outbox[rmt_pid].count *
                                               push_msg_size),
                                 TOTEM_MEM_HOST_PINNED,
                                 &(outbox[rmt_pid].push_values)));
        }
        if (pull_msg_size > 0) {
          CALL_SAFE(totem_malloc(bits_to_bytes(outbox[rmt_pid].count *
                                               pull_msg_size),
                                 TOTEM_MEM_HOST_PINNED,
                                 &(outbox[rmt_pid].pull_values)));
        }
      }
    }
  }
}

PRIVATE void init_outbox(partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  for (int pid = 0; pid < pcount; pid++) {
    partition_t* partition = &pset->partitions[pid];

    if (!partition->subgraph.vertex_count ||
        !partition->subgraph.edge_count) continue;

    // identify the remote nbrs and their count per remote partition
    vid_t* rmt_nbrs[MAX_PARTITION_COUNT];
    vid_t count_per_par[MAX_PARTITION_COUNT];
    init_get_rmt_nbrs(pset, pid, rmt_nbrs, count_per_par);
    // build the outbox
    if (partition->rmt_vertex_count) {
      // build the outbox tables for this partition
      init_outbox_table(partition, pcount, rmt_nbrs, count_per_par,
                        pset->push_msg_size, pset->pull_msg_size);
    }
  }
}

PRIVATE void init_inbox(partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  for (int pid = 0; pid < pcount; pid++) {
    partition_t* partition = &pset->partitions[pid];

    if (!partition->subgraph.vertex_count) continue;

    for (int src_pid = (pid + 1) % pcount; src_pid != pid;
         src_pid = (src_pid + 1) % pcount) {
      partition_t* remote_par = &pset->partitions[src_pid];
      // An inbox in a partition is an outbox in the source partition.
      // Therefore, we just need to copy the state of the already built
      // source partition's outbox into the destination partition's inbox.
      partition->inbox[src_pid] = remote_par->outbox[pid];
      if (remote_par->processor.type == PROCESSOR_GPU) {
        // if the remote processor is GPU, then a values array for this inbox
        // needs to be allocated on the host
        if (pset->push_msg_size > 0) {
          CALL_SAFE(totem_malloc(bits_to_bytes(partition->inbox[src_pid].count *
                                               pset->push_msg_size),
                                 TOTEM_MEM_HOST_PINNED,
                                 &(partition->inbox[src_pid].push_values)));
          CALL_SAFE(totem_malloc(bits_to_bytes(partition->inbox[src_pid].count *
                                               pset->push_msg_size),
                                 TOTEM_MEM_HOST_PINNED,
                                 &(partition->inbox[src_pid].push_values_s)));
        }
        if (pset->pull_msg_size > 0) {
          CALL_SAFE(totem_malloc(bits_to_bytes(partition->inbox[src_pid].count *
                                               pset->pull_msg_size),
                                 TOTEM_MEM_HOST_PINNED,
                                 &(partition->inbox[src_pid].pull_values)));
          CALL_SAFE(totem_malloc(bits_to_bytes(partition->inbox[src_pid].count *
                                               pset->pull_msg_size),
                                 TOTEM_MEM_HOST_PINNED,
                                 &(partition->inbox[src_pid].pull_values_s)));
        }
      }
    }
  }
}

PRIVATE void init_gpu_enable_peer_access(uint32_t pid, partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  partition_t* partition = &pset->partitions[pid];
  for (int remote_pid = (pid + 1) % pcount; remote_pid != pid;
       remote_pid = (remote_pid + 1) % pcount) {
    partition_t* remote_par = &pset->partitions[remote_pid];
    if (remote_par->processor.type == PROCESSOR_GPU &&
        remote_par->processor.id != partition->processor.id) {
      int can_access_peer = 0;
      CALL_CU_SAFE(hipDeviceCanAccessPeer(&can_access_peer,
                                           partition->processor.id,
                                           remote_par->processor.id));
      if (can_access_peer == 1) {
        CALL_CU_SAFE(hipDeviceEnablePeerAccess(remote_par->processor.id, 0));
      }
    }
  }
}

PRIVATE void init_table_gpu(partition_t* par, partition_set_t* pset,
                            bool inbox) {
  // set device context, create the tables for this gpu
  CALL_CU_SAFE(hipSetDevice(par->processor.id));
  grooves_box_table_t* box = par->outbox;
  if (inbox) box = par->inbox;
  // initialize the tables on the gpu
  for (uint32_t rmt_pid = 0; rmt_pid < pset->partition_count; rmt_pid++) {
    if (rmt_pid == par->id) continue;
    vid_t count = box[rmt_pid].count;
    if (count) {
      vid_t* rmt_nbrs = box[rmt_pid].rmt_nbrs;
      CALL_CU_SAFE(hipMalloc(
          reinterpret_cast<void**>(&(box[rmt_pid].rmt_nbrs)),
          count * sizeof(vid_t)));
      CALL_CU_SAFE(hipMemcpy(box[rmt_pid].rmt_nbrs, rmt_nbrs,
                              count * sizeof(vid_t), hipMemcpyDefault));
      if ((pset->partitions[rmt_pid].processor.type == PROCESSOR_GPU) &&
          inbox) {
        free(rmt_nbrs);
      }
      if (pset->push_msg_size > 0) {
        CALL_CU_SAFE(hipMalloc(
            reinterpret_cast<void**>(&(box[rmt_pid].push_values)),
            bits_to_bytes(count * pset->push_msg_size)));
        if (inbox) {
          CALL_CU_SAFE(hipMalloc(
              reinterpret_cast<void**>(&(box[rmt_pid].push_values_s)),
              bits_to_bytes(count * pset->push_msg_size)));
        }
      }
      if (pset->pull_msg_size > 0) {
        CALL_CU_SAFE(hipMalloc(
            reinterpret_cast<void**>(&(box[rmt_pid].pull_values)),
            bits_to_bytes(count * pset->pull_msg_size)));
        if (inbox) {
          CALL_CU_SAFE(hipMalloc(
              reinterpret_cast<void**>(&(box[rmt_pid].pull_values_s)),
              bits_to_bytes(count * pset->pull_msg_size)));
        }
      }
    }
  }
}

PRIVATE void init_gpu_state(partition_set_t* pset) {
  for (int pid = 0; pid < pset->partition_count; pid++) {
    partition_t* partition = &pset->partitions[pid];
    if (partition->processor.type == PROCESSOR_GPU) {
      init_table_gpu(partition, pset, false);
    }
  }
  for (int pid = 0; pid < pset->partition_count; pid++) {
    partition_t* partition = &pset->partitions[pid];
    if (partition->processor.type == PROCESSOR_GPU) {
      init_table_gpu(partition, pset, true);
      init_gpu_enable_peer_access(pid, pset);
    }
  }
}

error_t grooves_initialize(partition_set_t* pset) {
  if (pset->partition_count > 1) {
    init_outbox(pset);
    init_inbox(pset);
    init_gpu_state(pset);
  }
  return SUCCESS;
}

PRIVATE void finalize_table_gpu(partition_set_t* pset,
                                grooves_box_table_t* btable, bool inbox) {
  // finalize the tables on the gpu
  for (uint32_t pid = 0; pid < pset->partition_count; pid++) {
    if (btable[pid].count) {
      CALL_CU_SAFE(hipFree(btable[pid].rmt_nbrs));
      if (pset->push_msg_size > 0) {
        CALL_CU_SAFE(hipFree(btable[pid].push_values));
        if (inbox) {
          CALL_CU_SAFE(hipFree(btable[pid].push_values_s));
        }
      }
      if (pset->pull_msg_size > 0) {
        CALL_CU_SAFE(hipFree(btable[pid].pull_values));
        if (inbox) {
          CALL_CU_SAFE(hipFree(btable[pid].pull_values_s));
        }
      }
    }
  }
}

PRIVATE
void finalize_gpu_disable_peer_access(uint32_t pid, partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  partition_t* partition = &pset->partitions[pid];
  for (int remote_pid = (pid + 1) % pcount; remote_pid != pid;
       remote_pid = (remote_pid + 1) % pcount) {
    partition_t* remote_par = &pset->partitions[remote_pid];
    if (remote_par->processor.type == PROCESSOR_GPU &&
        remote_par->processor.id != partition->processor.id) {
      int can_access_peer = 0;
      CALL_CU_SAFE(hipDeviceCanAccessPeer(&can_access_peer,
                                           partition->processor.id,
                                           remote_par->processor.id));
      if (can_access_peer == 1) {
        CALL_CU_SAFE(hipDeviceDisablePeerAccess(remote_par->processor.id));
      }
    }
  }
}

PRIVATE void finalize_outbox(partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  for (int pid = 0; pid < pcount; pid++) {
    partition_t* partition = &pset->partitions[pid];
    if (partition->processor.type == PROCESSOR_GPU) {
      CALL_CU_SAFE(hipSetDevice(partition->processor.id));
      finalize_gpu_disable_peer_access(pid, pset);
      finalize_table_gpu(pset, partition->outbox, false);
    } else {
      assert(partition->processor.type == PROCESSOR_CPU);
      for (uint32_t rmt_pid = 0; rmt_pid < pcount; rmt_pid++) {
        if (rmt_pid == pid) continue;
        if (partition->outbox[rmt_pid].count) {
          free(partition->outbox[rmt_pid].rmt_nbrs);
          if (pset->push_msg_size > 0) {
            totem_free(partition->outbox[rmt_pid].push_values,
                       TOTEM_MEM_HOST_PINNED);
          }
          if (pset->pull_msg_size > 0) {
            totem_free(partition->outbox[rmt_pid].pull_values,
                       TOTEM_MEM_HOST_PINNED);
          }
        }
      }
    }
  }
}

PRIVATE void finalize_inbox(partition_set_t* pset) {
  uint32_t pcount = pset->partition_count;
  for (int pid = 0; pid < pcount; pid++) {
    partition_t* partition = &pset->partitions[pid];
    if (partition->processor.type == PROCESSOR_GPU) {
      CALL_CU_SAFE(hipSetDevice(partition->processor.id));
      finalize_table_gpu(pset, partition->inbox, true);
    } else {
      assert(partition->processor.type == PROCESSOR_CPU);
      for (int rmt_pid = 0; rmt_pid < pcount; rmt_pid++) {
        if (rmt_pid == pid) continue;
        partition_t* remote_par = &pset->partitions[rmt_pid];
        // free only the inboxes that are the destination of an outbox of a gpu-
        // partition. Others that are destinations to a cpu-partition will be
        // freed as an outbox in the source partition.
        if (remote_par->processor.type == PROCESSOR_GPU &&
            partition->inbox[rmt_pid].count) {
          free(partition->inbox[rmt_pid].rmt_nbrs);
          if (pset->push_msg_size > 0) {
            totem_free(partition->inbox[rmt_pid].push_values,
                       TOTEM_MEM_HOST_PINNED);
            totem_free(partition->inbox[rmt_pid].push_values_s,
                       TOTEM_MEM_HOST_PINNED);
          }
          if (pset->pull_msg_size > 0) {
            totem_free(partition->inbox[rmt_pid].pull_values,
                       TOTEM_MEM_HOST_PINNED);
            totem_free(partition->inbox[rmt_pid].pull_values_s,
                       TOTEM_MEM_HOST_PINNED);
          }
        }
      }
    }
  }
}

error_t grooves_finalize(partition_set_t* pset) {
  if (pset->partition_count > 1) {
    finalize_outbox(pset);
    finalize_inbox(pset);
  }
  return SUCCESS;
}

PRIVATE
void launch_communications_setup(partition_set_t* pset,
                                  grooves_direction_t direction, int local_pid,
                                  int remote_pid, void** src, void** dst,
                                  vid_t* count, size_t* msg_size,
                                  hipStream_t** stream) {
  if (direction == GROOVES_PUSH) {
    *msg_size = pset->push_msg_size;
    *src = pset->partitions[local_pid].outbox[remote_pid].push_values;
    *dst = pset->partitions[remote_pid].inbox[local_pid].push_values_s;
    *count = pset->partitions[local_pid].outbox[remote_pid].count;
    if (pset->partitions[local_pid].processor.type == PROCESSOR_GPU) {
      CALL_CU_SAFE(hipSetDevice(pset->partitions[local_pid].processor.id));
      *stream = &pset->partitions[local_pid].streams[1];
    } else {
      CALL_CU_SAFE(hipSetDevice(pset->partitions[remote_pid].processor.id));
      *stream = &pset->partitions[remote_pid].streams[0];
    }
  } else if (direction == GROOVES_PULL) {
    *msg_size = pset->pull_msg_size;
    *src = pset->partitions[local_pid].inbox[remote_pid].pull_values_s;
    *dst = pset->partitions[remote_pid].outbox[local_pid].pull_values;
    *count = pset->partitions[local_pid].inbox[remote_pid].count;

    if (pset->partitions[remote_pid].processor.type == PROCESSOR_GPU) {
      CALL_CU_SAFE(hipSetDevice(pset->partitions[remote_pid].processor.id));
      *stream = &pset->partitions[remote_pid].streams[1];
    } else {
      CALL_CU_SAFE(hipSetDevice(pset->partitions[local_pid].processor.id));
      *stream = &pset->partitions[local_pid].streams[0];
    }
  } else {
    printf("Direction not supported: %s", direction);
    fflush(stdout);
    exit(EXIT_FAILURE);
  }
}

error_t grooves_launch_communications(partition_set_t* pset, int pid,
                                      grooves_direction_t direction) {
  uint32_t pcount = pset->partition_count;
  for (int remote_pid = (pid + 1) % pcount; remote_pid != pid;
       remote_pid = (remote_pid + 1) % pcount) {
    // if both partitions are on the host, then, by design the source
    // partition's outbox is shared with the destination partition's inbox,
    // hence no need to copy data
    if ((pset->partitions[pid].processor.type == PROCESSOR_CPU) &&
        (pset->partitions[remote_pid].processor.type == PROCESSOR_CPU)) {
      continue;
    }

    if ((direction == GROOVES_PULL) &&
        !engine_get_comm_prev(remote_pid)) {
      continue;
    }

    size_t msg_size = 0;
    void* src = NULL;
    void* dst = NULL;
    vid_t count = 0;
    hipStream_t* stream = NULL;
    launch_communications_setup(pset, direction, pid, remote_pid,
                                &src, &dst, &count, &msg_size, &stream);

    if (count == 0) continue;
    CALL_CU_SAFE(hipMemcpyAsync(dst, src, bits_to_bytes(count * msg_size),
                                 hipMemcpyDefault, *stream));
  }
  return SUCCESS;
}

error_t grooves_synchronize(partition_set_t* pset,
                            grooves_direction_t direction) {
  for (int pid = 0; pid < pset->partition_count; pid++) {
    if (pset->partitions[pid].processor.type == PROCESSOR_GPU) {
      CALL_CU_SAFE(hipSetDevice(pset->partitions[pid].processor.id));
      CALL_CU_SAFE(hipStreamSynchronize(pset->partitions[pid].streams[0]));
      CALL_CU_SAFE(hipStreamSynchronize(pset->partitions[pid].streams[1]));
    }
  }
  if (pset->partition_count <= 1) return SUCCESS;
  for (int pid = 0; pid < pset->partition_count; pid++) {
    partition_t* par = &pset->partitions[pid];
    for (int rmt_pid = 0; rmt_pid < pset->partition_count; rmt_pid++) {
      if (rmt_pid == pid) continue;
      // For push-based communication
      void* tmp = par->inbox[rmt_pid].push_values;
      par->inbox[rmt_pid].push_values = par->inbox[rmt_pid].push_values_s;
      par->inbox[rmt_pid].push_values_s = tmp;

      // For pull-based communication
      tmp = par->inbox[rmt_pid].pull_values;
      par->inbox[rmt_pid].pull_values = par->inbox[rmt_pid].pull_values_s;
      par->inbox[rmt_pid].pull_values_s = tmp;
    }
  }
  return SUCCESS;
}
